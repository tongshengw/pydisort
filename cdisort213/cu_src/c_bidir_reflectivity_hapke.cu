#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_bidir_reflectivity_hapke() ============*/

/*
 * Hapke's BRDF model (times Pi/Mu0):
 *   Hapke, B., Theory of reflectance and emittance spectroscopy, Cambridge University Press, 1993,
 * eq. 8.89 on page 233. Parameters are from Fig. 8.15 on page 231, except for w.

  INPUT:

    wvnmlo : Lower wavenumber (inv cm) of spectral interval
    wvnmhi : Upper wavenumber (inv cm) of spectral interval
    mu     : Cosine of angle of reflection (positive)
    mup    : Cosine of angle of incidence (positive)
    dphi   : Difference of azimuth angles of incidence and reflection
                (radians)

  LOCAL VARIABLES:

    iref   : bidirectional reflectance options; 1 - Hapke's BDR model
    b0     : empirical factor to account for the finite size of particles in Hapke's BDR model
    b      : term that accounts for the opposition effect (retroreflectance, hot spot) in Hapke's BDR model
    ctheta : cosine of phase angle in Hapke's BDR model
    gamma  : albedo factor in Hapke's BDR model
    h0     : H(mu0) in Hapke's BDR model
    h      : H(mu) in Hapke's BDR model
    hh     : angular width parameter of opposition effect in Hapke's BDR model
    p      : scattering phase function in Hapke's BDR model
    theta  : phase angle (radians); the angle between incidence and reflection directions in Hapke's BDR model
    w      : single scattering albedo in Hapke's BDR model

   Called by- c_bidir_reflectivity
-------------------------------------------------------------------------*/

double c_bidir_reflectivity_hapke ( double wvnmlo,
				    double wvnmhi,
				    double mu,
				    double mup,
				    double dphi )
{
  double
    b0,b,ctheta,Xgamm,
    h0,h,hh,p,thetah,w;

  ctheta = mu*mup+sqrt((1.-mu*mu)*(1.-mup*mup))*cos(dphi);
  thetah = acos(ctheta);
  p      = 1.+.5*ctheta;
  hh     =  .06;
  b0     = 1.;
  b      = b0*hh/(hh+tan(.5*thetah));
  w      = 0.6;
  Xgamm  = sqrt(1.-w);
  h0     = (1.+2.*mup)/(1.+2.*Xgamm*mup);
  h      = (1.+2.*mu )/(1.+2.*Xgamm*mu );

  return .25*w*((1.+b)*p+h0*h-1.0)/(mu+mup);
}

/*============================= end of c_bidir_reflectivity_hapke() =====*/
