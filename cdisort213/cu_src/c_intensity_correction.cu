#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_intensity_correction() ================*/

/*
       Corrects intensity field by using Nakajima-Tanaka algorithm
       (1988). For more details, see Section 3.6 of STWL NASA report.
                I N P U T   V A R I A B L E S

       ds      Disort state variables
       dither  small multiple of machine precision
       flyr    separated fraction in delta-M method
       layru   index of UTAU in multi-layered system
       lyrcut  logical flag for truncation of computational layer
       ncut    total number of computational layers considered
       oprim   delta-M-scaled single-scatter albedo
       phirad  azimuthal angles in radians
       tauc    optical thickness at computational levels
       taucpr  delta-M-scaled optical thickness
       utaupr  delta-M-scaled version of UTAU

                O U T P U T   V A R I A B L E S

       out->UU  corrected intensity field; UU(IU,LU,J)
                 iu=1,ds->numu; lu=1,ds->ntau; j=1,ds->nphi

                I N T E R N A L   V A R I A B L E S

       ctheta  cosine of scattering angle
       dtheta  angle (degrees) to define aureole region as
                    direction of beam source +/- DTHETA
       phasa   actual (exact) phase function
       phasm   delta-M-scaled phase function
       phast   phase function used in TMS correction; actual phase
                    function divided by (1-FLYR*SSALB)
       pl      ordinary Legendre polynomial of degree l, P-sub-l
       plm1    ordinary Legendre polynomial of degree l-1, P-sub-(l-1)
       plm2    ordinary Legendre polynomial of degree l-2, P-sub-(l-2)
       theta0  incident zenith angle (degrees)
       thetap  emergent angle (degrees)
       ussndm  single-scattered intensity computed by using exact
                   phase function and scaled optical depth
                   (first term in STWL(68a))
       ussp    single-scattered intensity from delta-M method
                   (second term in STWL(68a))
       duims   intensity correction term from IMS method
                   (delta-I-sub-IMS in STWL(A.19))

   Called by- c_disort
   Calls- c_single_scat, c_secondary_scat
 -------------------------------------------------------------------*/

void c_intensity_correction(disort_state  *ds,
                            disort_output *out,
                            double         dither,
                            double        *flyr,
                            int           *layru,
                            int            lyrcut,
                            int            ncut,
                            double        *oprim,
                            double        *phasa,
                            double        *phast,
                            double        *phasm,
                            double        *phirad,
                            double        *tauc,
                            double        *taucpr,
                            double        *utaupr)
{
  register int
    iu,jp,k,lc,ltau,lu;
  double
    ctheta,dtheta,duims,pl,plm1,plm2,
    theta0=0,thetap=0,ussndm,ussp;

  dtheta = 10.;

  /*
   * Start loop over zenith angles
   */
  for (iu = 1; iu <= ds->numu; iu++) {
    if (UMU(iu) < 0.) {
      /*
       * Calculate zenith angles of incident and emerging directions
       */
      theta0 = acos(-ds->bc.umu0)/DEG;
      thetap = acos(UMU(iu))/DEG;
    }
    /*
     * Start loop over azimuth angles
     */
    for (jp = 1; jp <= ds->nphi; jp++) {
      /*
       * Calculate cosine of scattering angle, eq. STWL(4)
       */
      ctheta = -ds->bc.umu0*UMU(iu)+sqrt((1.-SQR(ds->bc.umu0))*(1.-SQR(UMU(iu))))*cos(PHIRAD(jp));
       /*
        * Initialize phase function
        */
      for (lc = 1; lc <= ncut; lc++) {
        PHASA(lc) = 1.;
        PHASM(lc) = 1.;
      }
      /*
       * Initialize Legendre poly. recurrence
       */

      plm1 = 1.;
      plm2 = 0.;
      for (k = 1; k <= ds->nmom; k++) {
        /*
         * Calculate Legendre polynomial of P-sub-l by upward recurrence
         */
        pl   = ((double)(2*k-1)*ctheta*plm1-(double)(k-1)*plm2)/k;
        plm2 = plm1;
        plm1 = pl;

        /*
         * Calculate actual phase function
         */
        for (lc = 1; lc <= ncut; lc++) {
          PHASA(lc) += (double)(2*k+1)*pl*PMOM(k,lc);
        }
        /*
         * Calculate delta-M transformed phase function
         */
        if (k <= ds->nstr-1) {
          for (lc = 1; lc <= ncut; lc++) {
            PHASM(lc) += (double)(2*k+1)*pl*(PMOM(k,lc)-FLYR(lc))/(1.-FLYR(lc));
          }
        }
      }
      /*
       * Apply TMS method, eq. STWL(68)
       */
      for (lc = 1; lc <= ncut; lc++) {
        PHAST(lc) = PHASA(lc)/(1.-FLYR(lc)*SSALB(lc));
      }
      for (lu = 1; lu <= ds->ntau; lu++) {
        if (!lyrcut || LAYRU(lu) < ncut) {
          ussndm        = c_single_scat(dither,LAYRU(lu),ncut,phast,ds->ssalb,taucpr,UMU(iu),ds->bc.umu0,UTAUPR(lu),ds->bc.fbeam);
          ussp          = c_single_scat(dither,LAYRU(lu),ncut,phasm,oprim,    taucpr,UMU(iu),ds->bc.umu0,UTAUPR(lu),ds->bc.fbeam);
          UU(iu,lu,jp) += ussndm-ussp;
        }
      }
      if (UMU(iu) < 0. && fabs(theta0-thetap) <= dtheta) {
        /*
         * Emerging direction is in the aureole (theta0 +/- dtheta).
         * Apply IMS method for correction of secondary scattering below top level.
         */
        ltau = 1;
        if (UTAU(1) <= dither) {
          ltau = 2;
        }
        for (lu = ltau; lu <= ds->ntau; lu++) {
          if(!lyrcut || LAYRU(lu) < ncut) {
            duims         = c_secondary_scat(ds,iu,lu,ctheta,flyr,LAYRU(lu),tauc);
	    UU(iu,lu,jp) -= duims;
          }
        }
      }
    } /* end loop over azimuth angles */
  } /* end loop over zenith angles */

  return;
}

/*============================= end of c_intensity_correction() =========*/
