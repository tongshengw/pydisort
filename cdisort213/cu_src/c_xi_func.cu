#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_xi_func() =============================*/

/*
   Calculates Xi function of eq. STWL (72)

         I N P U T   V A R I A B L E S

   umu1,2    cosine of zenith angle_1, _2
   tau       optical thickness of the layer

   NOTE: Original Fortran version also had argument umu3, but was only
         called for the case umu2 == umu3, so these two arguments are
         fused together here to reduce conditional testing.

   Called by- c_secondary_scat
 -------------------------------------------------------------------*/

double c_xi_func(double umu1,
               double umu2,
               double tau)
{
  double
    exp1,x1;

  x1   = (umu2-umu1)/(umu2*umu1);
  exp1 = exp(-tau/umu1);

  if (x1 != 0.) {
    return ((tau*x1-1.)*exp(-tau/umu2)+exp1)/(x1*x1*umu1*umu2);
  }
  else {
    return tau*tau*exp1/(2.*umu1*umu2);
  }
}

/*============================= end of c_xi_func() ======================*/
