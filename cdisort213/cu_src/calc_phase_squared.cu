#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= calc_phase_squared() ====================*/

/*
   Calculates squared phase function (see BDE)

                I N P U T   V A R I A B L E S

        nphase  number of angles for which original phase function
                   (ds->phase) is defined
        lu        index of user level
        ctheta  cosine of scattering angle
        nf        number of angular phase integration grid point
                     (zenith angle, theta)
        mu_phase  cos(theta) grid of phase function
        phas2     residual phase function
        mu_eq     cos(theta) phase integration grid points,
                     equidistant in abs(f_phas2)
        neg_phas  index whether phas2 is negative
        norm_phas normalization factor for phase integration

                I N T E R N A L   V A R I A B L E S

        pspike2  p"*p", where p" is the residual phase function; return value
	mu1arr
	stheta   corresponding sin of ctheta
	smueq    corresponding sin of mu_eq
	phint    phase function integrated over phi
	scr

   Called by- c_new_secondary_scat
 -------------------------------------------------------------------*/

double calc_phase_squared (int           nphase,
			   int           lu,
			   double        ctheta,
			   int           nf,
			   double       *mu_phase,
			   double       *phas2,
			   double       *mu_eq,
			   int          *neg_phas,
			   double        norm_phas)
{
  int j=0, k=0, it=0;

  double pspike2=0.0, stheta=0.0;
  double smueq=0.0, phint=0.0;

  double mumin=0.0, mumax=0.0;
  int imin=0, imax=0;
  double D=0.0, C=0.0, Dp=0.0, Cp=0.0;
  int cutting=FALSE;

  stheta = sqrt( 1.0 - ctheta * ctheta );

  /* calculate pspike2 */

  /* Note: MU_EQ(j.lu) is mu_1; ctheta is mu; MUP(k) is mu_i in BDE(201X) */


  for (j=1;j<=nf;j++) {

    /* special case: second scattering angle does not depend on
       azimuth of first scattering angle */
    if (ctheta==1.0 || MU_EQ(j,lu)==1.0) {
      it = locate_disort ( mu_phase, nphase, MU_EQ(j,lu)*ctheta ) + 1;
      phint = M_PI * ( PHAS2(it,lu)
		       + ( MU_EQ(j,lu)*ctheta - MUP(it) )
		       / ( MUP (it+1) - MUP(it) )
		       * ( PHAS2(it+1,lu) - PHAS2(it,lu) ) );
      if (ctheta==1.0)
	phint /= 2.0;
    }
    else {
      phint = 0.0;

      smueq = sqrt ( 1. - MU_EQ(j,lu)*MU_EQ(j,lu) );

      /* locate integration borders */
      mumin = ctheta *  MU_EQ(j,lu) - stheta * smueq;
      mumax = ctheta *  MU_EQ(j,lu) + stheta * smueq;

      /* cut where mu_1 = mu_2 */
      if (MU_EQ(j,lu) < mumax) {
	mumax = MU_EQ(j,lu);
	cutting=TRUE;
      }
      else
	cutting=FALSE;

      if (mumin<mumax) {
	imin = locate_disort ( mu_phase, nphase, mumin)+1;
	imax = locate_disort ( mu_phase, nphase, mumax)+1;

	k=imin;
	/* assuming SPF is linear in mu */
	D = ( PHAS2(k+1,lu) - PHAS2(k,lu) ) / ( MUP(k+1) - MUP(k) );
	C = PHAS2(k,lu) - MUP(k) * D;

	phint +=  ( D * ctheta * MU_EQ(j,lu) + C ) * M_PI / 2.0;

	for (k=imin+1;k<=imax;k++) {

	  Dp = ( PHAS2(k+1,lu) - PHAS2(k,lu) ) / ( MUP(k+1) - MUP(k) );
	  Cp = PHAS2(k,lu) - MUP(k) * Dp;

	  phint +=
	    ( Dp - D ) * sqrt ( 1.0 - ctheta * ctheta
				- MU_EQ(j,lu) * MU_EQ(j,lu)
				+ 2.0 * ctheta * MU_EQ(j,lu) * MUP(k)
				- MUP(k) * MUP(k) )
	    + ( ( Dp - D )* ctheta * MU_EQ(j,lu) + Cp - C ) *
	    asin ( ( ctheta * MU_EQ(j,lu) - MUP(k) )
		   / ( smueq * stheta ) );

	  D=Dp;
	  C=Cp;
	}

	if (cutting==TRUE)
	  phint += - D * sqrt ( 1.0 - ctheta * ctheta
			      + 2.0 * MU_EQ(j,lu) * MU_EQ(j,lu) *
			      ( ctheta - 1.0 ) )
	    - ( D * ctheta * MU_EQ(j,lu) + C ) *
	    asin ( ( ctheta - 1.0 ) * MU_EQ(j,lu)
		   / ( smueq * stheta ) );
	else
	  phint += ( D * ctheta * MU_EQ(j,lu) + C ) * M_PI / 2.0;
      }
    }

    if (j==1 || j==nf) {
      if ( NEG_PHAS(j,lu) == TRUE )
	pspike2 = pspike2 - 0.5 * phint;
      else
	pspike2 = pspike2 + 0.5 * phint;
    }
    else {
      if ( NEG_PHAS(j,lu) == TRUE )
	pspike2 = pspike2 - phint;
      else
	pspike2 = pspike2 + phint;
    }

  }

  pspike2 *= norm_phas;

  return pspike2;
}

/*============================= end of calc_phase_squared() =============*/
