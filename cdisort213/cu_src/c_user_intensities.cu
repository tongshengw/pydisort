#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_user_intensities() ====================*/

/*
   Computes intensity components at user output angles for azimuthal
   expansion terms in eq. SD(2), STWL(6)

   I N P U T    V A R I A B L E S:

       ds     :  Disort state variables
       bplanck:  Integrated Planck function for emission from
                 bottom boundary
       cmu    :  Abscissae for Gauss quadrature over angle cosine
       cwt    :  Weights for Gauss quadrature over angle cosine
       delm0  :  Kronecker delta, delta-sub-M0
       emu    :  Surface directional emissivity (user angles)
       expbea :  Transmission of incident beam, EXP(-TAUCPR/UMU0)
       gc     :  Eigenvectors at polar quadrature angles, SC(1)
       gu     :  Eigenvectors interpolated to user polar angles
                    (i.e., G in eq. SC(1) )
       kk     :  Eigenvalues of coeff. matrix in eq. SS(7), STWL(23b)
       layru  :  Layer number of user level UTAU
       ll     :  Constants of integration in eq. SC(1), obtained
                 by solving scaled version of eq. SC(5);
                 exponential term of eq. SC(12) not included
       lyrcut :  Logical flag for truncation of computational layer
       mazim  :  Order of azimuthal component
       ncut   :  Total number of computational layers considered
       nn     :  Order of double-Gauss quadrature (NSTR/2)
       rmu    :  Surface bidirectional reflectivity (user angles)
       taucpr :  Cumulative optical depth (delta-M-Scaled)
       tplanck:  Integrated Planck function for emission from
                 top boundary
       utaupr :  Optical depths of user output levels in delta-M
                 coordinates;  equal to UTAU if no delta-M
       zgu    :  General source function at user angles
       zu     :  Z-sub-zero, Z-sub-one in eq. SS(16) interpolated to user angles from an equation derived from SS(16),
                 Y-sub-zero, Y-sub-one on STWL(26b,a); zu[].zero, zu[].one (see cdisort.h)
       zz     :  Beam source vectors in eq. SS(19), STWL(24b)
       zzg    :  Beam source vectors in eq. KS(10)for a general source constant over a layer
       plk    :  Thermal source vectors z0,z1 by solving eq. SS(16),
                 Y-sub-zero,Y-sub-one in STWL(26)
       zbeam  :  Incident-beam source vectors


    O U T P U T    V A R I A B L E S:

       uum    :  Azimuthal components of the intensity in eq. STWJ(5),
                 STWL(6)

    I N T E R N A L    V A R I A B L E S:

       bnddir :  Direct intensity down at the bottom boundary
       bnddfu :  Diffuse intensity down at the bottom boundary
       bndint :  Intensity attenuated at both boundaries, STWJ(25-6)
       dtau   :  Optical depth of a computational layer
       lyrend :  End layer of integration
       lyrstr :  Start layer of integration
       palint :  Intensity component from parallel beam
       plkint :  Intensity component from planck source
       wk     :  Scratch vector for saving exp evaluations

       All the exponential factors (exp1, expn,... etc.)
       come from the substitution of constants of integration in
       eq. SC(12) into eqs. S1(8-9).  They all have negative
       arguments so there should never be overflow problems.

   Called by- c_disort
 -------------------------------------------------------------------*/

void c_user_intensities(disort_state   *ds,
                        double          bplanck,
                        double         *cmu,
                        double         *cwt,
                        double          delm0,
                        double         *dtaucpr,
                        double         *emu,
                        double         *expbea,
                        double         *gc,
                        double         *gu,
                        double         *kk,
                        int            *layru,
                        double         *ll,
                        int             lyrcut,
                        int             mazim,
                        int             ncut,
                        int             nn,
                        double         *rmu,
                        double         *taucpr,
                        double          tplanck,
                        double         *utaupr,
                        double         *wk,
			disort_triplet *zbu,
                        double         *zbeam,
			disort_pair    *zbeamsp,
                        double         *zbeama,
                        double         *zgu,
                        disort_pair    *zu,
                        double         *zz,
                        double         *zzg,
                        disort_pair    *plk,
                        double         *uum)
{
  register int
    negumu,
    iq,iu,jq,lc,lu,lyrend,lyrstr,lyu;
  double
    alfa,bnddfu,bnddir,bndint,
    denom,dfuint,dtau,dtau1,dtau2,
    exp0=0,exp1=0,exp2=0,expn,
    f0n,f1n,fact,genint,
    palint,plkint,sgn;

  /*
   * Incorporate constants of integration into interpolated eigenvectors
   */
  for (lc = 1; lc <= ncut; lc++) {
    for (iq = 1; iq <= ds->nstr; iq++) {
      for (iu = 1; iu <= ds->numu; iu++) {
        GU(iu,iq,lc) *= LL(iq,lc);
      }
    }
  }

  /*
   * Loop over levels at which intensities are desired ('user output levels')
   */
  for (lu = 1; lu <= ds->ntau; lu++) {
    if (ds->bc.fbeam > 0.) {
      exp0 = exp(-UTAUPR(lu)/ds->bc.umu0);
    }
    lyu = LAYRU(lu);
    /*
     * Loop over polar angles at which intensities are desired
     */
    for (iu = 1; iu <= ds->numu; iu++) {
      if (lyrcut && lyu > ncut) {
        continue;
      }
      negumu = (UMU(iu) < 0.);
      if (negumu) {
        lyrstr = 1;
        lyrend = lyu-1;
        sgn    = -1.;
      }
      else {
        lyrstr = lyu+1;
        lyrend = ncut;
        sgn    = 1.;
      }

      /*
       * For downward intensity, integrate from top to LYU-1 in eq. S1(8); for upward,
       * integrate from bottom to LYU+1 in S1(9)
       */
      genint = 0.;
      palint = 0.;
      plkint = 0.;
      for (lc = lyrstr; lc <= lyrend; lc++) {
        dtau = DTAUCPR(lc);
        exp1 = exp((UTAUPR(lu)-TAUCPR(lc-1))/UMU(iu));
        exp2 = exp((UTAUPR(lu)-TAUCPR(lc  ))/UMU(iu));

        if (ds->flag.planck && mazim == 0) {
          /*
           * Eqs. STWL(36b,c, 37b,c)
           */
          f0n     = sgn*(exp1-exp2);
          f1n     = sgn*((TAUCPR(lc-1)+UMU(iu))*exp1
                        -(TAUCPR(lc  )+UMU(iu))*exp2);
          plkint += Z0U(iu,lc)*f0n+Z1U(iu,lc)*f1n;
        }

        if (ds->bc.fbeam > 0.) {
	  if ( ds->flag.spher == TRUE ) {
	    denom  =  sgn*1.0/(ZBAU(iu,lc)*UMU(iu)+1.0);
	    palint += (ZB0U(iu,lc)*denom*(exp(-ZBAU(iu,lc)*TAUCPR(lc-1)) *exp1
					  -exp(-ZBAU(iu,lc)*TAUCPR(lc)) *exp2 )
		       +ZB1U(iu,lc)*denom*((TAUCPR(lc-1)+sgn*denom*UMU(iu))
					   *exp(-ZBAU(iu,lc)*TAUCPR(lc-1)) *exp1
					   -(TAUCPR(lc)+sgn*denom*UMU(iu) )
					   *exp(-ZBAU(iu,lc)*TAUCPR(lc))*exp2));
	  }
	  else {
	    denom = 1.+UMU(iu)/ds->bc.umu0;
	    if (fabs(denom) < 0.0001) {
	      /*
	       * L'Hospital limit
	       */
	      expn = (dtau/ds->bc.umu0)*exp0;
	    }
	    else {
	      expn = (exp1*EXPBEA(lc-1)
		      -exp2*EXPBEA(lc  ))*sgn/denom;
	    }
	    palint += ZBEAM(iu,lc)*expn;
	  }
        }
	if ( ds->flag.general_source ) {
          genint += ZGU(iu,lc)*sgn*(exp1-exp2);
	}
        /*
         * KK is negative
         */
        for (iq = 1; iq <= nn; iq++) {
          WK(iq) = exp(KK(iq,lc)*dtau);
          denom  = 1.+UMU(iu)*KK(iq,lc);
          if (fabs(denom) < 0.0001) {
            /*
             * L'Hospital limit
             */
            expn = (dtau/UMU(iu))*exp2;
          }
          else {
            expn = sgn*(exp1*WK(iq)-exp2)/denom;
          }
          palint += GU(iu,iq,lc)*expn;
        }

        /*
         * KK is positive
         */
        for (iq = nn+1; iq <= ds->nstr; iq++) {
          denom = 1.+UMU(iu)*KK(iq,lc);
          if (fabs(denom) < 0.0001) {
            /*
             * L'Hospital limit
             */
            expn = -(dtau/UMU(iu))*exp1;
          }
          else {
            expn = sgn*(exp1-exp2*WK(ds->nstr+1-iq))/denom;
          }
          palint += GU(iu,iq,lc)*expn;
        }
      }

      /*
       * Calculate contribution from user output level to next computational level
       */
      dtau1 = UTAUPR(lu)-TAUCPR(lyu-1);
      dtau2 = UTAUPR(lu)-TAUCPR(lyu  );

      if ((fabs(dtau1) >= 1.e-6 || !negumu) && (fabs(dtau2) >= 1.e-6 ||  negumu)) {
        if(negumu) {
          exp1 = exp(dtau1/UMU(iu));
        }
        else {
          exp2 = exp(dtau2/UMU(iu));
        }
        if (ds->bc.fbeam > 0.) {
	  if ( ds->flag.spher == TRUE ) {
	    if ( negumu ) {
	      expn = exp1;
	      alfa = ZBAU(iu,lyu);
	      denom = (-1.0/(alfa*UMU(iu)+1.));
	      palint += ZB0U(iu,lyu)*denom*(-exp(-alfa*UTAUPR(lu))
					    + expn*exp(-alfa*TAUCPR(lyu-1)))
		+ZB1U(iu,lyu)*denom*( -(UTAUPR(lu)-UMU(iu)*denom)*exp(-alfa*UTAUPR(lu))
				      +(TAUCPR(lyu-1)-UMU(iu)*denom)*expn*exp(-alfa*TAUCPR(lyu-1)));
	    }
	    else {
	      expn = exp2;
	      alfa = ZBAU(iu,lyu);
	      denom = (1.0/(alfa*UMU(iu)+1.0));
	      palint += ZB0U(iu,lyu)*denom*(exp(-alfa*UTAUPR(lu))
					    -exp(-alfa*TAUCPR(lyu))*expn)
		+ZB1U(iu,lyu)*denom*( (UTAUPR(lu) +UMU(iu)*denom)*exp(-alfa*UTAUPR(lu))
				      -(TAUCPR(lyu)+UMU(iu)*denom)*exp(-alfa*TAUCPR(lyu))*expn );
	    }
	  }
	  else {
	    denom = 1.+UMU(iu)/ds->bc.umu0;
	    if (fabs(denom) < 0.0001) {
	      expn = (dtau1/ds->bc.umu0)*exp0;
	    }
	    else if (negumu) {
	      expn = (exp0-EXPBEA(lyu-1)*exp1)/denom;
	    }
	    else {
	      expn = (exp0-EXPBEA(lyu  )*exp2)/denom;
	    }
	    palint += ZBEAM(iu,lyu)*expn;
	  }
        }
	if ( ds->flag.general_source ) {
          if (negumu) {
            expn = exp1;
          }
          else {
            expn = exp2;
          }
          genint += ZGU(iu,lyu)*(1.-expn);
	}
        /*
         * KK is negative
         */
        dtau = DTAUCPR(lyu);
        for (iq = 1; iq <= nn; iq++) {
          denom = 1.+UMU(iu)*KK(iq,lyu);
          if (fabs(denom) < 0.0001) {
            expn = -dtau2/UMU(iu)*exp2;
          }
          else if (negumu) {
            expn = (exp(-KK(iq,lyu)*dtau2)
                   -exp( KK(iq,lyu)*dtau )*exp1)/denom;
          }
          else {
            expn = (exp(-KK(iq,lyu)*dtau2)-exp2)/denom;
          }
          palint += GU(iu,iq,lyu)*expn;
        }

        /*
         * KK is positive
         */
        for (iq = nn+1; iq <= ds->nstr; iq++) {
          denom = 1.+UMU(iu)*KK(iq,lyu);
          if (fabs(denom) < 0.0001) {
            expn = -(dtau1/UMU(iu))*exp1;
          }
          else if (negumu) {
            expn = (exp(-KK(iq,lyu)*dtau1)-exp1)/denom;
          }
          else {
            expn = (exp(-KK(iq,lyu)*dtau1)
                   -exp(-KK(iq,lyu)*dtau )*exp2)/denom;
          }
          palint += GU(iu,iq,lyu)*expn;
        }

        if (ds->flag.planck && mazim == 0) {
          /*
           * Eqs. STWL (35-37) with tau-sub-n-1 replaced by tau for upward, and
           * tau-sub-n replaced by tau for downward directions
           */
          if (negumu) {
            expn = exp1;
            fact = TAUCPR(lyu-1)+UMU(iu);
          }
          else {
            expn = exp2;
            fact = TAUCPR(lyu  )+UMU(iu);
          }
          f0n     = 1.-expn;
          f1n     = UTAUPR(lu)+UMU(iu)-fact*expn;
          plkint += Z0U(iu,lyu)*f0n+Z1U(iu,lyu)*f1n;
        }
      }

      /*
       * Calculate intensity components attenuated at both boundaries.
       * NOTE: no azimuthal intensity component for isotropic surface
       */
      bndint = 0.;
      if (negumu && mazim == 0) {
        bndint = (ds->bc.fisot+tplanck)*exp(UTAUPR(lu)/UMU(iu));
      }
      else if (!negumu) {
        if (lyrcut || ( ds->flag.lamber && mazim > 0 ) ) {
          UUM(iu,lu) = palint+plkint;
          continue;
        }

        for (jq = nn+1; jq <= ds->nstr; jq++) {
          WK(jq) = exp(-KK(jq,ds->nlyr)*DTAUCPR(ds->nlyr));
        }
        bnddfu = 0.;
        for (iq = nn; iq >= 1; iq--) {
          dfuint = 0.;
          for (jq = 1; jq <= nn; jq++) {
            dfuint += GC(iq,jq,ds->nlyr)*LL(jq,ds->nlyr);
          }
          for (jq= nn+1; jq <= ds->nstr; jq++) {
            dfuint += GC(iq,jq,ds->nlyr)*LL(jq,ds->nlyr)*WK(jq);
          }
          if (ds->bc.fbeam > 0.) {
	    if ( ds->flag.spher == TRUE ) {
	      dfuint += exp(-ZBEAMA(ds->nlyr)*TAUCPR(ds->nlyr)) *
		(ZBEAM0(iq,ds->nlyr)+ZBEAM1(iq,ds->nlyr)*TAUCPR(ds->nlyr));
	    }
	    else {
	      dfuint += ZZ(iq,ds->nlyr)*EXPBEA(ds->nlyr);
	    }
          }
	  if ( ds->flag.general_source ) {
	    dfuint += ZZG(iq,ds->nlyr);
	  }
          dfuint += delm0*(ZPLK0(iq,ds->nlyr)+ZPLK1(iq,ds->nlyr)*TAUCPR(ds->nlyr));
          bnddfu += (1.+delm0)*RMU(iu,nn+1-iq)*CMU(nn+1-iq)*CWT(nn+1-iq)*dfuint;
        }
        bnddir = 0.;
        if (ds->bc.fbeam > 0. || ds->bc.umu0 >0.) {
          bnddir = ds->bc.umu0*ds->bc.fbeam/M_PI*RMU(iu,0)*EXPBEA(ds->nlyr);
        }
        bndint = (bnddfu+bnddir+delm0*EMU(iu)*bplanck+ds->bc.fluor)*exp((UTAUPR(lu)-TAUCPR(ds->nlyr))/UMU(iu));
      }
      UUM(iu,lu) = palint+plkint+bndint+genint;
    }
  }

  return;
}

/*============================= end of c_user_intensities() =============*/
