#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_disort_set() ==========================*/

/*
    Perform miscellaneous setting-up operations

    I N P U T  V A R I A B L E S

       ds         Disort state variables
       deltam
       tauc

    O U T P U T     V A R I A B L E S:

       If ds->flag.usrtau is FALSE
       ds->ntau
       ds->utau

       If ds->flag.usrang is FALSE
       ds->numu
       ds->umu

       cmu,cwt     computational polar angles and corresponding quadrature weights
       dtaucpr
       expbea      transmission of direct beam
       flyr        separated fraction in delta-m method
       gl          phase function legendre coefficients multiplied by (2l+1) and single-scatter albedo
       layru       computational layer in which utau falls
       lyrcut      flag as to whether radiation will be zeroed below layer ncut
       ncut        computational layer where absorption optical depth first exceeds  abscut
       nn          ds->nstr/2
       oprim       delta-m-scaled single-scatter albedo
       taucpr      delta-m-scaled optical depth
       utaupr      delta-m-scaled version of  utau

   Called by- c_disort
   Calls- c_gaussian_quadrature, c_errmsg

 ---------------------------------------------------------------------*/

void c_disort_set(disort_state *ds,
                  double       *ch,
                  double       *chtau,
                  double       *cmu,
                  double       *cwt,
                  int           deltam,
                  double       *dtaucpr,
                  double       *expbea,
                  double       *flyr,
                  double       *gl,
                  int          *layru,
                  int          *lyrcut,
                  int          *ncut,
                  int          *nn,
                  int          *corint,
                  double       *oprim,
                  double       *tauc,
                  double       *taucpr,
                  double       *utaupr,
                  emission_func_t emi_func)
{
  register int
    iq,iu,k,lc,lu;
  const double
    abscut = 10.;
  double
    abstau,chtau_tmp,f,taup,zenang;

  if (!ds->flag.usrtau) {
   /*
    * Set output levels at computational layer boundaries
    */
    for (lc = 0;  lc <= ds->ntau-1; lc++) {
      UTAU(lc+1) = TAUC(lc);
    }
  }

  /*
   * Apply delta-M scaling and move description of computational layers to local variables
   */
  TAUCPR(0) = 0.;
  abstau    = 0.;
  for (lc = 1; lc <= ds->nlyr; lc++) {
    PMOM(0,lc)  = 1.;
    if (abstau < abscut) {
      *ncut = lc;
    }
    abstau += (1.-SSALB(lc))*DTAUC(lc);
    if (!deltam) {
      OPRIM(lc)   = SSALB(lc);
      DTAUCPR(lc) = DTAUC(lc);
      TAUCPR(lc)  = TAUC(lc);
      for (k = 0; k <= ds->nstr-1; k++) {
        GL(k,lc)  = (double)(2*k+1)*OPRIM(lc)*PMOM(k,lc);
      }
      f = 0.;
    }
    else {
      /*
       * Do delta-M transformation
       */
      f           = PMOM(ds->nstr,lc);
      OPRIM(lc)   = SSALB(lc)*(1.-f)/(1.-f*SSALB(lc));
      DTAUCPR(lc) = (1.-f*SSALB(lc))*DTAUC(lc);
      TAUCPR(lc)  = TAUCPR(lc-1)+DTAUCPR(lc);
      for (k = 0; k <= ds->nstr-1; k++) {
        GL(k,lc)  = (double)(2*k+1)*OPRIM(lc)*(PMOM(k,lc)-f)/(1.-f);
      }
    }

    FLYR(lc)   = f;
  }

  /*
   * Calculate Chapman function if spherical geometry, set expbea and
   * ch for beam source.
   */
  if( (ds->flag.ibcnd == GENERAL_BC && ds->bc.fbeam > 0.) ||
      (ds->flag.ibcnd == GENERAL_BC && ds->flag.general_source )) {

    CHTAU(0)  = 0.;
    EXPBEA(0) = 1.;
    zenang    = acos(ds->bc.umu0)/DEG;

    if( ds->flag.spher == TRUE && ds->bc.umu0 < 0. ) {
      EXPBEA(0) = exp(-c_chapman(1,0.,tauc,ds->nlyr,ds->zd,
				 ds->dtauc,zenang,ds->radius));
    }
    if ( ds->flag.spher == TRUE ) {
      for (lc = 1; lc <= *ncut; lc++) {
        taup        = TAUCPR(lc-1) + DTAUCPR(lc)/2.;
	/* Need Chapman function at top (0.0) and middle (0.5) of layer */
        CHTAU(lc  ) = c_chapman(lc, 0.,   taucpr,ds->nlyr,ds->zd,
				dtaucpr,zenang,ds->radius);
        chtau_tmp   = c_chapman(lc, 0.5,  taucpr,ds->nlyr,ds->zd,
				dtaucpr,zenang,ds->radius);
        CH(lc)      = taup/chtau_tmp;
        EXPBEA(lc)  = exp(-CHTAU(lc));
      }
    }
    else {
      for (lc = 1; lc <= *ncut; lc++) {
        CH(lc)     = ds->bc.umu0;
        EXPBEA(lc) = exp(-TAUCPR(lc)/ds->bc.umu0);
      }
    }
  }
  else {
    for (lc = 1; lc <= *ncut; lc++) {
      EXPBEA(lc) = 0.;
    }
  }

  /*
   * If no thermal emission, cut off medium below absorption optical depth = abscut ( note that
   * delta-M transformation leaves absorption optical depth invariant ).  Not worth the
   * trouble for one-layer problems, though.
   */
  *lyrcut = FALSE;
  if (abstau >= abscut && !ds->flag.planck && ds->flag.ibcnd != SPECIAL_BC && ds->nlyr > 1) {
    *lyrcut = TRUE;
  }
  if(!*lyrcut) *ncut = ds->nlyr;

  /*
   * Set arrays defining location of user output levels within delta-M-scaled computational mesh
   */
  for (lu = 1; lu <= ds->ntau; lu++) {
    for (lc = 1; lc < ds->nlyr; lc++) {
      if (UTAU(lu) >= TAUC(lc-1) && UTAU(lu) <= TAUC(lc)) {
        break;
      }
    }

    UTAUPR(lu) = UTAU(lu);
    if (deltam) {
      UTAUPR(lu) = TAUCPR(lc-1)+(1.-SSALB(lc)*FLYR(lc))*(UTAU(lu)-TAUC(lc-1));
    }
    LAYRU(lu) = lc;
  }

  /*
   * Calculate computational polar angle cosines and associated quadrature weights for Gaussian
   * quadrature on the interval (0,1) (upward)
   */
  *nn = ds->nstr/2;
  c_gaussian_quadrature(*nn,cmu,cwt);

  /*
   * Downward (neg) angles and weights
   */
  for (iq = 1; iq <= *nn; iq++) {
    CMU(iq+*nn) = -CMU(iq);
    CWT(iq+*nn) =  CWT(iq);
  }

  if (ds->flag.ibcnd == GENERAL_BC && ds->bc.fbeam > 0.) {
    /*
     * Compare beam angle to comput. angles
     */
    for (iq = 1; iq <= *nn; iq++) {
      if (fabs(ds->bc.umu0-CMU(iq))/fabs(ds->bc.umu0) < 1.e-4) {
        // suppress error msg by adding a small difference
        ds->bc.umu0 = (1. + 1.E-4)*CMU(iq);
        // c_errmsg("cdisort_set--beam angle=computational angle; change ds.nstr",DS_ERROR);
      }
    }
  }

  if (!ds->flag.usrang || ds->flag.onlyfl) {
    /*
     * Set output polar angles to computational polar angles
     */
    for (iu = 1; iu <= *nn; iu++) {
      UMU(iu) = -CMU(*nn+1-iu);
    }
    for (iu = *nn+1; iu <=ds->nstr; iu++) {
      UMU(iu) =  CMU(iu-*nn);
    }
  }

  if (ds->flag.usrang && ds->flag.ibcnd == SPECIAL_BC) {
    /*
     * Shift positive user angle cosines to upper locations and put negatives in lower locations
     */
    for (iu = 1; iu <= ds->numu/2; iu++) {
      UMU(iu+ds->numu/2) = UMU(iu);
    }
    for (iu = 1; iu <= ds->numu/2; iu++) {
      UMU(iu) = -UMU((ds->numu/2)+1-iu);
    }
  }

  return;
}

/*============================= end of c_disort_set() ===================*/
