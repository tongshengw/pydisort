#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_self_test() ===========================*/

/*
 * If  compare is FALSE, set up self-test disort_state ds_test.
 * If  compare is TRUE, compare self-test results with correct
 * answers, abort if error, and free self-test memory.
 *
 * (See file 'DISORT.txt' for variable definitions.)
 *
 *    I N T E R N A L    V A R I A B L E S:
 *
 *       acc        Relative accuracy required for passing self-test
 *       error      Relative errors in DISORT output variables
 *       ok         Logical variable for determining failure of self-test
 *
 * Called by- c_disort
 * Calls- c_errmsg
 */

void c_self_test(int            compare,
                 int           *prntu0,
                 disort_state  *ds,
                 disort_output *out)
{
  const double
    acc = 1.e-4;
  int
    i,ok;
  double
    error;

  if(compare == FALSE) {
    for (i = 0; i < 5; i++) {
      ds->flag.prnt[i] = FALSE;
    }
    ds->flag.ibcnd     = GENERAL_BC;
    ds->flag.usrang    = TRUE;
    ds->flag.usrtau    = TRUE;
    ds->flag.lamber    = TRUE;
    ds->flag.onlyfl    = FALSE;
    ds->flag.planck    = TRUE;
    ds->flag.quiet     = QUIET;
    ds->flag.spher     = FALSE;
    ds->flag.general_source = FALSE;
    ds->flag.brdf_type = BRDF_NONE;
    ds->flag.intensity_correction     = TRUE;
    ds->flag.old_intensity_correction = TRUE;
    ds->flag.output_uum=FALSE;

    ds->nstr = 4;
    ds->nlyr = 1;
    ds->nmom = 4;
    ds->numu = 1;
    ds->ntau = 1;
    ds->nphi = 1;

    /* Allocate memory for self test */
    c_disort_state_alloc(ds);
    c_disort_out_alloc(ds,out);

    ds->accur  = 1.e-4;
    ds->wvnmlo =     0.;
    ds->wvnmhi = 50000.;

    ds->bc.fbeam  =  M_PI;
    ds->bc.umu0   =   .866;
    ds->bc.phi0   =   0.;
    ds->bc.fisot  =   1.;
    ds->bc.fluor  =   0.;
    ds->bc.albedo =    .7;
    ds->bc.ttemp  = 100.;
    ds->bc.btemp  = 300.;
    ds->bc.temis  =    .8;

    TEMPER(0) = 210.;
    TEMPER(1) = 200.;

    DTAUC(1)  = 1.;
    SSALB(1)  =  .9;

    /* Haze L moments */
    PMOM(0,1) = 1.;
    PMOM(1,1) =  .8042;
    PMOM(2,1) =  .646094;
    PMOM(3,1) =  .481851;
    PMOM(4,1) =  .359056;

    UMU(1)  =  0.5;
    UTAU(1) =  0.5;
    PHI(1)  = 90.0;

    return;
  }
  else if (compare == TRUE) {
    /*
     * Compare test case results with correct answers and abort if bad
     */
    ok = TRUE;

    error = (out->uu[0]-47.865571)/47.865571;
    if (fabs(error) > acc) {
      ok = FALSE;
      printf("Output variable uu differed by %g percent from correct value.\n",100.*error);
    }

    error = (out->rad[0].rfldir-1.527286)/1.527286;
    if (fabs(error) > acc) {
      ok = FALSE;
      printf("Output variable rfldir differed by %g percent from correct value.\n",100.*error);
    }

    error = (out->rad[0].rfldn-28.372225)/28.372225;
    if (fabs(error) > acc) {
      ok = FALSE;
      printf("Output variable rfldn differed by %g percent from correct value.\n",100.*error);
    }

    error = (out->rad[0].flup-152.585284)/152.585284;
    if (fabs(error) > acc) {
      ok = FALSE;
      printf("Output variable flup differed by %g percent from correct value.\n",100.*error);
    }

    /* Free allocated memory for self test */
    c_disort_out_free(ds,out);
    c_disort_state_free(ds);

    if (!ok) {
      c_errmsg("DISORT--self-test failed",DS_ERROR);
    }

    return;
  }
  else {
    printf("**error--self_test(): compare=%d not recognized\n",compare);
    __trap();
  }
}

/*============================= end of c_self_test() =====================*/
