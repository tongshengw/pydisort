#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_fluxes() ==============================*/

/*
    Calculates the radiative fluxes, mean intensity, and flux
    derivative with respect to optical depth from the m=0 intensity
    components (the azimuthally-averaged intensity)

    I N P U T    V A R I A B L E S:

       ds       :  Disort state variables
       cmu      :  Abscissae for Gauss quadrature over angle cosine
       cwt      :  Weights for Gauss quadrature over angle cosine
       gc       :  Eigenvectors at polar quadrature angles, SC(1)
       kk       :  Eigenvalues of coeff. matrix in eq. SS(7), STWL(23b)
       layru    :  Layer number of user level UTAU
       ll       :  Constants of integration in eq. SC(1), obtained by solving scaled version of eq. SC(5);
                   exponential term of eq. SC(12) not included
       lyrcut   :  Logical flag for truncation of comput. layer
       ncut     :  Number of computational layer where absorption optical depth exceeds ABSCUT
       nn       :  Order of double-Gauss quadrature (NSTR/2)
       prntu0   :  TRUE, print azimuthally-averaged intensity at quadrature angles
       taucpr   :  Cumulative optical depth (delta-M-scaled)
       utaupr   :  Optical depths of user output levels in delta-M coordinates;  equal to UTAU if no delta-M
       xr       :  Expansion of thermal source function in eq. SS(14,16), STWL(24c); xr[].zero, xr[].one (see cdisort.h)
       zz       :  Beam source vectors in eq. SS(19), STWL(24b)
       zzg      :  Beam source vectors in eq. KS(10)for a general source constant over a layer
       plk      :  Thermal source vectors z0,z1 by solving eq. SS(16), Y0,Y1 in STWL(26b,a);
                   plk[].zero, plk[].one (see cdisort.h)

    O U T P U T    V A R I A B L E S:

       out      : Disort output variables
       u0c      :  Azimuthally averaged intensities (at polar quadrature angles)

    I N T E R N A L    V A R I A B L E S:

       dirint   :  Direct intensity attenuated
       fdntot   :  Total downward flux (direct + diffuse)
       fl       :  fl[].zero: 'fldir' = direct-beam flux (delta-M scaled), fl[].one 'fldn' = diffuse down-flux (delta-M scaled)
       fnet     :  Net flux (total_down-diffuse_up)
       fact     :  EXP(- UTAUPR/UMU0)
       plsorc   :  Planck source function (thermal)
       zint     :  Intensity of m = 0 case, in eq. SC(1)

   Called by- c_disort
 -------------------------------------------------------------------*/

void c_fluxes(disort_state  *ds,
              disort_output *out,
              double        *ch,
              double        *cmu,
              double        *cwt,
              double        *gc,
              double        *kk,
              int           *layru,
              double        *ll,
              int            lyrcut,
              int            ncut,
              int            nn,
              int            prntu0,
              double        *taucpr,
              double        *utaupr,
              disort_pair   *xr,
              disort_pair   *zbeamsp,
              double        *zbeama,
              double        *zz,
              double        *zzg,
              disort_pair   *plk,
              disort_pair   *fl,
              double        *u0c)
{
  register int
    iq,jq,lu,lyu;
  double
    ang1,ang2,dirint,
    fact=0,fdntot,fnet,plsorc,zint;

  if (ds->flag.prnt[1]) {
    printf("\n\n                     <----------------------- FLUXES ----------------------->\n"                   "   Optical  Compu    Downward    Downward    Downward      Upward                    Mean      Planck   d(Net Flux)\n"
                   "     Depth  Layer      Direct     Diffuse       Total     Diffuse         Net   Intensity      Source   / d(Op Dep)\n");
  }

  /*
   * Zero DISORT output arrays
   */
  memset(u0c,0,ds->ntau*ds->nstr*sizeof(double));
  memset(fl,0,ds->ntau*sizeof(disort_pair));

  /*
   * Loop over user levels
   */
  for (lu = 1; lu <= ds->ntau; lu++) {
    lyu = LAYRU(lu);

    if (lyrcut && lyu > ncut) {
      /*
       * No radiation reaches this level
       */
      fdntot = 0.;
      fnet   = 0.;
      plsorc = 0.;
      if (ds->flag.prnt[1]) {
        printf("%10.4f%7d%12.3e%12.3e%12.3e%12.3e%12.3e%12.3e%12.3e%14.3e\n",                        UTAU(lu),lyu,RFLDIR(lu),RFLDN(lu),fdntot,FLUP(lu),fnet,UAVG(lu),plsorc,DFDT(lu));
      }
      continue;
    }

    if (ds->bc.fbeam > 0.) {
      if ( ds->flag.spher == TRUE ) {
	fact         = exp( - UTAUPR(lu) / CH(lyu) );
	RFLDIR( lu ) = fabs(ds->bc.umu0)*ds->bc.fbeam*
	  exp( - UTAU( lu ) / CH(lyu) );
      }
      else {
	fact       = exp(-UTAUPR(lu)/ds->bc.umu0);
	RFLDIR(lu) = ds->bc.umu0*ds->bc.fbeam*exp(-UTAU(lu)/ds->bc.umu0);
      }
      dirint     = ds->bc.fbeam*fact;
      FLDIR(lu)  = ds->bc.umu0*ds->bc.fbeam*fact;
    }
    else {
      dirint     = 0.;
      FLDIR(lu)  = 0.;
      RFLDIR(lu) = 0.;
    }

    for (iq = 1; iq <= nn; iq++) {
      zint = 0.;
      for (jq = 1; jq <= nn; jq++) {
        zint += GC(iq,jq,lyu)*LL(jq,lyu)*exp(-KK(jq,lyu)*(UTAUPR(lu)-TAUCPR(lyu  )));
      }
      for (jq = nn+1; jq <= ds->nstr; jq++) {
        zint += GC(iq,jq,lyu)*LL(jq,lyu)*exp(-KK(jq,lyu)*(UTAUPR(lu)-TAUCPR(lyu-1)));
      }

      U0C(iq,lu) = zint;
      if (ds->bc.fbeam > 0. ) {
	if ( ds->flag.spher == TRUE ) {
	  U0C(iq,lu) += exp(-ZBEAMA(lyu)*UTAUPR(lu))*
	    ( ZBEAM0(iq,lyu)+ZBEAM1(iq,lyu)*UTAUPR(lu) );
	}
	else {
	  U0C(iq,lu) += ZZ(iq,lyu)*fact;
	}
      }
      if ( ds->flag.general_source == TRUE ) {
	U0C(iq,lu) += ZZG(iq,lyu);
      }
      U0C(iq,lu) += ZPLK0(iq,lyu)+ZPLK1(iq,lyu)*UTAUPR(lu);
      UAVG(lu)   += CWT(nn+1-iq)*U0C(iq,lu);
      UAVGDN(lu) += CWT(nn+1-iq)*U0C(iq,lu);
      FLDN(lu)   += CWT(nn+1-iq)*U0C(iq,lu)*CMU(nn+1-iq);
    }

    for (iq = nn+1; iq <= ds->nstr; iq++) {
      zint = 0.;
      for (jq = 1; jq <= nn; jq++) {
        zint += GC(iq,jq,lyu)*LL(jq,lyu)*exp(-KK(jq,lyu)*(UTAUPR(lu)-TAUCPR(lyu  )));
      }
      for (jq = nn+1; jq <= ds->nstr; jq++) {
        zint += GC(iq,jq,lyu)*LL(jq,lyu)*exp(-KK(jq,lyu)*(UTAUPR(lu)-TAUCPR(lyu-1)));
      }

      U0C(iq,lu) = zint;
      if (ds->bc.fbeam > 0.) {
	if ( ds->flag.spher == TRUE ) {
	  U0C(iq,lu) += exp(-ZBEAMA(lyu)*UTAUPR(lu))*
	    ( ZBEAM0(iq,lyu)+ZBEAM1(iq,lyu)*UTAUPR(lu) );
	}
	else {
	  U0C(iq,lu) += ZZ(iq,lyu)*fact;
	}
      }
      if ( ds->flag.general_source == TRUE ) {
	U0C(iq,lu) += ZZG(iq,lyu);
      }
      U0C(iq,lu) += ZPLK0(iq,lyu)+ZPLK1(iq,lyu)*UTAUPR(lu);
      UAVG(lu)   += CWT(iq-nn)*U0C(iq,lu);
      UAVGUP(lu) += CWT(iq-nn)*U0C(iq,lu);
      FLUP(lu)   += CWT(iq-nn)*U0C(iq,lu)*CMU(iq-nn);
    }
    FLUP(lu)  *= 2.*M_PI;
    FLDN(lu)  *= 2.*M_PI;
    fdntot     = FLDN(lu)+FLDIR(lu);
    fnet       = fdntot-FLUP(lu);
    RFLDN(lu)  = fdntot-RFLDIR(lu);
    UAVG(lu)   = (2.*M_PI*UAVG(lu)+dirint)/(4.*M_PI);
    UAVGSO(lu) =  dirint / (4.*M_PI);
    UAVGDN(lu) = (2.*M_PI*UAVGDN(lu) )/(4.*M_PI);
    UAVGUP(lu) = (2.*M_PI*UAVGUP(lu) )/(4.*M_PI);
    plsorc     = XR0(lyu)+XR1(lyu)*UTAUPR(lu);
    DFDT(lu)   = (1.-SSALB(lyu))*4.*M_PI*(UAVG(lu)-plsorc);

    if (ds->flag.prnt[1]) {
      printf("%10.4f%7d%12.3e%12.3e%12.3e%12.3e%12.3e%12.3e%12.3e%14.3e\n",                      UTAU(lu),lyu,RFLDIR(lu),RFLDN(lu),fdntot,FLUP(lu),fnet,UAVG(lu),plsorc,DFDT(lu));
    }
  }

  if (prntu0) {
    printf("\n\n%s\n"," ******** AZIMUTHALLY AVERAGED INTENSITIES ( at polar quadrature angles) *******");
    for (lu = 1; lu <= ds->ntau; lu++) {
      printf("\n%s%10.4f\n\n%s\n",                     " Optical depth =",UTAU(lu),
                     "     Angle (deg)   cos(Angle)     Intensity     Angle (deg)   cos(Angle)     Intensity");
      for (iq = 1; iq <= nn; iq++) {
        ang1 = acos(CMU(2*nn-iq+1))/DEG;
        ang2 = acos(CMU(     iq  ))/DEG;
        printf("%16.4f%13.5f%14.3e%16.4f%13.5f%14.3e\n",                        ang1,CMU(2*nn-iq+1),U0C(iq,   lu),
                        ang2,CMU(     iq  ),U0C(iq+nn,lu));
      }
    }
  }

  return;
}

/*============================= end of c_fluxes() =======================*/
