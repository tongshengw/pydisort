#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_sgeco() ================================*/

/*
   Factors a real matrix by Gaussian elimination
   and estimates the condition of the matrix.
   Revision date:  8/1/82
   Author:  Moler, C. B. (Univ. of New Mexico)
   If rcond is not needed, sgefa is slightly faster.
   To solve  A*X = B, follow sgeco by sgesl.

     Inputs:
        a       double(lda, n), the matrix to be factored.
        lda     int, the leading dimension of the array a.
        n       int, the order of the matrix a.

     Outputs:
        a       an upper triangular matrix and the multipliers
                which were used to obtain it.
                The factorization can be written  A = L*U , where
                L  is a product of permutation and unit lower
                triangular matrices and U is upper triangular.
        ipvt    int(n), an integer vector of pivot indices.
        rcond   double, an estimate of the reciprocal condition of a.
                For the system A*X = B, relative perturbations
                in A and B of size epsilon may cause relative
                perturbations in X of size epsilon/rcond.
                If rcond is so small that the logical expression
                  1.+rcond == 1.
                is true, then A may be singular to working precision.
                In particular, rcond is zero if exact singularity
                is detected or the estimate underflows.
        z       double(n), a work vector whose contents are usually
                unimportant. If A is close to a singular matrix, then z
                is an approximate null vector in the sense that
                norm(A*Z) = rcond*norm(A)*norm(Z) .
 ------------------------------------------------------------------*/

void c_sgeco(double *a,
             int     lda,
             int     n,
             int    *ipvt,
             double *rcond,
             double *z)
{
  int
    info;
  register int
    j,k,kb,kp1,l;
  double
    anorm,ek,s,sm,t,wk,wkm,ynorm;

  /*
   * compute 1-norm of A
   */
  anorm = 0.;
  for (j = 1; j <= n; j++) {
    anorm = MAX(anorm,c_sasum(n,&A(1,j)));
  }

  /*
   * factor
   */
  c_sgefa(a,lda,n,ipvt,&info);

  /*
   * rcond = 1/(norm(A)*(estimate of norm(inverse(A)))).
   * estimate = norm(Z)/norm(Y) where A*Z = Y and trans(A)*Y = E.
   * trans(A) is the transpose of A. The components of E are
   * chosen to cause maximum local growth in the elements of W where
   * trans(U)*W = E.  The vectors are frequently rescaled to avoid overflow.
   * solve trans(U)*W = E
   */
  ek = 1.;
  memset(z,0,n*sizeof(double));

  for (k = 1; k <= n; k++) {
    if (Z(k) != 0.) {
      ek = F77_SIGN(ek,-Z(k));
    }
    if (fabs(ek-Z(k)) > fabs(A(k,k))) {
      s = fabs(A(k,k))/fabs(ek-Z(k));
      c_sscal(n,s,z);
      ek *= s;
    }
    wk  =  ek-Z(k);
    wkm = -ek-Z(k);
    s   = fabs(wk);
    sm  = fabs(wkm);
    if (A(k,k) != 0.) {
      wk  /= A(k,k);
      wkm /= A(k,k);
    }
    else {
      wk  = 1.;
      wkm = 1.;
    }
    kp1 = k+1;
    if (kp1 <= n) {
      for (j = kp1; j <= n; j++) {
        sm   += fabs(Z(j)+wkm*A(k,j));
        Z(j) += wk*A(k,j);
        s    += fabs(Z(j));
      }
      if (s < sm) {
        t  = wkm-wk;
        wk = wkm;
        for (j = kp1; j <= n; j++) {
          Z(j) += t*A(k,j);
        }
      }
    }
    Z(k) = wk;
  }

  s = 1./c_sasum(n,z);
  c_sscal(n,s,z);
  /*
   * solve trans(L)*Y = W
   */
  for (kb = 1; kb <= n; kb++) {
    k = n+1-kb;
    if (k < n) {
      Z(k) += c_sdot(n-k,&A(k+1,k),&Z(k+1));
    }
    if (fabs(Z(k)) > 1.) {
      s = 1./fabs(Z(k));
      c_sscal(n,s,z);
    }
    l    = IPVT(k);
    t    = Z(l);
    Z(l) = Z(k);
    Z(k) = t;
  }
  s = 1./c_sasum(n,z);
  c_sscal(n,s,z);
  /*
   * solve L*V = Y
   */
  ynorm = 1.;
  for (k = 1; k <= n; k++) {
    l    = IPVT(k);
    t    = Z(l);
    Z(l) = Z(k);
    Z(k) = t;
    if (k < n) {
      c_saxpy(n-k,t,&A(k+1,k),&Z(k+1));
    }
    if (fabs(Z(k)) > 1.) {
      s = 1./fabs(Z(k));
      c_sscal(n,s,z);
      ynorm *= s;
    }
  }
  s = 1./c_sasum(n,z);
  c_sscal(n,s,z);
  /*
   * solve U*Z = V
   */
  ynorm *= s;
  for (kb = 1; kb <= n; kb++) {
    k = n+1-kb;
    if (fabs(Z(k)) > fabs(A(k,k))) {
      s = fabs(A(k,k))/fabs(Z(k));
      c_sscal(n,s,z);
      ynorm *= s;
    }
    if (A(k,k) != 0.) {
      Z(k) /= A(k,k);
    }
    else {
      Z(k) = 1.;
    }
    t = -Z(k);
    c_saxpy(k-1,t,&A(1,k),&Z(1));
  }
  /*
   * make znorm = 1.0
   */
  s = 1./c_sasum(n,z);
  c_sscal(n,s,z);
  ynorm *= s;
  if (anorm != 0.) {
    *rcond = ynorm/anorm;
  }
  else {
    *rcond = 0.;
  }

  return;
}

/*============================= end of c_sgeco() =========================*/
