#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_solve1() ===============================*/

/*
     Construct right-hand side vector -b- for isotropic incidence
     (only) on either top or bottom boundary and solve system
     of equations obtained from the boundary conditions and the
     continuity-of-intensity-at-layer-interface equations

     I N P U T      V A R I A B L E S:

       ds       :  Disort state variables
       cband    :  Left-hand side matrix of banded linear system
                   eq. SC(5), scaled by eq. SC(12); assumed already
                   in LU-decomposed form, ready for LINPACK solver
       ihom     :  Direction-of-illumination flag (TOP_ILLUM, top; BOT_ILLUM, bottom)
       ipvt     :
       ncol     :  Number of columns in CBAND
       ncut     :
       nn       :  Order of double-Gauss quadrature (NSTR/2)

    O U T P U T     V A R I A B L E S:

       b        :  Right-hand side vector of eq. SC(5) going into
                   sgbsl; returns as solution vector of eq.
                   SC(12), constants of integration without
                   exponential term
       ll       :  permanent storage for -b-, but re-ordered


    I N T E R N A L    V A R I A B L E S:

       ipvt     :  INTEGER vector of pivot indices
       ncd      :  Number of diagonals below or above main diagonal

   Called by- c_albtrans
   Calls- c_sgbsl
 +-------------------------------------------------------------------+
*/

void c_solve1(disort_state *ds,
              double       *cband,
              int           ihom,
              int          *ipvt,
              int           ncol,
              int           ncut,
              int           nn,
              double       *b,
              double       *ll)
{
  register int
    i,ipnt,iq,lc,ncd;

  memset(b,0,ds->nstr*ds->nlyr*sizeof(double));

  if (ihom == TOP_ILLUM) {
    /*
     * Because there are no beam or emission sources, remainder of B array is zero
     */
    for (i = 1; i <= nn; i++) {
      B(i)         = ds->bc.fisot;
      B(ncol-nn+i) = 0.;
    }
  }
  else if (ihom == BOT_ILLUM) {
    for (i = 1; i <= nn; i++) {
      B(i)         = 0.;
      B(ncol-nn+i) = ds->bc.fisot;
    }
  }
  else {
    c_errmsg("solve1---unrecognized ihom",DS_ERROR);
  }

  ncd = 3*nn-1;
  c_sgbsl(cband,(9*(ds->nstr/2)-2),ncol,ncd,ncd,ipvt,b,0);
  for (lc = 1; lc <= ncut; lc++) {
    ipnt = lc*ds->nstr-nn;
    for (iq = 1; iq <= nn; iq++) {
      LL(nn-iq+1,lc) = B(ipnt-iq+1);
      LL(nn+iq,  lc) = B(ipnt+iq  );
    }
  }

  return;
}

/*============================= end of c_solve1() ========================*/
