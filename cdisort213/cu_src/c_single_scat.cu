#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_single_scat() =========================*/

/*
        Calculates single-scattered intensity from eqs. STWL (65b,d,e)

                I N P U T   V A R I A B L E S

        dither   small multiple of machine precision
        layru    index of utau in multi-layered system
        nlyr     number of sublayers
        phase    phase functions of sublayers
        omega    single scattering albedos of sublayers
        tau      optical thicknesses of sublayers
        umu      cosine of emergent angle
        umu0     cosine of incident zenith angle
        utau     user defined optical depth for output intensity
        fbeam   incident beam radiation at top


   Called by- c_intensity_correction
 -------------------------------------------------------------------*/

double c_single_scat(double   dither,
                     int      layru,
                     int      nlyr,
                     double  *phase,
                     double  *omega,
                     double  *tau,
                     double   umu,
                     double   umu0,
                     double   utau,
                     double   fbeam)
{
  register int
    lyr;
  double
    ans,exp0,exp1;

  ans  = 0.;
  exp0 = exp(-utau/umu0);

  if (fabs(umu+umu0) <= dither) {
    /*
     * Calculate downward intensity when umu=umu0, eq. STWL (65e)
     */
    for (lyr = 1; lyr <= layru-1; lyr++) {
      ans += OMEGA(lyr)*PHASE(lyr)*(TAU(lyr)-TAU(lyr-1));
    }
    ans = fbeam/(4.*M_PI*umu0)*exp0*(ans+OMEGA(layru)*PHASE(layru)*(utau-TAU(layru-1)));
    return ans;
  }

  if (umu > 0.) {
    /*
     * Upward intensity, eq. STWL (65b)
     */
    for (lyr = layru; lyr <= nlyr; lyr++) {
      exp1  = exp(-((TAU(lyr)-utau)/umu+TAU(lyr)/umu0));
      ans  += OMEGA(lyr)*PHASE(lyr)*(exp0-exp1);
      exp0  = exp1;
    }
  }
  else {
    /*
     * Downward intensity, eq. STWL (65d)
     */
    for (lyr = layru; lyr >= 1; lyr--) {
      exp1  = exp(-((TAU(lyr-1)-utau)/umu+TAU(lyr-1)/umu0));
      ans  += OMEGA(lyr)*PHASE(lyr)*(exp0-exp1);
      exp0  = exp1;
    }
  }
  ans *= fbeam/(4.*M_PI*(1.+umu/umu0));

  return ans;
}

/*============================= end of c_single_scat() ==================*/
