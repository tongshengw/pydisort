#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_albtrans() =============================*/

/*
   DISORT special case to get only albedo and transmissivity of entire medium as a function of incident beam angle
   (many simplifications because boundary condition is just isotropic illumination, there are no thermal sources, and
   particular solutions do not need to be computed).  See Ref. S2 and references therein for details.
   The basic idea is as follows.  The reciprocity principle leads to the following relationships for a plane-parallel,
   vertically inhomogeneous medium lacking thermal (or other internal) sources:

      albedo(theta) = u_0(theta) for unit-intensity isotropic
                       illumination at *top* boundary
       trans(theta) =  u_0(theta) for unit-intensity isotropic
                       illumination at *bottom* boundary
    where

       albedo(theta) = albedo for beam incidence at angle theta
       trans(theta)  = transmissivity for beam incidence at angle theta
       u_0(theta)    = upward azim-avg intensity at top boundary
                       at angle theta

   O U T P U T    V A R I A B L E S:

       ALBMED(iu)   Albedo of the medium as a function of incident
                    beam angle cosine UMU(IU)

       TRNMED(iu)   Transmissivity of the medium as a function of
                    incident beam angle cosine UMU(IU)

    I N T E R N A L   V A R I A B L E S:

       ncd         number of diagonals below/above main diagonal
       rcond       estimate of the reciprocal condition of matrix CBAND; for system  CBAND*X = B, relative
                   perturbations in CBAND and B of size epsilon may cause relative perturbations in X of size
                   epsilon/RCOND.  If RCOND is so small that
                          1.0 + RCOND .eq. 1.0
                   is true, then CBAND may be singular to working precision.
       cband       Left-hand side matrix of linear system eq. SC(5), scaled by eq. SC(12);
                   in banded form required by LINPACK solution routines
       ncol        number of columns in CBAND matrix
       ipvt        INTEGER vector of pivot indices (most others documented in DISORT)

   Called by- c_disort
   Calls- c_legendre_poly, c_sgbco, c_solve_eigen, c_interp_eigenvec, c_set_matrix, c_solve1,
          c_albtrans_intensity, c_albtrans_spherical, c_print_albtrans
 --------------------------------------------------------------------------------------*/

void c_albtrans(disort_state  *ds,
                disort_output *out,
                disort_pair   *ab,
                double        *array,
                double        *b,
                double        *bdr,
                double        *cband,
                double        *cc,
                double        *cmu,
                double        *cwt,
                double        *dtaucpr,
                double        *eval,
                double        *evecc,
                double        *gl,
                double        *gc,
                double        *gu,
                int           *ipvt,
                double        *kk,
                double        *ll,
                int            nn,
                double        *taucpr,
                double        *ylmc,
                double        *ylmu,
                double        *z,
                double        *wk)
{
  int
    lyrcut,ncol;
  register int
    iq,iu,l,lc,mazim,ncd,ncut;
  double
    delm0,rcond,sgn,sphalb,sphtrn;

  mazim = 0;
  delm0 = 1.;
  /*
   * Set DISORT variables that are ignored in this special case but are needed below in argument
   * lists of subroutines shared with general case
   */
  ncut            = ds->nlyr;
  lyrcut          = FALSE;
  ds->bc.fisot    = 1.;
  ds->bc.fluor    = 0.;
  ds->flag.lamber = TRUE;

  /*
   * Get Legendre polynomials for computational and user polar angle cosines
   */
  c_legendre_poly(ds->numu,mazim,ds->nstr,ds->nstr-1,ds->umu,ylmu);
  c_legendre_poly(nn,      mazim,ds->nstr,ds->nstr-1,cmu,    ylmc);

  /*
   * Evaluate Legendre polynomials with negative arguments from those with positive arguments;
   * Dave/Armstrong eq. (15), STWL(59)
   */
  sgn = -1.0;
  for (l = mazim; l <= ds->nstr-1; l++) {
    sgn *= -1;
    for (iq = nn+1; iq <= ds->nstr; iq++) {
      YLMC(l,iq) = sgn*YLMC(l,iq-nn);
    }
  }

  /*
   * Zero out bottom reflectivity (ALBEDO is used only in analytic formulae involving ALBEDO = 0
   * solutions; eqs 16-17 of Ref S2)
   */
  memset(bdr,0,(ds->nstr/2)*((ds->nstr/2)+1)*sizeof(double));

  /*-------------------  BEGIN LOOP ON COMPUTATIONAL LAYERS  -------------*/
  for (lc = 1; lc <= ds->nlyr; lc++) {
    /*
     * Solve eigenfunction problem in eq. STWJ(8b), STWL(23f)
     */
    c_solve_eigen(ds,lc,ab,array,cmu,cwt,gl,mazim,nn,ylmc,cc,evecc,eval,kk,gc,wk);
    /*
     * Interpolate eigenvectors to user angles
     */
    c_interp_eigenvec(ds,lc,cwt,evecc,gl,gu,mazim,nn,wk,ylmc,ylmu);
  }
  /*------------------  END LOOP ON COMPUTATIONAL LAYERS  ---------------*/

  /*
   * Set coefficient matrix (CBAND) of equations
   * combining boundary and layer interface
   * conditions (in band-storage mode required by
   * LINPACK routines)
   */
  c_set_matrix(ds,bdr,cband,cmu,cwt,delm0,dtaucpr,gc,kk,lyrcut,&ncol,ncut,taucpr,wk);

  /*
   * LU-decompose the coeff. matrix (LINPACK)
   */
  ncd = 3*nn-1;
  c_sgbco(cband,(9*(ds->nstr/2)-2),ncol,ncd,ncd,ipvt,&rcond,z);
  if (1.+rcond == 1.) {
    c_errmsg("albtrans--sgbco says matrix near singular",DS_WARNING);
  }

  /*
   * First, illuminate from top; if only one layer, this will give us everything
   * Solve for constants of integration in homogeneous solution
   */
  c_solve1(ds,cband,TOP_ILLUM,ipvt,ncol,ncut,nn,b,ll);

  /*
   * Compute azimuthally-averaged intensity at user angles; gives albedo if multi-layer (eq. 9 of Ref S2);
   * gives both albedo and transmissivity if single layer (eqs. 3-4 of Ref S2)
   */
  c_albtrans_intensity(ds,out,gu,kk,ll,nn,taucpr,wk);

  /*
   * Get beam-incidence albedos from reciprocity principle
   */

  for (iu = 1; iu <= ds->numu/2; iu++) {
    ALBMED(iu) = U0U(iu+ds->numu/2,1);
  }
  if (ds->nlyr == 1) {
    for (iu = 1; iu <= ds->numu/2; iu++) {
      /*
       * Get beam-incidence transmissivities from reciprocity principle (1 layer);
       * flip them end over end to correspond to positive UMU instead of negative
       */
      TRNMED(iu) = U0U(ds->numu/2+1-iu,2)+exp(-TAUCPR(ds->nlyr)/UMU(iu+ds->numu/2));
    }
  }
  else {
    /*
     * Second, illuminate from bottom (if multiple layers)
     */
    c_solve1(ds,cband,BOT_ILLUM,ipvt,ncol,ncut,nn,b,ll);
    c_albtrans_intensity(ds,out,gu,kk,ll,nn,taucpr,wk);
    /*
     * Get beam-incidence transmissivities from reciprocity principle
     */
    for (iu = 1; iu <= ds->numu/2; iu++) {
      TRNMED(iu) = U0U(iu+ds->numu/2,1)+exp(-TAUCPR(ds->nlyr)/UMU(iu+ds->numu/2));
    }
  }

  if (ds->bc.albedo > 0.) {
    /*
     * Get spherical albedo and transmissivity
     */
    if (ds->nlyr == 1) {
      c_albtrans_spherical(ds,cmu,cwt,gc,kk,ll,nn,taucpr,&sphalb,&sphtrn);
    }
    else {
      c_albtrans_spherical(ds,cmu,cwt,gc,kk,ll,nn,taucpr,&sphtrn,&sphalb);
    }
    /*
     * Ref. S2, eqs. 16-17 (these eqs. have a simple physical interpretation
     * like that of adding-doubling eqs.)
     */
    for (iu = 1; iu <= ds->numu; iu++) {

      ALBMED(iu) += ds->bc.albedo/(1.-ds->bc.albedo*sphalb)*sphtrn*TRNMED(iu);
      TRNMED(iu) += ds->bc.albedo/(1.-ds->bc.albedo*sphalb)*sphalb*TRNMED(iu);
    }
  }
  /*
   * Return UMU to all positive values, to agree with ordering in ALBMED, TRNMED
   */
  ds->numu /= 2;
  for (iu = 1; iu <= ds->numu; iu++) {
    UMU(iu) = UMU(iu+ds->numu);
  }
  if (ds->flag.prnt[3]) {
    c_print_albtrans(ds,out);
  }

  /* CE: I want to output the the spherical albedo and transmittance, and use the */
  /* variables ALBMED and TRNMED for this. They are not used so far otherwise in uvspec */
  /* If somebody needs these variables I will include new variables for sphtrn and sphalb*/
  ALBMED(1)=sphalb;
  TRNMED(1)=sphtrn;

  return;
}

/*============================= end of c_albtrans() ======================*/
