#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_isamax() ===============================*/

/*
 INPUT--  n        Number of elements in vector of interest
          sx       Array, length n, containing vector

 OUTPUT-- ans      First i, i = 1 to n, to maximize fabs(SX(i))

 NOTE: Fortran input incx removed because it is not used by
       disort or twostr
 ---------------------------------------------------------------------*/

int c_isamax(int     n,
             double *sx)
{
  register int
    ans=0,i;
  double
   smax,xmag;

  if (n <= 0) {
    ans = 0;
  }
  else if (n == 1) {
    ans = 1;
  }
  else {
    smax = 0.;
    for (i = 1; i <= n; i++) {
      xmag = fabs(SX(i));
      if (smax < xmag) {
        smax = xmag;
        ans  = i;
      }
    }
  }

  return ans;
}

/*============================= end of c_isamax() ========================*/
