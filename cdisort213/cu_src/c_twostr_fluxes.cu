#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
#undef  KK
#define KK(lyu) kk[lyu-1]
/*============================= c_twostr_fluxes() ========================*/

/*
 Calculates the radiative fluxes, mean intensity, and flux derivative
 with respect to optical depth from the azimuthally-averaged intensity

 I n p u t     v a r i a b l e s:

   ds         :  'Disort' state variables
   ts         :  twostr_xyz structure variables (xp_0, yb_0d, zb_a...; see cdisort.h)
   ch         :  Chapman factor
   cmu        :  Abscissa for gauss quadrature over angle cosine
   kk         :  Eigenvalues
   layru      :  Layer numbers of user levels -utau-
   ll         :  Constants of integration in eqs. KST(42-43), obtaine by solving eqs. KST(38-41)
   lyrcut     :  Logical flag for truncation of comput. layer
   ncut       :  Number of computational layer where absorption optical depth exceeds -abscut-
   oprim      :  Delta-m scaled single scattering albedo
   rr         :  Eigenvectors at polar quadrature angles
   flag.spher :  TRUE turns on pseudo-spherical effects
   taucpr     :  Cumulative optical depth (delta-m-scaled)
   utaupr     :  Optical depths of user output levels in delta-m coordinates; equal to  -utau- if no delta-m

 O u t p u t     v a r i a b l e s:

   out      :  'Disort' output variables
   u0c      :  Azimuthally averaged intensities at polar quadrature angle cmu

 I n t e r n a l       v a r i a b l e s:

   dirint   :  direct intensity attenuated
   fdntot   :  total downward flux (direct + diffuse)
   fldir    :  fl[].zero, direct-beam flux (delta-m scaled)
   fldn     :  fl[].one, diffuse down-flux (delta-m scaled)
   fnet     :  net flux (total-down - diffuse-up)
   fact     :  EXP( - utaupr / ch ), where ch is the Chapman factor
   plsorc   :  Planck source function (thermal)
 ---------------------------------------------------------------------*/

void c_twostr_fluxes(disort_state  *ds,
                     twostr_xyz    *ts,
                     double        *ch,
                     double         cmu,
                     double        *kk,
                     int           *layru,
                     double        *ll,
                     int            lyrcut,
                     int            ncut,
                     double        *oprim,
                     double        *rr,
                     double        *taucpr,
                     double        *utaupr,
                     disort_output *out,
                     double        *u0c,
                     disort_pair   *fl)
{
  register int
    lu,lyu;
  double
    fdntot,fnet,plsorc,dirint;
  register double
    fact1,fact2;

  if (ds->flag.prnt[1]) {
    printf("\n\n                     <----------------------- Fluxes ----------------------->\n"                   "   optical  compu    downward    downward    downward       upward                    mean      Planck   d(net flux)\n"
                   "     depth  layer      direct     diffuse       total      diffuse         net   intensity      source   / d(op dep)\n");
  }

  memset(out->rad,0,ds->ntau*sizeof(disort_radiant));

  /*
   * Loop over user levels
   */
  if (ds->flag.planck) {
    for (lu = 1; lu <= ds->ntau; lu++) {
      lyu        = LAYRU(lu);
      fact1      = exp(-ZP_A(lyu)*UTAUPR(lu));
      U0C(1,lu) += fact1*(YP_0D(lyu)+YP_1D(lyu)*UTAUPR(lu));
      U0C(2,lu) += fact1*(YP_0U(lyu)+YP_1U(lyu)*UTAUPR(lu));
    }
  }
  for (lu = 1; lu <= ds->ntau; lu++) {
    lyu = LAYRU(lu);
    if (lyrcut && lyu > ncut) {
      /*
       * No radiation reaches this level
       */
      fdntot = 0.;
      fnet   = 0.;
      plsorc = 0.;
    }
    else {
      if (ds->bc.fbeam > 0.) {
        fact1      = exp(-ZB_A(lyu)*UTAUPR(lu));
        U0C(1,lu) += fact1*(YB_0D(lyu)+YB_1D(lyu)*UTAUPR(lu));
        U0C(2,lu) += fact1*(YB_0U(lyu)+YB_1U(lyu)*UTAUPR(lu));
        if (ds->bc.umu0 > 0. || ds->flag.spher) {
          fact1      = ds->bc.fbeam*exp(-UTAUPR(lu)/CH(lyu));
          dirint     = fact1;
          FLDIR(lu)  = fabs(ds->bc.umu0)*fact1;
          RFLDIR(lu) = fabs(ds->bc.umu0)*ds->bc.fbeam*exp(-UTAU(lu)/CH(lyu));
        }
        else {
          dirint     = 0.;
          FLDIR(lu)  = 0.;
          RFLDIR(lu) = 0.;
        }
      }
      else {
        dirint     = 0.;
        FLDIR(lu)  = 0.;
        RFLDIR(lu) = 0.;
      }
      fact1      = LL(1,lyu)*exp( KK(lyu)*(UTAUPR(lu)-TAUCPR(lyu  )));
      fact2      = LL(2,lyu)*exp(-KK(lyu)*(UTAUPR(lu)-TAUCPR(lyu-1)));
      U0C(1,lu) += fact2+RR(lyu)*fact1;
      U0C(2,lu) += fact1+RR(lyu)*fact2;
      /*
       * Calculate fluxes and mean intensities; downward and upward fluxes from eq. KST(9)
       */
      fact1     = 2.*M_PI*cmu;
      FLDN(lu)  = fact1*U0C(1,lu);
      FLUP(lu)  = fact1*U0C(2,lu);
      fdntot    = FLDN(lu)+FLDIR(lu);
      fnet      = fdntot-FLUP(lu);
      RFLDN(lu) = fdntot-RFLDIR(lu);
      /*
       * Mean intensity from eq. KST(10)
       */
      UAVG(lu) = U0C(1,lu)+U0C(2,lu);
      UAVG(lu) = (2.*M_PI*UAVG(lu)+dirint)/(4.*M_PI);

      /*
       * Flux divergence from eqs. KST(11-12)
       */
      plsorc   = 1./(1.-OPRIM(lyu))*exp(-ZP_A(lyu)*UTAUPR(lu))*(XP_0(lyu)+XP_1(lyu)*UTAUPR(lu));
      DFDT(lu) = (1.-SSALB(lyu))*4.*M_PI*(UAVG(lu)-plsorc);
    }
    if (ds->flag.prnt[1]) {
      printf("%10.4f%7d%12.3e%12.3e%12.3e%12.3e%12.3e%12.3e%12.3e%14.3e\n",                     UTAU(lu),lyu,RFLDIR(lu),RFLDN(lu),fdntot,FLUP(lu),fnet,UAVG(lu),plsorc,DFDT(lu));
    }
  }

  return;
}

/*============================= end of c_twostr_fluxes() =================*/
