#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_solve_eigen() =========================*/

/*
   Solves eigenvalue/vector problem necessary to construct homogeneous
   part of discrete ordinate solution; STWJ(8b), STWL(23f)
   ** NOTE ** Eigenvalue problem is degenerate when single scattering
              albedo = 1;  present way of doing it seems numerically more
              stable than alternative methods that we tried

   I N P U T     V A R I A B L E S:

       ds     :  Disort state variables
       lc     :
       gl     :  Delta-M scaled Legendre coefficients of phase function
                 (including factors 2l+1 and single-scatter albedo)
       cmu    :  Computational polar angle cosines
       cwt    :  Weights for quadrature over polar angle cosine
       mazim  :  Order of azimuthal component
       nn     :  Half the total number of streams
       ylmc   :  Normalized associated Legendre polynomial
                 at the quadrature angles CMU


   O U T P U T    V A R I A B L E S:

       cc     :  C-sub-ij in eq. SS(5); needed in SS(15&18)
       eval   :  NN eigenvalues of eq. SS(12), STWL(23f) on return
                 from asymmetric_matrix but then square roots taken
       evecc  :  NN eigenvectors  (G+) - (G-)  on return
                 from asymmetric_matrix ( column j corresponds to EVAL(j) )
                 but then  (G+) + (G-)  is calculated from SS(10),
                 G+  and  G-  are separated, and  G+  is stacked on
                 top of  G-  to form NSTR eigenvectors of SS(7)
       gc     :  Permanent storage for all NSTR eigenvectors, but
                 in an order corresponding to KK
       kk     :  Permanent storage for all NSTR eigenvalues of SS(7),
                 but re-ordered with negative values first ( square
                 roots of EVAL taken and negatives added )


   I N T E R N A L   V A R I A B L E S:

       ab            :  Matrices AMB (alpha-beta), APB (alpha+beta) in reduced eigenvalue problem (see cdisort.h)
       array         :  Complete coefficient matrix of reduced eigenvalue
                        problem: (alpha+beta)*(alpha-beta)
       gpplgm        :  (g+) + (g-) (cf. eqs. SS(10-11))
       gpmigm        :  (g+) - (g-) (cf. eqs. SS(10-11))
       wk            :  Scratch array required by asymmetric_matrix

   Called by- c_disort, c_albtrans
   Calls- c_asymmetric_matrix, c_errmsg
 -------------------------------------------------------------------*/

/*
 * NOTE: Here the scratch array ARRAY(,) is half the size in each dimension compared to other subroutines
 */
#undef  ARRAY
#define ARRAY(iq,jq) array[iq-1+(jq-1)*(ds->nstr/2)]

void c_solve_eigen(disort_state *ds,
                   int           lc,
                   disort_pair  *ab,
                   double       *array,
                   double       *cmu,
                   double       *cwt,
                   double       *gl,
                   int           mazim,
                   int           nn,
                   double       *ylmc,
                   double       *cc,
                   double       *evecc,
                   double       *eval,
                   double       *kk,
                   double       *gc,
                   double       *wk)
{
  int
    ier;
  register int
    iq,jq,kq,l;
  double
    alpha,beta,gpmigm,gpplgm,sum;

  /*
   * Calculate quantities in eqs. SS(5-6), STWL(8b,15,23f)
   */
  for (iq = 1; iq <= nn; iq++) {
    for (jq = 1; jq <= ds->nstr; jq++) {
      sum = 0.;
      for (l = mazim; l <= ds->nstr-1; l++) {
        sum += GL(l,lc)*YLMC(l,iq)*YLMC(l,jq);
      }
      CC(iq,jq) = .5*sum*CWT(jq);
    }
    for (jq = 1; jq <= nn; jq++) {
      /*
       * Fill remainder of array using symmetry relations  C(-mui,muj) = C(mui,-muj) and C(-mui,-muj) = C(mui,muj)
       */
      CC(iq+nn,jq   ) = CC(iq,jq+nn);
      CC(iq+nn,jq+nn) = CC(iq,jq   );
      /*
       * Get factors of coeff. matrix of reduced eigenvalue problem
       */
      alpha      = CC(iq,jq   )/CMU(iq);
      beta       = CC(iq,jq+nn)/CMU(iq);
      AMB(iq,jq) = alpha-beta;
      APB(iq,jq) = alpha+beta;
    }
    AMB(iq,iq) -= 1./CMU(iq);
    APB(iq,iq) -= 1./CMU(iq);
  }
  /*
   * Finish calculation of coefficient matrix of reduced eigenvalue problem:
   * get matrix product (alpha+beta)*(alpha-beta); SS(12),STWL(23f)
   */
  for (iq = 1; iq <= nn; iq++) {
    for (jq = 1; jq <= nn; jq++) {
      sum = 0.;
      for (kq = 1; kq <= nn; kq++) {
        sum += APB(iq,kq)*AMB(kq,jq);
      }
      ARRAY(iq,jq) = sum;
    }
  }

  /*
   * Find (real) eigenvalues and eigenvectors
   */
  c_asymmetric_matrix(array,evecc,eval,nn,ds->nstr/2,ds->nstr,&ier,wk);

  if (ier > 0) {
    printf("\n\n asymmetric_matrix--eigenvalue no. %4d didn't converge.  Lower-numbered eigenvalues wrong.\n",ier);
    c_errmsg("asymmetric_matrix--convergence problems",DS_ERROR);
  }

  for (iq = 1; iq <= nn; iq++) {
    EVAL(iq)     = sqrt(fabs(EVAL(iq)));
    KK(iq+nn,lc) = EVAL(iq);
    /*
     * Add negative eigenvalue
     */
    KK(nn+1-iq,lc) = -EVAL(iq);
  }

  /*
   * Find eigenvectors (G+) + (G-) from SS(10) and store temporarily in APB array
   */
  for (jq = 1; jq <= nn; jq++) {
    for (iq = 1; iq <= nn; iq++) {
      sum = 0.;
      for (kq = 1; kq <= nn; kq++) {
        sum += AMB(iq,kq)*EVECC(kq,jq);
      }
      APB(iq,jq) = sum/EVAL(jq);
    }
  }
  for (jq = 1; jq <= nn; jq++) {
    for (iq = 1; iq <= nn; iq++) {
      gpplgm = APB(  iq,jq);
      gpmigm = EVECC(iq,jq);
      /*
       * Recover eigenvectors G+,G- from their sum and difference; stack them to get eigenvectors of full system
       * SS(7) (JQ = eigenvector number)
       */
      EVECC(iq,   jq) = .5*(gpplgm+gpmigm);
      EVECC(iq+nn,jq) = .5*(gpplgm-gpmigm);
      /*
       * Eigenvectors corresponding to negative eigenvalues (corresp. to reversing sign of 'k' in SS(10) )
       */
      gpplgm *= -1;
      EVECC(iq,   jq+nn)     = .5*(gpplgm+gpmigm);
      EVECC(iq+nn,jq+nn)     = .5*(gpplgm-gpmigm);
      GC(nn+iq,  nn+jq,  lc) = EVECC(iq,   jq   );
      GC(nn-iq+1,nn+jq,  lc) = EVECC(iq+nn,jq   );
      GC(nn+iq,  nn-jq+1,lc) = EVECC(iq,   jq+nn);
      GC(nn-iq+1,nn-jq+1,lc) = EVECC(iq+nn,jq+nn);
    }
  }

  return;
}

/*============================= end of c_solve_eigen() ==================*/
