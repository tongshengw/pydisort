#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_new_intensity_correction() ============*/

/*
       Corrects intensity field by using alternative Buras-Emde algorithm
       (201X).

                I N P U T   V A R I A B L E S

       ds      Disort state variables
       dither  small multiple of machine precision
       flyr    separated fraction in delta-M method
       layru   index of UTAU in multi-layered system
       lyrcut  logical flag for truncation of computational layer
       ncut    total number of computational layers considered
       oprim   delta-M-scaled single-scatter albedo
       phirad  azimuthal angles in radians
       tauc    optical thickness at computational levels
       taucpr  delta-M-scaled optical thickness
       utaupr  delta-M-scaled version of UTAU

                O U T P U T   V A R I A B L E S

       out->UU  corrected intensity field; UU(IU,LU,J)
                 iu=1,ds->numu; lu=1,ds->ntau; j=1,ds->nphi

                I N T E R N A L   V A R I A B L E S

       ctheta    cosine of scattering angle
       dtheta    angle (degrees) to define aureole region as
                      direction of beam source +/- DTHETA
       phasa     actual (exact) phase function
       phasm     delta-M-scaled phase function
       phast     phase function used in TMS correction; actual phase
                      function divided by (1-FLYR*SSALB)
       pl        ordinary Legendre polynomial of degree l, P-sub-l
       plm1      ordinary Legendre polynomial of degree l-1, P-sub-(l-1)
       plm2      ordinary Legendre polynomial of degree l-2, P-sub-(l-2)
       theta0    incident zenith angle (degrees)
       thetap    emergent angle (degrees)
       ussndm    single-scattered intensity computed by using exact
                     phase function and scaled optical depth
                     (first term in STWL(68a))
       ussp      single-scattered intensity from delta-M method
                     (second term in STWL(68a))
       duims     intensity correction term from IMS method
                     (delta-I-sub-IMS in STWL(A.19))
       nf        number of angular phase integration grid point
                     (zenith angle, theta)
       np        number of angular phase integration grid point
                     (azimuth angle, phi)
       nphase    number of angles for which original phase function
                     (ds->phase) is defined
       mu_eq     cos(theta) phase integration grid points,
                     equidistant in abs(f_phas2)
       norm_phas normalization factor for phase integration
       norm      normalization factor for preparation of phas2
       neg_phas  index whether phas2 is negative
       phas2     residual phase function
       phasr     delta-M scaled phase function
       f_phas2   cumulative integrated phase function phas2
       fbar      mean value of separated fraction f

   Called by- c_disort
   Calls- c_single_scat, c__new_secondary_scat,
          prep_double_scat_integr, c_dbl_vector
 -------------------------------------------------------------------*/

void c_new_intensity_correction(disort_state  *ds,
				disort_output *out,
				double         dither,
				double        *flyr,
				int           *layru,
				int            lyrcut,
				int            ncut,
				double        *oprim,
				double        *phasa,
				double        *phast,
				double        *phasm,
				double        *phirad,
				double        *tauc,
				double        *taucpr,
				double        *utaupr)
{
  register int
    iu,jp,k,lc,ltau,lu;
  double
    ctheta,dtheta,duims,pl,plm1,plm2,
    theta0=0,thetap=0,ussndm,ussp;

  const int
    nf = 100;
  const double
    tiny = 1e-4;
  int it=0, lyr=0;
  int nphase=ds->nphase;

  double *mu_eq=NULL, *norm_phas=NULL, norm=0.0;
  int *neg_phas=NULL;

  double *phas2=NULL, *phasr=NULL;
  double f_phas2=0.0;
  double fbar=0.0;
  int need_secondary_scattering=0;

  dtheta = 10.;

  /* beginning of BDE stuff */

  /* check whether secondary scattering is performed at all */
  for (iu = 1; iu <= ds->numu; iu++) {
    if (UMU(iu) < 0.) {
      /*
       * Calculate zenith angles of incident and emerging directions
       */
      theta0 = acos(-ds->bc.umu0)/DEG;
      thetap = acos(UMU(iu))/DEG;
      if (fabs(theta0-thetap) <= dtheta) {
	need_secondary_scattering=TRUE;
	break;
      }
    }
  }

  if (need_secondary_scattering==TRUE) {
    /* Initialization of new PSPIKE.                                      */

    mu_eq  = c_dbl_vector(0,nf*ds->ntau-1,"mu_eq");
    norm_phas = c_dbl_vector(0,ds->ntau-1,"norm_phas");
    neg_phas  = c_int_vector(0,nf*ds->ntau-1,"neg_phas");
    phas2 = c_dbl_vector(0,ds->nphase*ds->ntau-1,"phas2");
    phasr = c_dbl_vector(0,ds->nlyr-1,"phasr");

    /* Calculate delta-scaled phase function (phasr) */

    for (it=1; it<=ds->nphase; it++) {

      ctheta = ds->MUP(it);

      for (lc=1; lc<=ds->nlyr; lc++)
	PHASR(lc) = 1.0 - FLYR(lc);

      plm1 = 1.0;
      plm2 = 0.0;

      for (k=1; k<=ds->nstr-1; k++) {

	/* ** Calculate Legendre polynomial of */
	/* ** P-sub-l by upward recurrence     */

	pl = ( (2*k-1) * ctheta * plm1 - (k-1) * plm2 ) / k;
	plm2 = plm1;
	plm1 = pl;

	for (lc=1; lc<=ds->nlyr; lc++)
	  PHASR(lc) += (2*k+1) * pl * ( PMOM(k,lc) - FLYR(lc) );

      }

      /* calculate difference between original and delta-scaled phase
	 functions (phas2) */

      for (lu=1; lu<=ds->ntau; lu++) {

	PHAS2(it,lu) = 0.0;

	/* this could be optimized */
	for (lyr=1; lyr<=LAYRU(lu)-1; lyr++)
	  PHAS2(it,lu) += ( DSPHASE(it,lyr) - PHASR(lyr) ) *
	    SSALB(lyr) * DTAUC(lyr);

	lyr = LAYRU(lu);
	PHAS2(it,lu) += ( DSPHASE(it,lyr) - PHASR(lyr) ) *
	  SSALB(lyr) * ( UTAU(lu) - TAUC(lyr-1) );

      }

    } /* end for it<nphas */

    /* normalize by 1/(ssa*beta*f) */

    for (lu=1; lu<=ds->ntau; lu++) {

      lyr = LAYRU(lu);
      fbar = FLYR(lyr) * SSALB(lyr) * ( UTAU(lu) - TAUC(lyr-1) );

      for (lyr=1; lyr<=LAYRU(lu)-1; lyr++)
	fbar += SSALB(lyr) * DTAUC(lyr) * FLYR(lyr);

      if ( fbar <= tiny || ds->bc.fbeam <= tiny )
	for (it=1; it<=ds->nphase; it++)
	  PHAS2(it,lu) = 0.0;
      else {
	fbar = 1. / fbar;
	for (it=1; it<=ds->nphase; it++)
	  PHAS2(it,lu) *= fbar;
      }

      /* normalize phas2 to 2.0 */

      f_phas2 = 0.0;
      for (it=2; it<=ds->nphase; it++)
	f_phas2 +=
	  ( ds->MUP(it) - ds->MUP(it-1) ) * 0.5 *
	  ( PHAS2(it,lu) + PHAS2(it-1,lu) );

      if (f_phas2 != 0.0) {
	norm = 2.0 / f_phas2;
	for (it=1; it<=ds->nphase; it++)
	  PHAS2(it,lu) *= norm;
      }

    } /* end for lu<ntau */

    prep_double_scat_integr (ds->nphase, ds->ntau, nf, ds->mu_phase,
			     phas2, mu_eq, neg_phas, norm_phas);
  } /* end if (need_secondary_scattering) */

  /* end of BDE stuff */

  /*
   * Start loop over zenith angles
   */
  for (iu = 1; iu <= ds->numu; iu++) {
    if (UMU(iu) < 0.) {
      /*
       * Calculate zenith angles of incident and emerging directions
       */
      theta0 = acos(-ds->bc.umu0)/DEG;
      thetap = acos(UMU(iu))/DEG;
    }
    /*
     * Start loop over azimuth angles
     */
    for (jp = 1; jp <= ds->nphi; jp++) {
      /*
       * Calculate cosine of scattering angle, eq. STWL(4)
       */
      ctheta = -ds->bc.umu0*UMU(iu)+sqrt((1.-SQR(ds->bc.umu0))*(1.-SQR(UMU(iu))))*cos(PHIRAD(jp));
      /*
       * Initialize phase function
       */
      for (lc = 1; lc <= ncut; lc++) {
        PHASM(lc) = 1.;
      }

      /* BDE ** Interpolate original phase function */
      /* BDE ** to actual phase function            */

      /* !!! +1: locate starts counting from 0! */
      it = locate_disort ( ds->mu_phase, ds->nphase, ctheta ) + 1;

      for (lc=1; lc<=ncut; lc++)
	PHASA(lc) = DSPHASE(it,lc)
	  + ( ctheta - ds->MUP(it) ) /
	  ( ds->MUP(it+1) - ds->MUP(it) ) *
	  ( DSPHASE(it+1,lc) - DSPHASE(it,lc) );
      /*
       * Initialize Legendre poly. recurrence
       */
      plm1 = 1.;
      plm2 = 0.;
      for (k = 1; k <= ds->nstr-1; k++) {
        /*
         * Calculate Legendre polynomial of P-sub-l by upward recurrence
         */
        pl   = ((double)(2*k-1)*ctheta*plm1-(double)(k-1)*plm2)/k;
        plm2 = plm1;
        plm1 = pl;

        /*
         * Calculate delta-M transformed phase function
         */
	for (lc=1; lc <= ncut; lc++) {
	  PHASM(lc) += (double)(2*k+1)*pl*(PMOM(k,lc)-FLYR(lc))/(1.-FLYR(lc));
	}
      }
      /*
       * Apply TMS method, eq. STWL(68)
       */
      for (lc = 1; lc <= ncut; lc++) {
        PHAST(lc) = PHASA(lc)/(1.-FLYR(lc)*SSALB(lc));
      }
      for (lu = 1; lu <= ds->ntau; lu++) {
        if (!lyrcut || LAYRU(lu) < ncut) {
          ussndm        = c_single_scat(dither,LAYRU(lu),ncut,phast,ds->ssalb,taucpr,UMU(iu),ds->bc.umu0,UTAUPR(lu),ds->bc.fbeam);
          ussp          = c_single_scat(dither,LAYRU(lu),ncut,phasm,oprim,    taucpr,UMU(iu),ds->bc.umu0,UTAUPR(lu),ds->bc.fbeam);
          UU(iu,lu,jp) += ussndm-ussp;
        }
      }
      if (UMU(iu) < 0. && fabs(theta0-thetap) <= dtheta) {
        /*
         * Emerging direction is in the aureole (theta0 +/- dtheta).
         * Apply IMS method for correction of secondary scattering below top level.
         */
        ltau = 1;
        if (UTAU(1) <= dither) {
          ltau = 2;
        }
        for (lu = ltau; lu <= ds->ntau; lu++) {
          if(!lyrcut || LAYRU(lu) < ncut) {
            duims = c_new_secondary_scat(ds,iu,lu,it,ctheta,flyr,
					 LAYRU(lu),tauc,
					 nf,
					 phas2, mu_eq, neg_phas,
					 NORM_PHAS(lu));
	    UU(iu,lu,jp) -= duims;
          }
        }
      }
    } /* end loop over azimuth angles */
  } /* end loop over zenith angles */

  free(mu_eq); free(norm_phas); free(neg_phas);
  free(phas2); free(phasr);

  return;
}

/*============================= end of c_new_intensity_correction() =====*/
