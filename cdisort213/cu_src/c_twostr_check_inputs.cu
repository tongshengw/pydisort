#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
#undef  KK
#define KK(lyu) kk[lyu-1]
/*============================= c_twostr_check_inputs() ==================*/

/*
 * Checks the twostr input dimensions and variables
 */

void c_twostr_check_inputs(disort_state *ds,
                           double       *gg,
                           int          *ierror,
                           double       *tauc)
{
  int
    inperr,lc,lu;
  double
    umumin;

  inperr = FALSE;

  if (ds->nlyr < 1) {
    inperr    = c_write_bad_var(ds->flag.quiet,"nlyr");
    IERROR(1) = 1;
  }

  for (lc = 1; lc <= ds->nlyr; lc++) {
    if (DTAUC(lc) < 0.) {
      inperr     = c_write_bad_var(ds->flag.quiet,"dtauc");
      IERROR(3) += 1;
    }
    if (SSALB(lc) < 0. || SSALB(lc) > 1.) {
      inperr     = c_write_bad_var(ds->flag.quiet,"ssalb");
      IERROR(4) += 1;
    }
    if (ds->flag.planck) {
      if (lc == 1 && TEMPER(0) < 0.) {
        inperr     = c_write_bad_var(ds->flag.quiet,"temper");
        IERROR(5) += 1;
      }
      if (TEMPER(lc) < 0.) {
        inperr     = c_write_bad_var(ds->flag.quiet,"temper");
        IERROR(5) += 1;
      }
    }
    if (GG(lc) < -1. || GG(lc) > 1.) {
      inperr     = c_write_bad_var(ds->flag.quiet,"gg");
      IERROR(6) += 1;
    }
  }

  if(ds->flag.spher==TRUE) {
    for (lc = 1; lc <= ds->nlyr; lc++) {
      if (ds->ZD(lc) > ds->ZD(lc-1)) {
        inperr     = c_write_bad_var(ds->flag.quiet,"zd");
        IERROR(7) += 1;
      }
    }
  }

  if (ds->flag.usrtau) {
    if (ds->ntau < 1) {
      inperr    = c_write_bad_var(ds->flag.quiet,"ntau");
      IERROR(8) = 1;
    }
    for (lu = 1; lu <= ds->ntau; lu++) {
      if (fabs(UTAU(lu)-TAUC(ds->nlyr)) <= 1.e-6*TAUC(ds->nlyr)) { /* relative check copied from c_check_inputs() */
        UTAU(lu)= TAUC(ds->nlyr);
      }
      if (UTAU(lu) < 0. || UTAU(lu) > TAUC(ds->nlyr)) {
        inperr      = c_write_bad_var(ds->flag.quiet,"utau");
        IERROR(10) += 1;
      }
    }
  }

  if (ds->bc.fbeam < 0.) {
    inperr     = c_write_bad_var(ds->flag.quiet,"fbeam");
    IERROR(12) = 1;
  }

  umumin = 0.;
  if(ds->flag.spher==TRUE) {
    umumin = -1.;
  }

  if (ds->bc.fbeam > 0. && (ds->bc.umu0 <= umumin || ds->bc.umu0 > 1.)) {
    inperr     = c_write_bad_var(ds->flag.quiet,"umu0");
    IERROR(13) = 1;
  }
  if (ds->bc.fisot < 0.) {
    inperr     = c_write_bad_var(ds->flag.quiet,"fisot");
    IERROR(14) = 1;
  }
  if (ds->bc.albedo < 0. || ds->bc.albedo > 1.) {
    inperr     = c_write_bad_var(ds->flag.quiet,"albedo");
    IERROR(15) = 1;
  }

  if(ds->flag.planck) {
    if (ds->wvnmlo < 0. || ds->wvnmhi < ds->wvnmlo) {
      inperr     = c_write_bad_var(ds->flag.quiet,"wvnmlo,hi");
      IERROR(16) = 1;
    }
    if (ds->bc.temis < 0. || ds->bc.temis > 1.) {
      inperr     = c_write_bad_var(ds->flag.quiet,"temis");
      IERROR(17) = 1;
    }
    if (ds->bc.btemp < 0.) {
      inperr     = c_write_bad_var(ds->flag.quiet,"btemp");
      IERROR(18) = 1;
    }
    if (ds->bc.ttemp < 0.) {
      inperr     = c_write_bad_var(ds->flag.quiet,"ttemp");
      IERROR(19) = 1;
    }
  }

  if (!ds->flag.usrtau && ds->ntau < ds->nlyr+1) {
    inperr = c_write_too_small_dim(ds->flag.quiet,"ds.ntau",ds->nlyr+1);
    IERROR(22) = 1;
  }

  if (ds->bc.fluor < 0.) {
    inperr     = c_write_bad_var(ds->flag.quiet,"fluor");
    IERROR(23) = 1;
  }

  if (inperr) {
    c_errmsg("twostr_check_inputs--input and/or dimension errors",DS_ERROR);
  }

  for (lc = 1; lc <= ds->nlyr; lc++) {
    if (ds->flag.planck && fabs(TEMPER(lc)-TEMPER(lc-1)) > 50. && ds->flag.quiet==VERBOSE) {
      c_errmsg("twostr_check_inputs--vertical temperature step may be too large for good accuracy",DS_WARNING);
    }
  }

  return;
}

/*============================= end of c_twostr_check_inputs() ===========*/
