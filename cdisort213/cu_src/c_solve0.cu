#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_solve0() ==============================*/

/*
        Construct right-hand side vector B for general boundary
        conditions STWJ(17) and solve system of equations obtained
        from the boundary conditions and the continuity-of-
        intensity-at-layer-interface equations.
        Thermal emission contributes only in azimuthal independence.

    I N P U T      V A R I A B L E S:

       ds       :  Disort input variables
       bdr      :  Surface bidirectional reflectivity
       bem      :  Surface bidirectional emissivity
       bplanck  :  Bottom boundary thermal emission
       cband    :  Left-hand side matrix of linear system eq. SC(5),
                   scaled by eq. SC(12); in banded form required
                   by LINPACK solution routines
       cmu,cwt  :  Abscissae, weights for Gauss quadrature
                   over angle cosine
       expbea   :  Transmission of incident beam, EXP(-TAUCPR/UMU0)
       lyrcut   :  Logical flag for truncation of computational layers
       mazim    :  Order of azimuthal component
       ncol     :  Number of columns in CBAND
       nn       :  Order of double-Gauss quadrature (NSTR/2)
       ncut     :  Total number of computational layers considered
       tplanck  :  Top boundary thermal emission
       taucpr   :  Cumulative optical depth (delta-M-scaled)
       zz       :  Beam source vectors in eq. SS(19), STWL(24b)
       zzg      :  Beam source vectors in eq. KS(10)for a general source constant over a layer
       plk      :  Thermal source vectors z0,z1 by solving eq. SS(16), Y0,Y1 in STWL(26b,a);
                   plk[].zero, plk[].one (see cdisort.h)

    O U T P U T     V A R I A B L E S:

       b        :  Right-hand side vector of eq. SC(5) going into
                   sgbsl; returns as solution vector of eq. SC(12),
                   constants of integration without exponential term
      ll        :  Permanent storage for B, but re-ordered

   I N T E R N A L    V A R I A B L E S:

       ipvt     :  Integer vector of pivot indices
       it       :  Pointer for position in  B
       ncd      :  Number of diagonals below or above main diagonal
       rcond    :  Indicator of singularity for cband
       z        :  Scratch array required by sgbco

   Called by- c_disort
   Calls- c_sgbco, c_errmsg, c_sgbsl
 +-------------------------------------------------------------------*/

void c_solve0(disort_state *ds,
              double       *b,
              double       *bdr,
              double       *bem,
              double        bplanck,
              double       *cband,
              double       *cmu,
              double       *cwt,
              double       *expbea,
              int          *ipvt,
              double       *ll,
              int           lyrcut,
              int           mazim,
              int           ncol,
              int           ncut,
              int           nn,
              double        tplanck,
              double       *taucpr,
              double       *z,
              disort_pair  *zbeamsp,
	      double       *zbeama,
              double       *zz,
              double       *zzg,
              disort_pair  *plk)
{
  register int
    ipnt,iq,it,jq,lc,ncd;
  double
    rcond,sum,diff;

  memset(b,0,ds->nstr*ds->nlyr*sizeof(double));

  /*
   * Construct B, STWJ(20a,c) for parallel beam+bottom
   * reflection+thermal emission at top and/or bottom
   */
  if (mazim > 0 && ( ds->bc.fbeam > 0.  || ds->flag.general_source) ) {
    /*
     * Azimuth-dependent case (never called if FBEAM = 0)
     */
    if ( lyrcut == TRUE || ds->flag.lamber == TRUE ) {
      /*
       * No azimuthal-dependent intensity for Lambert surface; no
       * intensity component for truncated bottom layer
       */
      for (iq = 1; iq <= nn; iq++) {
        /*
         * Top boundary
         */
	if ( ds->flag.spher == TRUE ) {
	  B(iq) = - ZBEAM0(nn+1-iq,1);
	}
	else {
	  B(iq) = - ZZ(nn+1-iq,1);
	}
	if ( ds->flag.general_source == TRUE ) {
	  B(iq) -= ZZG(nn+1-iq,1);
	  //aky	  B(iq) = B(iq) - ZZG(nn+1-iq,1);
	}
        /*
         * Bottom boundary
         */
	if ( ds->flag.spher == TRUE ) {
	  B(ncol-nn+iq) = - exp(-ZBEAMA(ncut)*TAUCPR(ncut))*
	    (ZBEAM0(iq+nn,ncut) + ZBEAM1(iq+nn,ncut)*TAUCPR(ncut));
	}
	else {
	  B(ncol-nn+iq) = - ZZ(iq+nn,ncut)*EXPBEA(ncut);
	}
	if ( ds->flag.general_source == TRUE ) {
	  B(ncol-nn+iq) -=  ZZG(iq+nn,ncut);
	  //aky	  B(ncol-nn+iq) = B(ncol-nn+iq)  - ZZG(iq+nn,ncut);
	}
      }
    }
    else {
      for (iq = 1; iq <= nn; iq++) {
	if ( ds->flag.spher == TRUE ) {
	  B(iq) = - ZBEAM0(nn+1-iq,1);
	}
	else {
	  B(iq) = - ZZ(nn+1-iq,1);
	}
	if ( ds->flag.general_source == TRUE ) {
	  B(iq) -= ZZG(nn+1-iq,1);
	  //aky	  B(iq) = B(iq) - ZZG(nn+1-iq,1);
	}
	if ( ds->flag.spher == TRUE ) {
	  c_errmsg("solve0--BDR not implemented for pseudo-spherical geometry",
		   DS_WARNING);
	}
	else {
	  sum   = 0.;
	  for (jq = 1; jq <= nn; jq++) {
	    sum += CWT(jq)*CMU(jq)*BDR(iq,jq)*ZZ(nn+1-jq,ncut)*EXPBEA(ncut);
	  }
	  B(ncol-nn+iq) = sum;
	  if ( ds->flag.general_source == TRUE ) {
	    sum   = 0.;
	    for (jq = 1; jq <= nn; jq++) {
	      sum += CWT(jq)*CMU(jq)*BDR(iq,jq)*ZZG(nn+1-jq,ncut);
	    }
	    B(ncol-nn+iq) += sum;
	  }
	}
        if (ds->bc.fbeam > 0.) {
	  if ( ds->flag.spher == TRUE ) {
	    c_errmsg("solve0--BDR not implemented for pseudo-spherical geometry",
		     DS_WARNING)  ;
	  }
	  else {
	    B(ncol-nn+iq) += (BDR(iq,0)*ds->bc.umu0*ds->bc.fbeam/
			      M_PI-ZZ(iq+nn,ncut))*EXPBEA(ncut);
	  }
        }
	if ( ds->flag.general_source == TRUE ) {
	    B(ncol-nn+iq) += -ZZG(iq+nn,ncut);
	}
      }
    }
    /*
     * Continuity condition for layer interfaces of eq. STWJ(20b)
     */
    it = nn;
    diff = 0;
    for (lc = 1; lc <= ncut-1; lc++) {
      for (iq = 1; iq <= ds->nstr; iq++) {
	if ( ds->flag.general_source == TRUE ) {
	  diff = (ZZG(iq,lc+1)-ZZG(iq,lc));
	}
	if ( ds->flag.spher == TRUE ) {
	  B(++it) = exp(-ZBEAMA(lc+1)*TAUCPR(lc))*
	    (ZBEAM0(iq,lc+1)+ZBEAM1(iq,lc+1)*TAUCPR(lc))
	    -  exp(-ZBEAMA(lc)*TAUCPR(lc))*
	    (ZBEAM0(iq,lc)+ZBEAM1(iq,lc)*TAUCPR(lc))
	    + diff;
	}
	else {
	  B(++it) = (ZZ(iq,lc+1)-ZZ(iq,lc))*EXPBEA(lc)  + diff;
	}
      }
    }
  }
  else {
    /*
     * Azimuth-independent case
     */
    if (ds->bc.fbeam == 0. && ds->flag.general_source == FALSE ) {
      for (iq = 1; iq <= nn; iq++) {
        /*
         * Top boundary
         */
        B(iq) = -ZPLK0(nn+1-iq,1)+ds->bc.fisot+tplanck;
      }
      if ( lyrcut == TRUE ) {
        /*
         * No intensity component for truncated bottom layer
         */
        for (iq = 1; iq <= nn; iq++) {
          /*
           * Bottom boundary
           */
          B(ncol-nn+iq) = -ZPLK0(iq+nn,ncut)-ZPLK1(iq+nn,ncut)*TAUCPR(ncut);
        }
      }
      else {
        for (iq = 1; iq <= nn; iq++) {
          sum = 0.;
          for (jq = 1; jq <= nn; jq++) {
            sum += CWT(jq)*CMU(jq)*BDR(iq,jq)*
	      (ZPLK0(nn+1-jq,ncut)+ZPLK1(nn+1-jq,ncut)*TAUCPR(ncut));
          }
          B(ncol-nn+iq) = 2.*sum+BEM(iq)*bplanck-
	    ZPLK0(iq+nn,ncut)-ZPLK1(iq+nn,ncut)*TAUCPR(ncut);
        }
      }
      /*
       * Continuity condition for layer interfaces, STWJ(20b)
       */
      it = nn;
      for (lc = 1; lc <= ncut-1; lc++) {
        for (iq = 1; iq <= ds->nstr; iq++) {
          B(++it) = ZPLK0(iq,lc+1)-ZPLK0(iq,lc)+
	    (ZPLK1(iq,lc+1)-ZPLK1(iq,lc))*TAUCPR(lc);
        }
      }
    }
    else {
      if ( ds->flag.spher == TRUE ) {
	for (iq = 1; iq <= nn; iq++)
	  B(iq) = -ZBEAM0(nn+1-iq,1)-ZPLK0(nn+1-iq,1)+ds->bc.fisot+tplanck;
      }
      else {
	for (iq = 1; iq <= nn; iq++)
	  B(iq) = -ZZ(nn+1-iq,1)-ZPLK0(nn+1-iq,1)+ds->bc.fisot+tplanck;
      }
      if ( ds->flag.general_source == TRUE ) {
	for (iq = 1; iq <= nn; iq++)
	  B(iq) -= ZZG(nn+1-iq,1);
	//aky	  B(iq) = B(iq) - ZZG(nn+1-iq,1);
      }
      if (lyrcut) {
	if ( ds->flag.spher == TRUE ) {
	  for (iq = 1; iq <= nn; iq++) {
	    B(ncol-nn+iq) = -exp(-ZBEAMA(ncut)*TAUCPR(ncut))*
	      (ZBEAM0(iq+nn,ncut)+ ZBEAM1(iq+nn,ncut)*TAUCPR(ncut))
	      -ZPLK0(iq+nn,ncut)-ZPLK1(iq+nn,ncut)*TAUCPR(ncut);
	  }
	}
	else {
	  for (iq = 1; iq <= nn; iq++) {
	    B(ncol-nn+iq) = -ZZ(iq+nn,ncut)*EXPBEA(ncut)
	      -ZPLK0(iq+nn,ncut)-ZPLK1(iq+nn,ncut)*TAUCPR(ncut);
	  }
	}
	if ( ds->flag.general_source == TRUE ) {
	  for (iq = 1; iq <= nn; iq++)
	    B(ncol-nn+iq) -= ZZG(iq+nn,ncut);
	  //aky	    B(ncol-nn+iq) = B(ncol-nn+iq) - ZZG(iq+nn,ncut);
	}
      }
      else {
	if ( ds->flag.spher == TRUE ) {
	  for (iq = 1; iq <= nn; iq++) {
	    sum = 0.;
	    for (jq = 1; jq <= nn; jq++) {
	      sum += CWT(jq)*CMU(jq)*BDR(iq,jq)*
		( exp(-ZBEAMA(ncut)*TAUCPR(ncut))*
		  (ZBEAM0(nn+1-jq,ncut)+ZBEAM1(nn+1-jq,ncut)*TAUCPR(ncut))
		  + ZZG(nn+1-jq,ncut)
		  + ZPLK0(nn+1-jq,ncut)+ZPLK1(nn+1-jq,ncut)*TAUCPR(ncut));
	    }
	    B(ncol-nn+iq) = 2.0*sum +
	      ( BDR(iq,0)*ds->bc.umu0*ds->bc.fbeam/M_PI) *EXPBEA(ncut)
	      -  exp(-ZBEAMA(ncut)*TAUCPR(ncut))*
	      (ZBEAM0(iq+nn,ncut)+ZBEAM1(iq+nn,ncut)*TAUCPR(ncut))
	      - ZZG(iq+nn,ncut)
	      + BEM(iq)*bplanck
	      -ZPLK0(iq+nn,ncut)-ZPLK1(iq+nn,ncut)*TAUCPR(ncut)
	      +ds->bc.fluor;
	  }
	}
	else {
	  for (iq = 1; iq <= nn; iq++) {
	    sum = 0.;
	    for (jq = 1; jq <= nn; jq++) {
	      sum += CWT(jq)*CMU(jq)*BDR(iq,jq)*
		(ZZ(nn+1-jq,ncut)*EXPBEA(ncut)+ZPLK0(nn+1-jq,ncut)
		 + ZZG(nn+1-jq,ncut)
		 +ZPLK1(nn+1-jq,ncut)*TAUCPR(ncut));
	    }
	    B(ncol-nn+iq) = 2.*sum+
	      (BDR(iq,0)*ds->bc.umu0*ds->bc.fbeam/M_PI-ZZ(iq+nn,ncut))
	      *EXPBEA(ncut)
	      - ZZG(iq+nn,ncut)
	      +BEM(iq)*bplanck-ZPLK0(iq+nn,ncut)-ZPLK1(iq+nn,ncut)*TAUCPR(ncut)
	      +ds->bc.fluor;
	  }
	}
      }
      it = nn;
      if ( ds->flag.spher == TRUE ) {
	for (lc = 1; lc <= ncut-1; lc++) {
	  for (iq = 1; iq <= ds->nstr; iq++) {
	    B(++it) = exp(-ZBEAMA(lc+1)*TAUCPR(lc))*
	      (ZBEAM0(iq,lc+1)+ZBEAM1(iq,lc+1)*TAUCPR(lc))
	      -exp(-ZBEAMA(lc)*TAUCPR(lc))*
	      (ZBEAM0(iq,lc)+ZBEAM1(iq,lc)*TAUCPR(lc))
	      +ZZG(iq,lc+1)-ZZG(iq,lc)
	      +ZPLK0(iq,lc+1)-ZPLK0(iq,lc)+
	      (ZPLK1(iq,lc+1)-ZPLK1(iq,lc))*TAUCPR(lc);
	  }
	}
      }
      else {
	for (lc = 1; lc <= ncut-1; lc++) {
	  for (iq = 1; iq <= ds->nstr; iq++) {
	    B(++it) = (ZZ(iq,lc+1)-ZZ(iq,lc))*EXPBEA(lc)
	      +ZZG(iq,lc+1)-ZZG(iq,lc)
	      +ZPLK0(iq,lc+1)-ZPLK0(iq,lc)
	      +(ZPLK1(iq,lc+1)-ZPLK1(iq,lc))*TAUCPR(lc);
	  }
	}
      }
    }
  }

  /*
   * Find L-U (lower/upper triangular) decomposition of band matrix
   * CBAND and test if it is nearly singular (note: CBAND is
   * destroyed) (CBAND is in LINPACK packed format)
   */
  rcond = 0.;
  ncd   = 3*nn-1;
  c_sgbco(cband,(9*(ds->nstr/2)-2),ncol,ncd,ncd,ipvt,&rcond,z);

  if (1.+rcond == 1.) {
    c_errmsg("solve0--sgbco says matrix near singular",DS_WARNING);
  }

  /*
   * Solve linear system with coeff matrix CBAND and R.H. side(s) B
   * after CBAND has been L-U decomposed. Solution is returned in B.
   */

  c_sgbsl(cband,(9*(ds->nstr/2)-2),ncol,ncd,ncd,ipvt,b,0);

  /*
   * Zero CBAND (it may contain 'foreign' elements upon returning from
   * LINPACK); necessary to prevent errors
   */
  memset(cband,0,(9*(ds->nstr/2)-2)*(ds->nstr*ds->nlyr)*sizeof(double));

  for (lc = 1; lc <= ncut; lc++) {
    ipnt = lc*ds->nstr-nn;
    for (iq = 1; iq <= nn; iq++) {
      LL(nn-iq+1,lc) = B(ipnt-iq+1);
      LL(nn+iq,  lc) = B(ipnt+iq  );
    }
  }

  return;
}
/*============================= end of c_solve0() =======================*/
