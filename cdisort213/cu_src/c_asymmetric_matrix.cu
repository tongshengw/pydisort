#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_asymmetric_matrix() ===================*/

/*
  Solves eigenfunction problem for real asymmetric matrix for which it
  is known a priori that the eigenvalues are real. This is an adaptation
  of a subroutine EIGRF in the IMSL library to use real instead of complex
  arithmetic, accounting for the known fact that the eigenvalues and
  eigenvectors in the discrete ordinate solution are real.

  EIGRF is based primarily on EISPACK routines.  The matrix is first
  balanced using the Parlett-Reinsch algorithm.  Then the Martin-Wilkinson
  algorithm is applied. There is a statement 'j = wk(i)' that converts a
  double precision variable to an integer variable; this seems dangerous
  to us in principle, but seems to work fine in practice.

  References:

  Dongarra, J. and C. Moler, EISPACK -- A Package for Solving Matrix
      Eigenvalue Problems, in Cowell, ed., 1984: Sources and Development of
      Mathematical Software, Prentice-Hall, Englewood Cliffs, NJ
  Parlett and Reinsch, 1969: Balancing a Matrix for Calculation of
      Eigenvalues and Eigenvectors, Num. Math. 13, 293-304
  Wilkinson, J., 1965: The Algebraic Eigenvalue Problem, Clarendon Press,
      Oxford

   I N P U T    V A R I A B L E S:

       aa    :  input asymmetric matrix, destroyed after solved
        m    :  order of aa
       ia    :  first dimension of aa
    ievec    :  first dimension of evec

   O U T P U T    V A R I A B L E S:

       evec  :  (unnormalized) eigenvectors of aa (column j corresponds to EVAL(J))
       eval  :  (unordered) eigenvalues of aa (dimension m)
       ier   :  if != 0, signals that EVAL(ier) failed to converge;
                   in that case eigenvalues ier+1,ier+2,...,m  are
                   correct but eigenvalues 1,...,ier are set to zero.

   S C R A T C H   V A R I A B L E S:

       wk    :  work area (dimension at least 2*m)

   Called by- c_solve_eigen
   Calls- c_errmsg
 -------------------------------------------------------------------*/

void c_asymmetric_matrix(double *aa,
                         double *evec,
                         double *eval,
                         int     m,
                         int     ia,
                         int     ievec,
                         int    *ier,
                         double *wk)
{
  const double
   c1 =    .4375,
   c2 =    .5,
   c3 =    .75,
   c4 =    .95,
   c5 =  16.,
   c6 = 256.;
  int
    noconv,notlas,
    i,ii,in,j,k,ka,kkk,l,lb=0,lll,n,n1,n2;
  double
    col,discri,f,g,h,p=0,q=0,r=0,repl,rnorm,row,
    s,scale,sgn,t,tol,uu,vv,w,x,y,z;

  *ier = 0;
  tol = DBL_EPSILON;
  if (m < 1 || ia < m || ievec < m) {
    c_errmsg("asymmetric_matrix--bad input variable(s)",DS_ERROR);
  }

  /*
   * Handle 1x1 and 2x2 special cases
   */
  if (m == 1) {
    EVAL(1)   = AA(1,1);
    EVEC(1,1) = 1.;
    return;
  }
  else if (m == 2) {
    discri = SQR(AA(1,1)-AA(2,2))+4.*AA(1,2)*AA(2,1);
    if(discri < 0.) {
      c_errmsg("asymmetric_matrix--complex evals in 2x2 case",DS_ERROR);
    }
    sgn = 1.;
    if (AA(1,1) < AA(2,2)) {
     sgn = -1.;
    }
    EVAL(1)   = .5*(AA(1,1)+AA(2,2)+sgn*sqrt(discri));
    EVAL(2)   = .5*(AA(1,1)+AA(2,2)-sgn*sqrt(discri));
    EVEC(1,1) = 1.;
    EVEC(2,2) = 1.;
    if (AA(1,1) == AA(2,2) && (AA(2,1) == 0. || AA(1,2) == 0.)) {
      rnorm     = fabs(AA(1,1))+fabs(AA(1,2))+fabs(AA(2,1))+fabs(AA(2,2));
      w         = tol*rnorm;
      EVEC(2,1) =  AA(2,1)/w;
      EVEC(1,2) = -AA(1,2)/w;
    }
    else {
      EVEC(2,1) = AA(2,1)/(EVAL(1)-AA(2,2));
      EVEC(1,2) = AA(1,2)/(EVAL(2)-AA(1,1));
    }
    return;
  }

  /*
   * Initialize output variables
   */
  *ier = 0;
  memset(eval,0,m*sizeof(double));
  memset(evec,0,ievec*ievec*sizeof(double));
  for (i = 1; i <= m; i++) {
    EVEC(i,i) = 1.;
  }

  /*
   * Balance the input matrix and reduce its norm by diagonal similarity transformation stored in wk;
   * then search for rows isolating an eigenvalue and push them down.
   */
  rnorm = 0.;
  l     = 1;
  k     = m;

S50:

  kkk = k;
  for (j = kkk; j >= 1; j--) {
    row = 0.;
    for (i = 1; i <= k; i++) {
      if (i != j) {
        row += fabs(AA(j,i));
      }
    }
    if (row == 0.) {
      WK(k) = (double)j;
      if (j != k) {
        for (i = 1; i <= k; i++) {
          repl    = AA(i,j);
          AA(i,j) = AA(i,k);
          AA(i,k) = repl;
        }
        for (i = l; i <= m; i++) {
          repl    = AA(j,i);
          AA(j,i) = AA(k,i);
          AA(k,i) = repl;
        }
      }
      k--;
      goto S50;
    }
  }

  /*
   * Search for columns isolating an eigenvalue and push them left.
   */

S100:

  lll = l;
  for (j = lll; j <= k; j++) {
    col = 0.;
    for (i = l; i <= k; i++) {
      if (i != j) {
        col += fabs(AA(i,j));
      }
    }
    if (col == 0.) {
      WK(l) = (double)j;
      if (j != l) {
        for (i = 1; i <= k; i++) {
          repl    = AA(i,j);
          AA(i,j) = AA(i,l);
          AA(i,l) = repl;
        }
        for (i = l; i <= m; i++) {
          repl    = AA(j,i);
          AA(j,i) = AA(l,i);
          AA(l,i) = repl;
        }
      }
      l++;
      goto S100;
    }
  }

  /*
   * Balance the submatrix in rows L through K
   */
  for (i = l; i <= k; i++) {
    WK(i) = 1.;
  }

  noconv = TRUE;
  while (noconv) {
    noconv = FALSE;
    for (i = l; i <= k; i++) {
      col = 0.;
      row = 0.;
      for (j = l; j <= k; j++) {
        if (j != i) {
          col += fabs(AA(j,i));
          row += fabs(AA(i,j));
        }
      }

      f = 1.;
      g = row/c5;
      h = col+row;

      while (col < g) {
        f   *= c5;
        col *= c6;
      }

      g = row*c5;

      while (col >= g) {
        f   /= c5;
        col /= c6;
      }

      /*
       * Now balance
       */
      if ((col+row)/f < c4*h) {
        WK(i)  *= f;
        noconv  = TRUE;
        for (j = l; j <= m; j++) {
          AA(i,j) /= f;
        }
        for (j = 1; j <= k; j++) {
          AA(j,i) *= f;
        }
      }
    }
  }

  if (k-1 >= l+1) {
    /*
     * Transfer A to a Hessenberg form.
     */
    for (n = l+1; n <= k-1; n++) {
      h       = 0.;
      WK(n+m) = 0.;
      scale   = 0.;
      /*
       * Scale column
       */
      for (i = n; i <= k; i++) {
        scale += fabs(AA(i,n-1));
      }
      if (scale != 0.) {
        for (i = k; i >= n; i--) {
          WK(i+m)  = AA(i,n-1)/scale;
          h       += SQR(WK(i+m));
        }
        g        = -F77_SIGN(sqrt(h),WK(n+m));
        h       -= WK(n+m)*g;
        WK(n+m) -= g;
        /*
         * Form (I-(U*UT)/H)*A
         */
        for (j = n; j <= m; j++) {
          f = 0.;
          for (i = k; i >= n; i--) {
            f += WK(i+m)*AA(i,j);
          }
          for (i = n; i <= k; i++) {
            AA(i,j) -= WK(i+m)*f/h;
          }
        }
        /*
         * Form (i-(u*ut)/h)*a*(i-(u*ut)/h)
         */
        for (i = 1; i <= k; i++) {
          f = 0.;
          for (j = k; j >= n; j--) {
            f += WK(j+m)*AA(i,j);
          }
          for (j = n; j <= k; j++) {
            AA(i,j) -= WK(j+m)*f/h;
          }
        }
        WK(n+m)   *= scale;
        AA(n,n-1)  = scale*g;
      }
    }

    for (n = k-2; n >= l; n--) {
      n1 = n+1;
      n2 = n+2;
      f = AA(n+1,n);
      if( f != 0.) {
        f *= WK(n+1+m);
        for (i = n+2; i <= k; i++) {
          WK(i+m) = AA(i,n);
        }
        if (n+1 <= k) {
          for (j = 1; j <= m; j++) {
            g = 0.;
            for (i = n+1; i <= k; i++) {
              g += WK(i+m)*EVEC(i,j);
            }
            g /= f;
            for (i = n+1; i <= k; i++) {
              EVEC(i,j) += g*WK(i+m);
            }
          }
        }
      }
    }
  }

  n = 1;
  for (i = 1; i <= m; i++) {
    for (j = n; j <= m; j++) {
      rnorm += fabs(AA(i,j));
    }
    n = i;
    if (i < l || i > k) {
      EVAL(i) = AA(i,i);
    }
  }

  n = k;
  t = 0.;
  /*
   * Search for next eigenvalues
   */

S400:

  if (n < l) {
    goto S550;
  }

  in = 0;
  n1 = n-1;
  n2 = n-2;

  /*
   * Look for single small sub-diagonal element
   */

S410:

  for (i = l; i <= n; i++) {
    lb = n+l-i;
    if (lb == l) {
      break;
    }
    s = fabs(AA(lb-1,lb-1))+fabs(AA(lb,lb));
    if (s == 0.) {
      s = rnorm;
    }
    if (fabs(AA(lb,lb-1)) <= tol*s) {
      break;
    }
  }

  x = AA(n,n);
  if (lb == n) {
    /*
     * One eigenvalue found
     */
    AA(n,n) = x+t;
    EVAL(n) = AA(n,n);
    n       = n1;
    goto S400;
  }

  y = AA(n1,n1);
  w = AA(n,n1)*AA(n1,n);

  if (lb == n1) {
    /*
     * Two eigenvalues found
     */
    p         = (y-x)*c2;
    q         = p*p+w;
    z         = sqrt(fabs(q));
    AA(n,n)   = x+t;
    x         = AA(n,n);
    AA(n1,n1) = y+t;
    /*
     * Real pair
     */
    z        = p+F77_SIGN(z,p);
    EVAL(n1) = x+z;
    EVAL(n)  = EVAL(n1);

    if (z != 0.) {
      EVAL(n) = x-w/z;
    }
    x = AA(n,n1);
    /*
     * Employ scale factor in case X and Z are very small
     */
    r = sqrt(x*x+z*z);
    p = x/r;
    q = z/r;
    /*
     * Row modification
     */
    for (j = n1; j <= m; j++) {
      z        = AA(n1,j);
      AA(n1,j) =  q*z+p*AA(n,j);
      AA(n, j) = -p*z+q*AA(n,j);
    }
    /*
     * Column modification
     */
    for (i = 1; i <= n; i++) {
      z        = AA(i,n1);
      AA(i,n1) =  q*z+p*AA(i,n);
      AA(i,n ) = -p*z+q*AA(i,n);
    }
    /*
     * Accumulate transformations
     */
    for (i = l; i <= k; i++) {
      z          = EVEC(i,n1);
      EVEC(i,n1) =  q*z+p*EVEC(i,n);
      EVEC(i,n ) = -p*z+q*EVEC(i,n);
    }
    n = n2;
    goto S400;
  }

  if (in == 30) {
    /*
     * No convergence after 30 iterations; set error indicator to
     * the index of the current eigenvalue, and return.
     */
    *ier = n;
    return;
  }

  /*
   * Form shift
   */
  if (in == 10 || in == 20) {
    t += x;
    for (i = l; i <= n; i++) {
      AA(i,i) -= x;
    }
    s = fabs(AA(n,n1))+fabs(AA(n1,n2));
    x = c3*s;
    y = x;
    w = -c1*s*s;
  }

  in++;

  /*
   * Look for two consecutive small sub-diagonal elements
   */
  for (j = lb; j <= n2; j++) {
    i  = n2+lb-j;
    z  = AA(i,i);
    r  = x-z;
    s  = y-z;
    p  = (r*s-w)/AA(i+1,i)+AA(i,i+1);
    q  = AA(i+1,i+1)-z-r-s;
    r  = AA(i+2,i+1);
    s  = fabs(p)+fabs(q)+fabs(r);
    p /= s;
    q /= s;
    r /= s;

    if (i == lb) {
      break;
    }

    uu = fabs(AA(i,i-1))*(fabs(q)+fabs(r));
    vv = fabs(p)*(fabs(AA(i-1,i-1))+fabs(z)+fabs(AA(i+1,i+1)));

    if (uu <= tol*vv) {
      break;
    }
  }

  AA(i+2,i) = 0.;
  for (j = i+3; j <= n; j++) {
    AA(j,j-2) = 0.;
    AA(j,j-3) = 0.;
  }

  /*
   * Double QR step involving rows K to N and columns M to N
   */
  for (ka = i; ka <= n1; ka++) {
    notlas = (ka != n1);
    if (ka == i) {
      s = F77_SIGN(sqrt(p*p+q*q+r*r),p);
      if (lb != i) {
        AA(ka,ka-1) *= -1;
      }
    }
    else {
      p = AA(ka,  ka-1);
      q = AA(ka+1,ka-1);
      r = 0.;
      if (notlas) {
        r = AA(ka+2,ka-1);
      }
      x = fabs(p)+fabs(q)+fabs(r);
      if (x == 0.) {
        continue;
      }
      p /= x;
      q /= x;
      r /= x;
      s  = F77_SIGN(sqrt(p*p+q*q+r*r),p);

      AA(ka,ka-1) = -s*x;
    }

    p += s;
    x  = p/s;
    y  = q/s;
    z  = r/s;
    q /= p;
    r /= p;

    /*
     * Row modification
     */
    for (j = ka; j <= m; j++) {
      p = AA(ka,j)+q*AA(ka+1,j);
      if (notlas) {
        p          += r*AA(ka+2,j);
        AA(ka+2,j) -= p*z;
      }
      AA(ka+1,j) -= p*y;
      AA(ka,  j) -= p*x;
    }

    /*
     * Column modification
     */
    for (ii = 1; ii <= IMIN(n,ka+3); ii++) {
      p = x*AA(ii,ka)+y*AA(ii,ka+1);
      if (notlas) {
        p           += z*AA(ii,ka+2);
        AA(ii,ka+2) -= p*r;
      }
      AA(ii,ka+1) -= p*q;
      AA(ii,ka  ) -= p;
    }

    /*
     * Accumulate transformations
     */
    for (ii = l; ii <= k; ii++) {
      p = x*EVEC(ii,ka)+y*EVEC(ii,ka+1);
      if (notlas) {
        p             += z*EVEC(ii,ka+2);
        EVEC(ii,ka+2) -= p*r;
      }
      EVEC(ii,ka+1) -= p*q;
      EVEC(ii,ka  ) -= p;
    }
  }

  goto S410;

  /*
   * All evals found, now backsubstitute real vector
   */

S550:

  if (rnorm != 0.) {
    for (n = m; n >= 1; n--) {
      n2      = n;
      AA(n,n) = 1.;
      for (i = n-1; i >= 1; i--) {
        w = AA(i,i)-EVAL(n);
        if (w == 0.) {
          w = tol*rnorm;
        }
        r = AA(i,n);
        for (j = n2; j <= n-1; j++) {
          r += AA(i,j)*AA(j,n);
        }
        AA(i,n) = -r/w;
        n2      = i;
      }
    }
    /*
     * End backsubstitution vectors of isolated evals
     */
    for (i = 1; i <= m; i++) {
      if (i < l || i > k) {
        for (j = i; j <= m; j++) {
          EVEC(i,j) = AA(i,j);
        }
      }
    }
    /*
     * Multiply by transformation matrix
     */
    if (k != 0) {
      for (j = m; j >= l; j--) {
        for (i = l; i <= k; i++) {
          z = 0.;
          for (n = l; n <= IMIN(j,k); n++) {
            z += EVEC(i,n)*AA(n,j);
          }
          EVEC(i,j) = z;
        }
      }
    }
  }
  for (i = l; i <= k; i++) {
    for (j = 1; j <= m; j++) {
      EVEC(i,j) *= WK(i);
    }
  }

  /*
   * Interchange rows if permutations occurred
   */
  for (i = l-1; i >= 1; i--) {
    j = WK(i);
    if (i != j) {
      for (n = 1; n <= m; n++) {
        repl      = EVEC(i,n);
        EVEC(i,n) = EVEC(j,n);
        EVEC(j,n) = repl;
      }
    }
  }
  for (i = k+1; i <= m; i++) {
    j = WK(i);
    if (i != j) {
      for (n = 1; n <= m; n++) {
        repl      = EVEC(i,n);
        EVEC(i,n) = EVEC(j,n);
        EVEC(j,n) = repl;
      }
    }
  }

  return;
}

/*============================= end of c_asymmetric_matrix() ============*/
