#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_legendre_poly() =======================*/

/*
       Computes the normalized associated Legendre polynomial, defined
       in terms of the associated Legendre polynomial Plm = P-sub-l-super-m as

          Ylm(MU) = sqrt( (l-m)!/(l+m)! ) * Plm(MU)

       for fixed order m and all degrees from l = m to TWONM1.
       When m.GT.0, assumes that Y-sub(m-1)-super(m-1) is available
       from a prior call to the routine.

       REFERENCE: Dave, J.V. and B.H. Armstrong, Computations of High-Order
                    Associated Legendre Polynomials, J. Quant. Spectrosc. Radiat. Transfer 10,
                    557-562, 1970. (hereafter D/A)

       METHOD: Varying degree recurrence relationship.

       NOTES:
       (1) The D/A formulas are transformed by setting m=n-1; l=k-1.
       (2) Assumes that routine is called first with  m = 0, then with
           m = 1, etc. up to  m = twonm1.


  I N P U T     V A R I A B L E S:

       nmu    :  Number of arguments of YLM
       m      :  Order of YLM
       maxmu  :
       twonm1 :  Max degree of YLM
       MU(i)  :  Arguments of YLM (i = 1 to nmu)

       If m > 0, YLM(m-1,i) for i = 1 to nmu is assumed to exist from a prior call.


  O U T P U T     V A R I A B L E:

       YLM(l,i) :  l = m to twonm1, normalized associated Legendre polynomials
                   evaluated at argument MU(i)

   Called by- c_disort, c_albtrans
 -------------------------------------------------------------------*/

void c_legendre_poly(int     nmu,
                     int     m,
                     int     maxmu,
                     int     twonm1,
                     double *mu,
                     double *ylm)
{
  register int
    i,l;
  register double
    tmp1,tmp2;

  if (m == 0) {
    /*
     * Upward recurrence for ordinary Legendre polynomials
     */
    for (i = 1; i <= nmu; i++) {
      YLM(0,i) = 1.;
      YLM(1,i) = MU(i);
    }
    for (l = 2; l <= twonm1; l++) {
      for (i = 1; i <= nmu; i++) {
        YLM(l,i) = ((double)(2*l-1)*MU(i)*YLM(l-1,i)-(double)(l-1)*YLM(l-2,i))/l;
      }
    }
  }
  else {
    for (i = 1; i <= nmu; i++) {
      /*
       * Y-sub-m-super-m; derived from D/A eqs. (11,12), STWL(58c)
       */
      YLM(m,i) = -sqrt((1.-1./(2*m))*(1.-SQR(MU(i))))*YLM(m-1,i);

      /*
       * Y-sub-(m+1)-super-m; derived from D/A eqs.(13,14) using eqs.(11,12), STWL(58f)
       */
      YLM(m+1,i) = sqrt(2.*m+1.)*MU(i)*YLM(m,i);
    }
    /*
     * Upward recurrence; D/A eq.(10), STWL(58a)
     */
    for (l = m+2; l <= twonm1; l++) {
      tmp1 = sqrt((l-m  )*(l+m  ));
      tmp2 = sqrt((l-m-1)*(l+m-1));
      for (i = 1; i <= nmu; i++) {
        YLM(l,i) = ((double)(2*l-1)*MU(i)*YLM(l-1,i)-tmp2*YLM(l-2,i))/tmp1;
      }
    }
  }

  return;
}

/*============================= end of c_legendre_poly() ================*/
