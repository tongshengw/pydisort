#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
#undef  KK
#define KK(lyu) kk[lyu-1]
/*============================= c_chapman() ==============================*/

/*
 Calculates the Chapman factor.

 I n p u t       v a r i a b l e s:

      lc        : Computational layer
      taup      :
      tauc      :
      nlyr      : Number of layers in atmospheric model
      zd(lc)    : lc = 0, nlyr. zd(lc) is distance from bottom
                  surface to top of layer lc. zd(nlyr) = 0. km
      dtau_c    : Optical thickness of layer lc (un-delta-m-scaled)
      zenang    : Solar zenith angle as seen from bottom surface
      r         : Radial parameter, see Velinow & Kostov (2001). NOTE: Use the same dimension as zd,
                  for instance both in km.

 O u t p u t      v a r i a b l e s:

      ch        : Chapman-factor. In a pseudo-spherical atmosphere, replace exp(-tau/umu0) by exp(-ch(lc)) in the
                  beam source in

 I n t e r n a l     v a r i a b l e s:

      dhj       : delta-h-sub-j in eq. B2 (DS)
      dsj       : delta-s-sub-j in eq. B2 (DS)
      fact      : =1 for first  sum in eq. B2 (DS)
                  =2 for second sum in eq. B2 (DS)
      rj        : r-sub-j   in eq. B1 (DS)
      rjp1      : r-sub-j+1 in eq. B1 (DS)
      xpsinz    : The length of the line OG in Fig. 1, (DS)


 NOTE: Assumes a spherical planet. One might consider generalizing following
       Velinow YPI, Kostov VI, 2001, Generalization on Chapman Function for the Atmosphere of an Oblate Rotating Planet,
         Comptes Rendus de l'Academie Bulgare des Sciences 54, 29-34.
*/

double c_chapman(int     lc,
                 double  taup,
                 double *tauc,
                 int     nlyr,
                 double *zd,
                 double *dtau_c,
                 double  zenang,
                 double  r)
{
  register int
    id,j;
  double
    zenrad,xp,xpsinz,
    sum,fact,fact2,rj,rjp1,dhj,dsj;

  zenrad = zenang*DEG;
  xp     = r+ZD(lc)+(ZD(lc-1)-ZD(lc))*taup;
  xpsinz = xp*sin(zenrad);

  if (zenang > 90. && xpsinz < r) {
    return 1.e+20;
  }

  /*
   * Find index of layer in which the screening height lies
   */
  id = lc;
  if (zenang > 90.) {
    for (j= lc; j <= nlyr; j++) {
      if (xpsinz < (ZD(j-1)+r) && (xpsinz >= ZD(j)+r)) {
        id = j;
      }
    }
  }

  sum = 0.;
  for (j = 1; j <= id; j++) {
    fact  = 1.;
    fact2 = 1.;
    /*
     * Include factor of 2 for zenang > 90., second sum in eq. B2 (DS)
     */
    if (j > lc) {
      fact = 2.;
    }
    else if (j == lc && lc == id && zenang > 90.) {
      fact2 = -1.;
    }

    rj   = r+ZD(j-1);
    rjp1 = r+ZD(j  );
    if (j == lc && id == lc) {
      rjp1 = xp;
    }

    dhj = ZD(j-1)-ZD(j);
    if (id > lc && j == id) {
      dsj = sqrt(rj*rj-xpsinz*xpsinz);
    }
    else {
      dsj = sqrt(rj*rj-xpsinz*xpsinz)-fact2*sqrt(rjp1*rjp1-xpsinz*xpsinz);
    }
    sum += DTAU_C(j)*fact*dsj/dhj;
  }
  /*
   * Add third term in eq. B2 (DS)
   */
  if (id > lc) {
    dhj  = ZD(lc-1)-ZD(lc);
    dsj  = sqrt(xp*xp-xpsinz*xpsinz)-sqrt(SQR(ZD(lc)+r)-xpsinz*xpsinz);
    sum += DTAU_C(lc)*dsj/dhj;
  }

  return sum;
}

/*============================= end of c_chapman() =======================*/
