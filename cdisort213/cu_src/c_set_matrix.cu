#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_set_matrix() ==========================*/

/*
    Calculate coefficient matrix for the set of equations obtained from the
    boundary conditions and the continuity-of-intensity-at-layer-interface equations.

    Store in the special banded-matrix format required by LINPACK routines


    I N P U T      V A R I A B L E S:

       ds       :  Disort state variables
       bdr      :  surface bidirectional reflectivity
       cmu,cwt  :  abscissae, weights for Gauss quadrature over angle cosine
       delm0    :  Kronecker delta, delta-sub-m0
       gc       :  Eigenvectors at polar quadrature angles, SC(1)
       kk       :  Eigenvalues of coeff. matrix in eq. SS(7), STWL(23b)
       lyrcut   :  Logical flag for truncation of computational layers
       nn       :  Number of streams in a hemisphere (NSTR/2)
       ncut     :  Total number of computational layers considered
       taucpr   :  Cumulative optical depth (delta-M-scaled)

   O U T P U T     V A R I A B L E S:

       cband    :  Left-hand side matrix of linear system eq. SC(5), scaled by eq. SC(12);
                   in banded form required by LINPACK solution routines
       ncol     :  Number of columns in cband


   I N T E R N A L    V A R I A B L E S:

       irow     :  Points to row in CBAND
       jcol     :  Points to position in layer block
       lda      :  Row dimension of CBAND
       ncd      :  Number of diagonals below or above main diagonal
       nshift   :  For positioning number of rows in band storage
       wk       :  Temporary storage for EXP evaluations


   BAND STORAGE

      LINPACK requires band matrices to be input in a special
      form where the elements of each diagonal are moved up or
      down (in their column) so that each diagonal becomes a row.
      (The column locations of diagonal elements are unchanged.)

      Example:  if the original matrix is

          11 12 13  0  0  0
          21 22 23 24  0  0
           0 32 33 34 35  0
           0  0 43 44 45 46
           0  0  0 54 55 56
           0  0  0  0 65 66

      then its LINPACK input form would be:

           *  *  *  +  +  +  , * = not used
           *  * 13 24 35 46  , + = used for pivoting
           * 12 23 34 45 56
          11 22 33 44 55 66
          21 32 43 54 65  *

      If A is a band matrix, the following program segment
      will convert it to the form (ABD) required by LINPACK
      band-matrix routines:

        n  = (column dimension of a, abd)
        ml = (band width below the diagonal)
        mu = (band width above the diagonal)
        m = ml+mu+1;
        for (j = 1; j <= n; j++) {
          i1 = IMAX(1,j-mu);
          i2 = IMIN(n,j+ml);
          for (i = i1; i <= i2; i++) {
            k = i-j+m;
            ABD(k,j) = A(i,j);
          }
        }

      This uses rows  ml+1 through  2*ml+mu+1  of ABD.
      The total number of rows needed in ABD is 2*ml+mu+1.
      In the example above, n = 6, ml = 1, mu = 2, and the
      row dimension of ABD must be >= 5.

   Called by- c_disort, c_albtrans
 -------------------------------------------------------------------*/

void c_set_matrix(disort_state *ds,
                  double       *bdr,
                  double       *cband,
                  double       *cmu,
                  double       *cwt,
                  double        delm0,
                  double       *dtaucpr,
                  double       *gc,
                  double       *kk,
                  int           lyrcut,
                  int          *ncol,
                  int           ncut,
                  double       *taucpr,
                  double       *wk)
{
  int
    mi     = ds->nstr/2,
    mi9m2  = 9*mi-2,
    nnlyri = ds->nstr*ds->nlyr,
    nn     = ds->nstr/2;
  register int
    iq,irow,jcol,jq,k,lc,lda,ncd,nncol,nshift;
  double
    expa,sum;

  memset(cband,0,mi9m2*nnlyri*sizeof(double));

  ncd    = 3*nn-1;
  lda    = 3*ncd+1;
  nshift = lda-2*ds->nstr+1;
  *ncol  = 0;

  /*
   * Use continuity conditions of eq. STWJ(17) to form coefficient matrix in STWJ(20);
   * employ scaling transformation STWJ(22)
   */
  for (lc = 1; lc <= ncut; lc++) {
    for (iq = 1; iq <= nn; iq++) {
      WK(iq) = exp(KK(iq,lc)*DTAUCPR(lc));
    }
    jcol = 0;
    for (iq = 1; iq <= nn; iq++) {
      *ncol += 1;
      irow   = nshift-jcol;
      for (jq = 1; jq <= ds->nstr; jq++) {
        CBAND(irow+ds->nstr,*ncol) =  GC(jq,iq,lc);
        CBAND(irow,         *ncol) = -GC(jq,iq,lc)*WK(iq);
        irow++;
      }
      jcol++;
    }

    for (iq = nn+1; iq <= ds->nstr; iq++) {
      *ncol += 1;
      irow = nshift-jcol;
      for (jq = 1; jq <= ds->nstr; jq++) {
        CBAND(irow+ds->nstr,*ncol) =  GC(jq,iq,lc)*WK(ds->nstr+1-iq);
        CBAND(irow,         *ncol) = -GC(jq,iq,lc);
        irow++;
      }
      jcol++;
    }
  }

  /*
   * Use top boundary condition of STWJ(20a) for first layer
   */
  jcol = 0;
  for (iq = 1; iq <= nn; iq++) {
    expa = exp(KK(iq,1)*TAUCPR(1));
    irow = nshift-jcol+nn;
    for (jq = nn; jq >= 1; jq--) {
      CBAND(irow,jcol+1) = GC(jq,iq,1)*expa;
      irow++;
    }
    jcol++;
  }

  for (iq = nn+1; iq <=ds->nstr; iq++) {
    irow = nshift-jcol+nn;
    for (jq = nn; jq >= 1; jq--) {
      CBAND(irow,jcol+1) = GC(jq,iq,1);
      irow++;
    }
    jcol++;
  }

  /*
   * Use bottom boundary condition of STWJ(20c) for last layer
   */
  nncol = *ncol-ds->nstr;
  jcol  = 0;
  for (iq = 1; iq <= nn; iq++) {
    nncol++;
    irow = nshift-jcol+ds->nstr;
    for (jq = nn+1; jq <= ds->nstr; jq++) {
      if (lyrcut || ( ds->flag.lamber && delm0 == 0. ) ) {
        /*
         * No azimuthal-dependent intensity if Lambert surface;
         * no intensity component if truncated bottom layer
         */
        CBAND(irow,nncol) = GC(jq,iq,ncut);
      }
      else {
        sum = 0.;
        for (k = 1; k <= nn; k++) {
          sum += CWT(k)*CMU(k)*BDR(jq-nn,k)*GC(nn+1-k,iq,ncut);
        }
        CBAND(irow,nncol) = GC(jq,iq,ncut)-(1.+delm0)*sum;
      }
      irow++;
    }
    jcol++;
  }

  for (iq = nn+1; iq <= ds->nstr; iq++) {
    nncol++;
    irow = nshift-jcol+ds->nstr;
    expa = WK(ds->nstr+1-iq);
    for (jq = nn+1; jq <= ds->nstr; jq++) {
      if (lyrcut || (ds->flag.lamber && delm0 == 0.)) {
        CBAND(irow,nncol) = GC(jq,iq,ncut)*expa;
      }
      else {
        sum = 0.;
        for (k = 1; k <= nn; k++) {
          sum += CWT(k)*CMU(k)*BDR(jq-nn,k)*GC(nn+1-k,iq,ncut);
        }
        CBAND(irow,nncol) = (GC(jq,iq,ncut)-(1.+delm0)*sum)*expa;
      }
      irow++;
    }
    jcol++;
  }

  return;
}

/*============================= end of c_set_matrix() ===================*/
