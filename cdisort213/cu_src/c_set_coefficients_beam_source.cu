#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_set_coefficients_beam_source() ========*/

/*
       Set coefficients in ks(7) for beam source

    I N P U T      V A R I A B L E S:

       cmu    :   Computational polar angles
       ch     :   The Chapman-factor to correct for pseudo-spherical geometry in the direct beam.
       chtau  :   The optical depth in spherical geometry.
       delmo  :   Kronecker delta, delta-sub-m0
       fbeam  :   incident beam radiation at top
       gl     :   Phase function Legendre coefficients multiplied by (2l+1) and single-scatter albedo
       lc:    :   layer index
       mazim  :   order of azimuthal component
       nstr   :   number of streams
       taucpr :   delta-m-scaled optical depth
       ylmc   :   Normalized associated Legendre polynomial at the quadrature angles -cmu-
       ylm0   :   Normalized associated Legendre polynomial at the beam angle

    O U T P U T     V A R I A B L E S:

       xba    :   alfa in eq. KS(7)
       xb0    :   x-sub-zero in KS(7)
       xb1    :   x-sub-one in KS(7)
       zj     :  Solution vector Z-sub-zero after solving eq. SS(19), STWL(24b)

   Called by- c_disort
 -------------------------------------------------------------------*/

void c_set_coefficients_beam_source(disort_state *ds,
				    double       *ch,
				    double       *chtau,
				    double       *cmu,
				    double        delm0,
				    double        fbeam,
				    double       *gl,
				    int           lc,
				    int           mazim,
				    int           nstr,
				    double       *taucpr,
				    double       *xba,
				    disort_pair  *xb,
				    double       *ylm0,
				    double       *ylmc,
				    double       *zj)
{

  register int
    iq,k;
  double
    deltat,sum,q0a,q2a,q0,q2;
  static double
    big;

  big    = sqrt(DBL_MAX)/1.e+10;

  /*     Calculate x-sub-zero in STWJ(6d)   */

  for (iq = 1; iq <= nstr; iq++) {
    sum = 0;
    for (k = mazim; k <= nstr-1; k++) {
      sum += GL(k,lc)*YLMC(k,iq)*YLM0(k);
    }
    ZJ(iq) = (2.-delm0)*fbeam*sum/(4.*M_PI);
  }

  q0a = exp( -CHTAU(lc-1) );
  q2a = exp( -CHTAU(lc) );

  /*     Calculate alfa coefficient  */

  deltat = TAUCPR(lc) - TAUCPR(lc-1);

  XBA(lc) = 1./CH(lc);

  if ( fabs(XBA(lc)) > big  &&  TAUCPR(lc) > 1.)  XBA(lc) = 0.0;

  if( fabs(XBA(lc)*TAUCPR(lc)) > log(big))	  XBA(lc) = 0.0;

  /*     Dither alfa if it is close to one of the quadrature angles */

  if (  fabs(XBA(lc)) > 0.00001 ) {
    for (iq = 1; iq <= nstr/2; iq++) {
      if (fabs((fabs(XBA(lc))-1.0/CMU(iq))/XBA(lc) ) < 0.05 ) XBA(lc) = XBA(lc) * 1.001;
    }
  }

  for (iq = 1; iq <= nstr; iq++) {

    q0 = q0a * ZJ(iq);
    q2 = q2a * ZJ(iq);

    /*     x-sub-zero and x-sub-one in Eqs. KS(48-49)   */

    XB1(iq,lc) = (1.0/deltat)*(q2*exp(XBA(lc)*TAUCPR(lc)) - q0*exp(XBA(lc)*TAUCPR(lc-1)));
    XB0(iq,lc) = q0 * exp(XBA(lc)*TAUCPR(lc-1)) - XB1(iq,lc)*TAUCPR(lc-1);

  }
  return;
}
/*============================= end of c_set_coefficients_beam_source() ====*/
