#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_disort() ==============================*/

/*-------------------------------------------------------------------------------*
 * Plane-parallel discrete ordinates radiative transfer program                  *
 * C version                                                                     *
 * Fortran ftp site: ftp://climate.gsfc.nasa.gov/pub/wiscombe/Multiple_Scatt/    *
 *-------------------------------------------------------------------------------*

  Calling Tree (omitting calls to c_errmsg, c_dbl_vector, c_int_vector, c_free_dbl_vector):

  c_disort-+-c_self_test-+-c_disort_state_alloc
         |               +-c_disort_out_alloc
         |               +-c_disort_state_free
         |               +-c_disort_out_free
         +-c_check_inputs-+-(c_write_bad_var)
         |                +-c_dref
         +-c_disort_set-+-c_gaussian_quadrature
         +-c_print_inputs
         +-c_albtrans-+-c_legendre_poly
         |            +-c_solve_eigen-+-c_asymmetric_matrix
         |            +-c_interp_eigenvec
         |            +-c_set_matrix
         |            +-(c_sgbco)
         |            +-c_solve1-+-(c_sgbsl)
         |            +-c_atltrin
         |            +-c_albtrans_spherical
         |            +-c_print_albtrans
         +-c_planck_func1
         +-c_legendre_poly
         +-c_surface_bidir-+-c_gaussian_quadrature
         |                 +-c_bidir_reflectivity
         +-c_solve_eigen-+-c_asymmetric_matrix
	 +-c_set_coefficients_beam_source
	 +-c_interp_coefficients_beam_source
         +-c_upbeam_pseudo_spherical-+-(c_sgeco)
         |                           +-(c_sgesl)
         +-c_upbeam-+-(c_sgeco)
         |          +-(c_sgesl)
         +-c_upbeam_general_source-+-(c_sgeco)
         |                         +-(c_sgesl)
         +-c_upisot-+-(c_sgeco)
         |          +-(c_sgesl)
         +-c_interp_eigenvec
         +-c_interp_source
         +-c_set_matrix
         +-c_solve0-+-(c_sgbco)
         |          +-(c_sgbsl)
         +-c_fluxes
         +-c_user_intensities
         +-c_intensity_components
         +-c_print_avg_intensities
         +-c_ratio
         +-c_intensity_correction-+-c_single_scat
         |                        +-c_secondary_scat-+-c_xi_func
         +-c_new_intensity_correction-+-c_single_scat
         |                            +-prep_double_scat_integr
         |                            +-c_new_secondary_scat-+-c_xi_func
         |                            +-calc_phase_squared
         +-c_print_intensities

 +-------------------------------------------------------------------+

  Index conventions (for all loops and all variable descriptions):

  iu       :  for user polar angles
  iq,jq,kq :  for computational polar angles ('quadrature angles')
  iq/2     :  for half the computational polar angles (just the ones in either 0-90 degrees, or 90-180 degrees)
  j        :  for user azimuthal angles
  k,l      :  for Legendre expansion coefficients or, alternatively, subscripts of associated Legendre polynomials
  lu       :  for user levels
  lc       :  for computational layers (each having a different single-scatter albedo and/or phase function)
  lev      :  for computational levels
  mazim    :  for azimuthal components in Fourier cosine expansion of intensity and phase function

 +------------------------------------------------------------------+

               I N T E R N A L    V A R I A B L E S

   AMB(iq/2,iq/2)....First  matrix factor in reduced eigenvalue problem of eqs. SS(12), STWJ(8E), STWL(23f) (used only in solve_eigen);
                     ab[].zero (see cdisort.h)
   APB(iq/2,iq/2)....Second matrix factor in reduced eigenvalue problem of eqs. SS(12), STWJ(8E), STWL(23f) (used only in solve_eigen);
                     ab[].one (see cdisort.h)
   ARRAY(iq,iq)......Scratch matrix for solve_eigen(), upbeam() and upisot()
                     (see each subroutine for definition)
   B()...............Right-hand side vector of eq. SC(5) going into SOLVE0,1;
                     returns as solution vector vector  L, the constants of integration
   BDR(iq/2,0:iq/2)..Bottom-boundary bidirectional reflectivity for a given azimuthal component.  First index always
                     refers to a computational angle.  Second index: if zero, refers to incident beam angle UMU0;
                     if non-zero, refers to a computational angle.
   BEM(iq/2).........Bottom-boundary directional emissivity at computational angles.
   bplanck...........Intensity emitted from bottom boundary
   callnum...........Number of surface calls
   CBAND()...........Matrix of left-hand side of the linear system eq. SC(5), scaled by eq. SC(12);
                     in banded form required by LINPACK solution routines
   CC(iq,iq).........C-sub-IJ in eq. SS(5)
   CH(lc)............The Chapman-factor to correct for pseudo-spherical geometry in the direct beam.
   CHTAU(lc).........The optical depth in spherical geometry.
   CMU(iq)...........Computational polar angles (Gaussian)
   CWT(iq)...........Quadrature weights corresponding to CMU
   corint............When set TRUE, correct intensities for delta-scaling effects (see Nakajima and Tanaka, 1988).
                     When FALSE, intensities are not corrected. In general, CORINT should be set true when beam
                     source is present (FBEAM is not zero) and DELTAM is TRUE in a problem including scattering.
                     However, execution is faster when CORINT is FALSE, and intensities outside the aureole may still be
                     accurate enough.  When CORINT is TRUE, it is important to have a sufficiently high order of
                     Legendre approximation of the phase function. This is because the intensities are corrected by
                     calculating the single-scattered radiation, for which an adequate representation of the phase
                     function is crucial.  In case of a low order Legendre approximation of an otherwise highly
                     anisotropic phase function, the intensities might actually be more accurate when corint is FALSE.
                     When only fluxes are calculated (ds->flag.onlyfl is TRUE), or there is no beam source (FBEAM=0.0), or there
                     is no scattering (SSALB =0. for all layers) corint is set FALSE by the code.
   delm0.............Kronecker delta, delta-sub-M0, where M = MAZIM is the number of the Fourier component in the
                     azimuth cosine expansion
   deltam............TRUE,  use delta-M method ( see Wiscombe, 1977 );
                     FALSE, do not use delta-M method.
                     In general, for a given number of streams, intensities and fluxes will be more accurate for phase functions
                     with a large forward peak if DELTAM is set true. Intensities close to the forward scattering
                     direction are often less accurate, however, when the delta-M method is applied. The intensity deltam
                     correction of Nakajima and Tanaka is used to improve the accuracy of the intensities.
   dither............Small quantity subtracted from single-scattering albedos of unity, in order to avoid using special
                     case formulas;  prevents an eigenvalue of exactly zero from occurring, which would cause an immediate overflow
   DTAUCPR(lc).......Computational-layer optical depths (delta-M-scaled if DELTAM = TRUE, otherwise equal to DTAUC)
   EMU(iu)...........Bottom-boundary directional emissivity at user angles.
   EVAL(iq)..........Temporary storage for eigenvalues of eq. SS(12)
   EVECC(iq,iq)......Complete eigenvectors of SS(7) on return from solve_eigen; stored permanently in  GC
   EXPBEA(lc)........Transmission of direct beam in delta-M optical depth coordinates
   FLYR(lc)..........Separated fraction in delta-M method
   GL(k,lc)..........Phase function Legendre polynomial expansion coefficients, calculated from PMOM by
                     including single-scattering albedo, factor 2K+1, and (if DELTAM=TRUE) the delta-M scaling
   GC(iq,iq,lc)......Eigenvectors at polar quadrature angles, g in eq. SC(1)
   GU(iu,iq,lc)......Eigenvectors interpolated to user polar angles (g  in eqs. SC(3) and S1(8-9), i.e. g without the l factor)
   IPVT(lc*iq).......Integer vector of pivot indices for LINPACK routines
   KK(iq,lc).........Eigenvalues of coeff. matrix in eq. SS(7)
   kconv.............Counter in azimuth convergence test
   LAYRU(lu).........Computational layer in which user output level UTAU(LU) is located
   LL(iq,lc).........Constants of integration L in eq. SC(1), obtained by solving scaled version of eq. SC(5)
   lyrcut............TRUE, radiation is assumed zero below layer ncut because of almost complete absorption
   naz...............Number of azimuthal components considered
   ncut..............Computational layer number in which absorption optical depth first exceeds ABSCUT
   OPRIM(lc).........Single scattering albedo after delta-M scaling
   pass1.............TRUE on first entry, FALSE thereafter
   PKAG(0:lc)........Integrated Planck function for internal emission
   PRNTU0(l).........logical flag to trigger printing of azimuthally-averaged intensities:
                       l    quantities printed
                      --    ------------------
                       0    azimuthally-averaged intensities at user
                               levels and computational polar angles
                       1    azimuthally-averaged intensities at user
                               levels and user polar angles
   PSI0(iq)..........Sum just after square bracket in  eq. SD(9); psi[].zero (see cdisort.h)
   PSI1(iq)..........Sum in  eq. STWL(31d); psi[].one
   RMU(iu,0:iq)......Bottom-boundary bidirectional reflectivity for a given azimuthal component.  First index always
                     refers to a user angle.  Second index: if zero, refers to incident beam angle UMU0;
                     if non-zero, refers to a computational angle.
   scat_yes..........int, TRUE if scattering, FALSE if not (added to C version)
   TAUC(0:lc)........Cumulative optical depth (un-delta-M-scaled)
   TAUCPR(0:lc)......Cumulative optical depth (delta-M-scaled if DELTAM = TRUE, otherwise equal to TAUC)
   tplanck...........Intensity emitted from top boundary
   UUM(iu,lu)........Expansion coefficients when the intensity (u-super-M) is expanded in Fourier cosine series
                     in azimuth angle
   U0C(iq,lu)........Azimuthally-averaged intensity at quadrature angle
   U0U(iu,lu)........If ds->flag.onlyfl = FALSE, azimuthally-averaged intensity at user angles and user levels
                     If ds->flag.onlyfl = TRUE, azimuthally-averaged intensity at computational
                     (Gaussian quadrature) angles and user levels; the corresponding quadrature angle cosines are
                     returned in UMU.
   UTAUPR(lu)........Optical depths of user output levels in delta-M coordinates; equal to UTAU(LU) if no delta-M
   WK(iq)............Scratch array
   XR0(lc)...........X-sub-zero in expansion of thermal source function preceding eq. SS(14)(has no mu-dependence); b-sub-zero in eq. STWL(24d)
   XR1(lc)...........X-sub-one in expansion of thermal source function; see eqs. SS(14-16); b-sub-one in STWL(24d)
   YLM0(l)...........Normalized associated Legendre polynomial of subscript L at the beam angle (not saved
                     as function of superscipt M)
   YLMC(l,iq)........Normalized associated Legendre polynomial of subscript L at the computational angles
                     (not saved as function of superscipt M)
   YLMU(l,iu)........Normalized associated Legendre polynomial of subscript L at the user angles
                     (not saved as function of superscipt M)
   Z()...............scratch array used in solve0(), albtrans() to solve a linear system for the constants of integration
   Z0(iq)............Solution vectors Z-sub-zero of eq. SS(16); zee[].zero (see cdisort.h)
   Z1(iq)............Solution vectors Z-sub-one  of eq. SS(16); zee[].one
   Z0U(iu,lc)........Z-sub-zero in eq. SS(16) interpolated to user angles from an equation derived from SS(16); zu[].zero (see cdisort.h)
   Z1U(iu,lc)........Z-sub-one  in eq. SS(16) interpolated to user angles from an equation derived from SS(16); zu[].one
   ZBEAM(iu,lc)......Particular solution for beam source
   ZGU(iu,lc)........General source function interpolated to user angles
   ZJ(iq)............Right-hand side vector  X-sub-zero in eq. SS(19), also the solution vector
                     Z-sub-zero after solving that system
   ZJG(iq)...........Right-hand side vector  X-sub-zero in eq. KS(10), also the solution vector
                     Z-sub-zero after solving that system for a general source constant over a layer
   ZZ(iq,lc).........Permanent storage for the beam source vectors ZJ
   ZZG(iq,lc)........Permanent storage for the beam source vectors ZJG
   ZPLK0(iq,lc)......Permanent storage for the thermal source vectors plk[].zero obtained by solving eq. SS(16)
   ZPLK1(iq,lc)......Permanent storage for the thermal source vectors plk[].one  obtained by solving eq. SS(16)

*/

int c_disort(disort_state  *ds,
	      disort_output *out,
        emission_func_t emi_func)
{
  static int
    self_tested = -1;
  int
    prntu0[2],
    corint,deltam,scat_yes,compare,lyrcut,needdeltam,
    iq,iu,j,kconv,l,lc,lev,lu,mazim,naz,ncol,ncos,ncut,nn;
  static int
    callnum=1;
  int
    *ipvt,
    *layru;
  ipvt = (int *)swappablemalloc(ds->nstr * ds->nlyr * sizeof(int));
  layru = (int *)swappablemalloc(ds->ntau * sizeof(int));

  double
    angcos,azerr,azterm,bplanck,cosphi,delm0,
    sgn,tplanck;
  double
    *array,*b,*bdr,*bem,*cband,*cc,*ch,*chtau,
    *cmu,*cwt, *dtaucpr,*emu,*eval,*evecc,*expbea,
    *flyr,*gc,*gl,*gu,*kk,*ll,
    *oprim,*phasa,*phast,*phasm,*phirad,*pkag,
    *rmu,*tauc,*taucpr,*u0c,*utaupr,*uum,
    *wk,*xba,*ylm0,*ylmc,*ylmu,
    *z,*zbeam,
    *zbeama,zbsa=0,*zj,*zjg,*zju,*zgu,*zz,*zzg;
  disort_pair
    *ab,*fl,*plk,*xr,*psi,*xb,*zbeamsp,*zbs,*zee,*zu;
  disort_triplet
    *zbu;
  const double
    dither = 100.*DBL_EPSILON;


  /* Set these here to ensure that memory is correctly allocated. */
  if (!ds->flag.usrtau) {
    ds->ntau = ds->nlyr+1;
  }
  if ( ((!ds->flag.usrang) || ds->flag.onlyfl)  && ( (!ds->flag.ibcnd) == SPECIAL_BC)) {
    ds->numu = ds->nstr;
  }
  if (ds->flag.usrang && ds->flag.ibcnd == SPECIAL_BC) {
    ds->numu *= 2;
  }

  if (self_tested == -1) {
    int
      prntu0_test[2] = {FALSE,FALSE};
    disort_state
      ds_test;
    disort_output
      out_test;
    /*
     * Set input values for self-test.
     * Be sure self_test() sets all print flags off.
     */
    self_tested = 0;
    compare     = FALSE;
    c_self_test(compare,prntu0_test,&ds_test,&out_test);
    c_disort(&ds_test,&out_test,emi_func);
  }

  /*
   * Determine whether there is scattering or not
   */
  scat_yes = FALSE;
  for (lc = 1; lc <= ds->nlyr; lc++) {
    if (SSALB(lc) > 0.) {
      scat_yes = TRUE;
      break;
    }
  }

  /*
   * Turn on delta-M tranformation
   */
  deltam = TRUE;

  /* delta-M scaling makes only sense if phase function has more
   * moments than streams
   */
  needdeltam = FALSE;
  if( deltam==TRUE ) {
    for (lc=1; lc<=ds->nlyr; lc++)
      if ( PMOM(ds->nstr,lc) != 0.0 )
	needdeltam = TRUE;
    if (needdeltam==FALSE)
      deltam=FALSE;
  }

  /*
   * Turn off intensity correction when only fluxes are calculated, there
   * is no beam source, no scattering, or delta-M transformation is not applied
   */
  corint = ds->flag.intensity_correction;
  if (ds->flag.onlyfl || ds->bc.fbeam == 0. || !scat_yes || !deltam)
    corint = FALSE;

  prntu0[0] = FALSE;
  prntu0[1] = FALSE;

  /*
   * Allocate zeroed memory
   */
  tauc = c_dbl_vector(0,ds->nlyr,"tauc");

  for (lc = 1; lc <= ds->nlyr; lc++) {
    if(SSALB(lc) == 1.) {
      SSALB(lc) = 1.-dither;
    }
    TAUC(lc) = TAUC(lc-1)+DTAUC(lc);
  }

  /* Check input dimensions and variables */
  int err = c_check_inputs(ds,scat_yes,deltam,corint,tauc,callnum);
  if (err) {
    free(tauc);
    return err;
  }

  /*-------------------------------------------------------------------------------------------*
   * Special case for getting albedo and transmissivity of medium for many beam angles at once *
   *-------------------------------------------------------------------------------------------*/

  if (ds->flag.ibcnd == SPECIAL_BC) {
    /*
     * Allocate zeroed memory
     */
    array    = c_dbl_vector(0,ds->nstr*ds->nstr-1,"array");
    b        = c_dbl_vector(0,ds->nstr*ds->nlyr-1,"b");
    bdr      = c_dbl_vector(0,((ds->nstr/2)+1)*(ds->nstr/2)-1,"bdr");
    cband    = c_dbl_vector(0,ds->nstr*ds->nlyr*(9*(ds->nstr/2)-2)-1,"cband");
    ch       = c_dbl_vector(0,ds->nlyr-1,"ch");
    chtau    = c_dbl_vector(0,(2*ds->nlyr+1)-1,"chtau");
    cc       = c_dbl_vector(0,ds->nstr*ds->nstr-1,"cc");
    cmu      = c_dbl_vector(0,ds->nstr-1,"cmu");
    cwt      = c_dbl_vector(0,ds->nstr-1,"cwt");
    dtaucpr  = c_dbl_vector(0,ds->nlyr-1,"dtaucpr");
    eval     = c_dbl_vector(0,(ds->nstr/2)-1,"eval");
    evecc    = c_dbl_vector(0,ds->nstr*ds->nstr-1,"evecc");
    expbea   = c_dbl_vector(0,ds->nlyr,"expbea");
    flyr     = c_dbl_vector(0,ds->nlyr-1,"flyr");
    gc       = c_dbl_vector(0,ds->nlyr*ds->nstr*ds->nstr-1,"gc");
    gl       = c_dbl_vector(0,ds->nlyr*(ds->nstr+1),"gl");
    gu       = c_dbl_vector(0,ds->nlyr*ds->nstr*ds->numu-1,"gu");
    kk       = c_dbl_vector(0,ds->nlyr*ds->nstr-1,"kk");
    ll       = c_dbl_vector(0,ds->nlyr*ds->nstr-1,"ll");
    oprim    = c_dbl_vector(0,ds->nlyr-1,"oprim");
    taucpr   = c_dbl_vector(0,ds->nlyr,"taucpr");
    utaupr   = c_dbl_vector(0,ds->ntau-1,"utaupr");
    wk       = c_dbl_vector(0,ds->nstr-1,"wk");
    ylmc     = c_dbl_vector(0,ds->nstr*(ds->nstr+1)-1,"ylmc");
    ylmu     = c_dbl_vector(0,(ds->numu)*(ds->nstr+1)-1,"ylmu");
    z        = c_dbl_vector(0,ds->nstr*ds->nlyr-1,"z");

    ab       = (disort_pair *)swappablecalloc((ds->nstr/2)*(ds->nstr/2),sizeof(disort_pair));
    if (!ab) {
      c_errmsg("disort alloc error for ab", DS_ERROR);
    }
    /*
     * Zero output arrays
     */
    if (!ds->flag.usrtau) {
      memset(ds->utau,0,ds->ntau*sizeof(double));
    }
    if (!ds->flag.usrang || ds->flag.onlyfl) {
      memset(ds->umu,0,(ds->numu+1)*sizeof(double));
    }
    memset(out->rad,   0,ds->ntau*sizeof(disort_radiant));
    memset(out->albmed,0,ds->numu*sizeof(double));
    memset(out->trnmed,0,ds->numu*sizeof(double));
    if (ds->flag.onlyfl == FALSE) {
      memset(out->uu,0,ds->numu*ds->ntau*ds->nphi*sizeof(double));
    }

    /* Perform various setup operations */
    c_disort_set(ds,ch,chtau,cmu,cwt,deltam,dtaucpr,expbea,flyr,gl,layru,&lyrcut,&ncut,&nn,&corint,oprim,tauc,taucpr,utaupr,emi_func);

    /*  Print input information */
    if(ds->flag.prnt[0]) {
      c_print_inputs(ds,dtaucpr,scat_yes,deltam,corint,flyr,lyrcut,oprim,tauc,taucpr);
    }

    c_albtrans(ds,out,ab,array,b,bdr,cband,cc,cmu,cwt,dtaucpr,eval,evecc,gl,gc,gu,ipvt,kk,ll,nn,taucpr,ylmc,ylmu,z,wk);

    callnum++;

    /*
     * Free allocated memory
     */
    free(array), free(b),    free(bdr), free(cband),  free(cc),    free(ch);
    free(chtau), free(cmu),  free(cwt), free(dtaucpr),free(eval),  free(evecc);
    free(expbea),free(flyr), free(gc),  free(gl),     free(gu),    free(kk);
    free(ll),    free(oprim),free(tauc),free(taucpr), free(utaupr),free(wk);
    free(ylmc),  free(ylmu), free(z),   free(ab);

    return 0;
  }

  /*--------------*
   * General case *
   *--------------*/

  /*
   * Allocate zeroed memory
   */
  array   = c_dbl_vector(0,ds->nstr*ds->nstr-1,"array");
  b       = c_dbl_vector(0,ds->nstr*ds->nlyr-1,"b");
  bdr     = c_dbl_vector(0,((ds->nstr/2)+1)*(ds->nstr/2)-1,"bdr");
  bem     = c_dbl_vector(0,(ds->nstr/2)-1,"bem");
  cband   = c_dbl_vector(0,ds->nstr*ds->nlyr*(9*(ds->nstr/2)-2)-1,"cband");
  cc      = c_dbl_vector(0,ds->nstr*ds->nstr-1,"cc");
  ch      = c_dbl_vector(0,ds->nlyr-1,"ch");
  chtau   = c_dbl_vector(0,(2*ds->nlyr+1)-1,"chtau");
  cmu     = c_dbl_vector(0,ds->nstr-1,"cmu");
  cwt     = c_dbl_vector(0,ds->nstr-1,"cwt");
  dtaucpr = c_dbl_vector(0,ds->nlyr-1,"dtaucpr");
  emu     = c_dbl_vector(0,ds->numu-1,"emu");
  eval    = c_dbl_vector(0,(ds->nstr/2)-1,"eval");
  evecc   = c_dbl_vector(0,ds->nstr*ds->nstr-1,"evecc");
  expbea  = c_dbl_vector(0,ds->nlyr,"expbea");
  flyr    = c_dbl_vector(0,ds->nlyr,"flyr");    // We need at least one element
  gc      = c_dbl_vector(0,ds->nlyr*ds->nstr*ds->nstr-1,"gc");
  gl      = c_dbl_vector(0,ds->nlyr*(ds->nstr+1),"gl");
  gu      = c_dbl_vector(0,ds->nlyr*ds->nstr*ds->numu-1,"gu");
  kk      = c_dbl_vector(0,ds->nlyr*ds->nstr-1,"kk");
  ll      = c_dbl_vector(0,ds->nlyr*ds->nstr-1,"ll");
  oprim   = c_dbl_vector(0,ds->nlyr-1,"oprim");
  phasa   = c_dbl_vector(0,ds->nlyr-1,"phasa");
  phast   = c_dbl_vector(0,ds->nlyr-1,"phast");
  phasm   = c_dbl_vector(0,ds->nlyr-1,"phasm");
  if (ds->nphi > 0) {
    phirad = c_dbl_vector(0,ds->nphi-1,"phirad");
  }
  else {
    phirad = NULL;
  }
  pkag   = c_dbl_vector(0,ds->nlyr,"pkag");
  rmu    = c_dbl_vector(0,((ds->nstr/2)+1)*ds->numu-1,"rmu");
  taucpr = c_dbl_vector(0,ds->nlyr,"taucpr");
  u0c    = c_dbl_vector(0,ds->ntau*ds->nstr-1,"u0c");
  utaupr = c_dbl_vector(0,ds->ntau-1,"utaupr");
  uum    = c_dbl_vector(0,ds->ntau*ds->numu-1,"uum");
  wk     = c_dbl_vector(0,ds->nstr-1,"wk");
  xba    = c_dbl_vector(0,ds->nlyr,"xba");
  ylm0   = c_dbl_vector(0,ds->nstr,"ylm0");
  ylmc   = c_dbl_vector(0,ds->nstr*(ds->nstr+1)-1,"ylmc");
  ylmu   = c_dbl_vector(0,ds->numu*(ds->nstr+1)-1,"ylmu");
  z      = c_dbl_vector(0,ds->nstr*ds->nlyr-1,"z");
  zbeam  = c_dbl_vector(0,ds->nlyr*ds->numu-1,"zbeam");
  zbeama = c_dbl_vector(0,ds->nlyr,"zbeama");
  zj     = c_dbl_vector(0,ds->nstr-1,"zj");
  zjg    = c_dbl_vector(0,ds->nstr-1,"zjg");
  zju    = c_dbl_vector(0,ds->numu,"zju");
  zgu    = c_dbl_vector(0,ds->nlyr*ds->numu-1,"zgu");
  zz     = c_dbl_vector(0,ds->nlyr*ds->nstr-1,"zz");
  zzg    = c_dbl_vector(0,ds->nlyr*ds->nstr-1,"zzg");
  /*
   * Using C structures facilitates cache-aware memory allocation, which can reduce
   * cache misses and potentially speed up computer execution.
   */
  fl     = (disort_pair *)swappablecalloc(ds->ntau,sizeof(disort_pair));                  if (!fl)      c_errmsg("disort alloc error for fl", DS_ERROR);
  plk    = (disort_pair *)swappablecalloc(ds->nlyr*ds->nstr,sizeof(disort_pair));         if (!plk)     c_errmsg("disort alloc error for plk",DS_ERROR);
  ab     = (disort_pair *)swappablecalloc((ds->nstr/2)*(ds->nstr/2),sizeof(disort_pair)); if (!ab)      c_errmsg("disort alloc error for ab", DS_ERROR);
  xr     = (disort_pair *)swappablecalloc(ds->nlyr,sizeof(disort_pair));                  if (!xr)      c_errmsg("disort alloc error for xr", DS_ERROR);
  psi    = (disort_pair *)swappablecalloc(ds->nstr,sizeof(disort_pair));                  if (!psi)     c_errmsg("disort alloc error for psi",DS_ERROR);
  xb     = (disort_pair *)swappablecalloc(ds->nlyr*ds->nstr,sizeof(disort_pair));         if (!xb)      c_errmsg("disort alloc error for xb",DS_ERROR);
  zbs    = (disort_pair *)swappablecalloc(ds->nstr,sizeof(disort_pair));                  if (!zbs)     c_errmsg("disort alloc error for zbs",DS_ERROR);
  zbeamsp= (disort_pair *)swappablecalloc(ds->nlyr*ds->nstr,sizeof(disort_pair));         if (!zbeamsp) c_errmsg("disort alloc error for zbeamsp",DS_ERROR);
  zee    = (disort_pair *)swappablecalloc(ds->nstr,sizeof(disort_pair));                  if (!zee)     c_errmsg("disort alloc error for zee",DS_ERROR);
  zu     = (disort_pair *)swappablecalloc(ds->nlyr*ds->numu,sizeof(disort_pair));         if (!zu)      c_errmsg("disort alloc error for zu", DS_ERROR);

  zbu    = (disort_triplet *)swappablecalloc(ds->nlyr*ds->numu,sizeof(disort_triplet));   if (!zbu)     c_errmsg("disort alloc error for zbu", DS_ERROR);

  /*
   * Zero output arrays
   */
  if (!ds->flag.usrtau) {
    memset(ds->utau,0,ds->ntau*sizeof(double));
  }
  if (!ds->flag.usrang || ds->flag.onlyfl) {
    memset(ds->umu,0,(ds->numu)*sizeof(double));
  }
  memset(out->rad,0,ds->ntau*sizeof(disort_radiant));
  if (ds->flag.onlyfl == FALSE) {
    memset(out->uu,0,ds->numu*ds->ntau*ds->nphi*sizeof(double));
  }

  /* Perform various setup operations */
  c_disort_set(ds,ch,chtau,cmu,cwt,deltam,dtaucpr,expbea,flyr,gl,layru,&lyrcut,&ncut,&nn,&corint,oprim,tauc,taucpr,utaupr,emi_func);


  /*  Print input information */
  if(ds->flag.prnt[0]) {
    c_print_inputs(ds,dtaucpr,scat_yes,deltam,corint,flyr,lyrcut,oprim,tauc,taucpr);
  }

  /*
   * Calculate Planck functions
   */
  if (!ds->flag.planck) {
    bplanck = 0.;
    tplanck = 0.;
  }
  else {
    tplanck = emi_func(ds->wvnmlo,ds->wvnmhi,ds->bc.ttemp)*ds->bc.temis;
    bplanck = emi_func(ds->wvnmlo,ds->wvnmhi,ds->bc.btemp);
    for (lev = 0; lev <= ds->nlyr; lev++) {
      PKAG(lev) = emi_func(ds->wvnmlo,ds->wvnmhi,TEMPER(lev));
    }
  }

  /*
   *--------  BEGIN LOOP TO SUM AZIMUTHAL COMPONENTS OF INTENSITY  ---------
   *          (eq STWJ 5, STWL 6)
   */
  kconv = 0;
  naz   = ds->nstr-1;

  /*
   * Azimuth-independent case
   */
  if (ds->bc.fbeam == 0.                         ||
      fabs(1.-ds->bc.umu0) < 1.e-5               ||
      ds->flag.onlyfl                            ||
      (ds->numu == 1 && fabs(1.-UMU(1)) < 1.e-5) ||
      (ds->numu == 1 && fabs(1.+UMU(1)) < 1.e-5) ||
      (ds->numu == 2 && fabs(1.+UMU(1)) < 1.e-5 && fabs(1.-UMU(2)) < 1.e-5)) {
    naz = 0;
  }

  for (mazim = 0; mazim <= naz; mazim++) {
    if (mazim == 0) {
      delm0 = 1.;
    }
    else {
      delm0 = 0.;
    }

    /*
     * Get normalized associated Legendre polynomials for
     *   (a) incident beam angle cosine
     *   (b) computational and user polar angle cosines
     */
    if (ds->bc.fbeam > 0.) {
      ncos   = 1;
      angcos = -ds->bc.umu0;
      c_legendre_poly(ncos,mazim,ds->nstr,ds->nstr-1,&angcos,ylm0);
    }

    if (!ds->flag.onlyfl && ds->flag.usrang) {
      c_legendre_poly(ds->numu,mazim,ds->nstr,ds->nstr-1,ds->umu,ylmu);
    }
    c_legendre_poly(nn,mazim,ds->nstr,ds->nstr-1,cmu,ylmc);

    /*
     * Get normalized associated Legendre polynomials with negative arguments from those with
     * positive arguments; Dave/Armstrong eq. (15), STWL(59)
     */
    sgn = -1.;
    for (l = mazim; l <= ds->nstr-1; l++) {
      sgn *= -1.;
      for (iq = nn+1; iq <= ds->nstr; iq++) {
        YLMC(l,iq) = sgn*YLMC(l,iq-nn);
      }
    }

    /*
     * Specify users bottom reflectivity and emissivity properties
     */
    if (!lyrcut) {
      c_surface_bidir(ds, delm0, cmu, mazim, nn, bdr, emu, bem, rmu,
		      callnum);
    }

    /*--------------  BEGIN LOOP ON COMPUTATIONAL LAYERS  ------------*/
    for (lc = 1; lc <= ncut; lc++) {
      /*
       * Solve eigenfunction problem in eq. STWJ(8B), STWL(23f); return eigenvalues and eigenvectors
       */
      c_solve_eigen(ds,lc,ab,array,cmu,cwt,gl,mazim,nn,ylmc,cc,evecc,eval,kk,gc,wk);
      /*
       * Calculate particular solutions of eq. SS(18), STWL(24a) for incident beam source
       */
      if (ds->bc.fbeam > 0.) {
	if ( ds->flag.spher == TRUE ) {
	  /* Pseudo-spherical approach */
	  c_set_coefficients_beam_source(ds,ch,chtau,cmu,delm0,ds->bc.fbeam,
					 gl,lc,mazim,ds->nstr,
					 taucpr,xba,xb,ylm0,ylmc,zj);

	  if ( ds->flag.usrang == TRUE  ) {
	    /* Get coefficients at umu for pseudo-spherical source */
	    c_interp_coefficients_beam_source(ds,chtau,delm0,ds->bc.fbeam,
					      gl,lc,mazim,ds->nstr,
					      ds->numu,taucpr,zbu,
					      xba,zju,ylm0,ylmu);
	  }
	  c_upbeam_pseudo_spherical(ds,lc,array,cc,cmu,ipvt,nn,wk,
				    xb,xba,zbs,&zbsa,zbeamsp,zbeama);
	}
	else {
	  /* Plane-parallel version */
	  c_upbeam(ds,lc,array,cc,cmu,delm0,gl,ipvt,mazim,nn,wk,ylm0,ylmc,zj,zz);
	}
      }

      /*
       * Calculate particular solutions of eq. SS(18), STWL(24a), KS(5) for
       * general user specified source.
       */
      if (ds->flag.general_source) {
	c_upbeam_general_source(ds,lc,mazim,array,cc,ipvt,nn,wk,zjg,zzg);
      }

      /*
       * Calculate particular solutions of eq. SS(15), STWL(25) for thermal emission source
       */
      if (ds->flag.planck && mazim == 0) {
        XR1(lc) = 0.;
        if (DTAUCPR(lc) > 1e-4) { /* fix by RPB, caused problems in make check AVHRR CH4/5 */
          XR1(lc) = (PKAG(lc)-PKAG(lc-1))/DTAUCPR(lc);
        }
        XR0(lc) = PKAG(lc-1)-XR1(lc)*TAUCPR(lc-1);
        c_upisot(ds,lc,array,cc,cmu,ipvt,nn,oprim,wk,xr,zee,plk);
      }

      if (!ds->flag.onlyfl && ds->flag.usrang) {
        /*
         * Interpolate eigenvectors to user angles
         */
        c_interp_eigenvec(ds,lc,cwt,evecc,gl,gu,mazim,nn,wk,ylmc,ylmu);
        /*
         * Interpolate source terms to user angles
         */
        c_interp_source(ds,lc,cwt,delm0,gl,mazim,oprim,ylm0,ylmc,ylmu,
			psi,xr,zee,zj,zjg,zbeam,zbu,zbs,zbsa,zgu,zu);
      }
    }
    /*-------------------  END LOOP ON COMPUTATIONAL LAYERS  ----------------*/

    /*
     *
     * Set coefficient matrix of equations combining boundary and layer interface conditions
     */
    c_set_matrix(ds,bdr,cband,cmu,cwt,delm0,dtaucpr,gc,kk,lyrcut,&ncol,ncut,taucpr,wk);

    /*
     * Solve for constants of integration in homogeneous solution (general boundary conditions)
     */
    c_solve0(ds,b,bdr,bem,bplanck,cband,cmu,cwt,expbea,ipvt,ll,lyrcut,
	     mazim,ncol,ncut,nn,tplanck,taucpr,z,zbeamsp,zbeama,zz,zzg,plk);

    /*
     * Compute upward and downward fluxes
     */
    if (mazim == 0) {
      c_fluxes(ds,out,ch,cmu,cwt,gc,kk,layru,ll,lyrcut,ncut,nn,PRNTU0(1),
	       taucpr,utaupr,xr,zbeamsp,zbeama,zz,zzg,plk,fl,u0c);
    }

    if (ds->flag.onlyfl) {
      /*
       * Save azimuthal-avg intensities at quadrature angles
       */
      for (lu = 1; lu <= ds->ntau; lu++) {
        for (iq = 1; iq <= ds->nstr; iq++) {
          U0U(iq,lu) = U0C(iq,lu);
        }
      }
      break;
    }

    memset(uum,0,ds->numu*ds->ntau*sizeof(double));

    if (ds->flag.usrang) {
      /*
       * Compute azimuthal intensity components at user angles
       */
      c_user_intensities(ds,bplanck,cmu,cwt,delm0,dtaucpr,emu,expbea,
			 gc,gu,kk,layru,ll,lyrcut,mazim,
			 ncut,nn,rmu,taucpr,tplanck,utaupr,wk,
			 zbu,zbeam,zbeamsp,
			 zbeama,zgu,zu,zz,zzg,plk,uum);
    }
    else {
      /*
       * Compute azimuthal intensity components at quadrature angles
       */
      c_intensity_components(ds,gc,kk,layru,ll,lyrcut,mazim,ncut,nn,taucpr,utaupr,zz,plk,uum);
    }

    if (mazim == 0) {
      /*
       * Save azimuthally averaged intensities
       */
      for (lu = 1; lu <= ds->ntau; lu++) {
        for (iu = 1; iu <= ds->numu; iu++) {
          U0U(iu,lu) = UUM(iu,lu);
          for (j = 1; j <= ds->nphi; j++) {
            UU(iu,lu,j) = UUM(iu,lu);
          }
        }
      }

      if ( ds->flag.output_uum)
	for (lu = 1; lu <= ds->ntau; lu++)
	  for (iu = 1; iu <= ds->numu; iu++)
            OUT_UUM(iu,lu,mazim) = UUM(iu,lu);

      /*
       * Print azimuthally averaged intensities at user angles
       */
      if (PRNTU0(2)) {
        c_print_avg_intensities(ds,out);
      }

      if (naz > 0) {
        memset(phirad,0,ds->nphi*sizeof(double));
        for (j = 1; j <= ds->nphi; j++) {
          PHIRAD(j) = (PHI(j)-ds->bc.phi0)*DEG;
        }
      }
    }
    else {
      /*
       * Increment intensity by current azimuthal component (Fourier cosine series);  eq SD(2), STWL(6)
       */
      azerr = 0.;
      for (j = 1; j <= ds->nphi; j++) {
        cosphi = cos((double)mazim*PHIRAD(j));
        for (lu = 1; lu <= ds->ntau; lu++) {
          for (iu = 1; iu <= ds->numu; iu++) {
            azterm       = UUM(iu,lu)*cosphi;
            UU(iu,lu,j) += azterm;
            azerr        = MAX(azerr,c_ratio(fabs(azterm),fabs(UU(iu,lu,j))));
          }
        }
      }
      if ( ds->flag.output_uum)
	for (lu = 1; lu <= ds->ntau; lu++)
	  for (iu = 1; iu <= ds->numu; iu++)
            OUT_UUM(iu,lu,mazim) = UUM(iu,lu);

      if(azerr <= ds->accur) {
        kconv++;
      }
      if (kconv >= 2) {
        break;
      }
    }
  }
  /*--------------  END LOOP ON AZIMUTHAL COMPONENTS  ----------------*/



  for (iu = 1; iu <= ds->numu; iu++) {
    lu = ds->ntau;
    j =  1;
  }
  if (corint) {
    /*
     * Apply Nakajima/Tanaka intensity corrections
     */
    if (!ds->flag.old_intensity_correction && self_tested == 1) {
      if (ds->flag.quiet==VERBOSE)
	printf("Using new intensity correction, with phase functions\n");
      c_new_intensity_correction(ds,out,dither,flyr,layru,lyrcut,ncut,oprim,phasa,phast,phasm,phirad,tauc,taucpr,utaupr);
    }
    else {
      if (ds->flag.quiet==VERBOSE)
	printf("Using original intensity correction, with phase moments\n");
      c_intensity_correction(ds,out,dither,flyr,layru,lyrcut,ncut,oprim,phasa,phast,phasm,phirad,tauc,taucpr,utaupr);
    }
  }


  for (iu = 1; iu <= ds->numu; iu++) {
    lu = ds->ntau;
    j =  1;
  }


  if (ds->flag.prnt[2] && !ds->flag.onlyfl) {
    /*
     * Print intensities
     */
    c_print_intensities(ds,out);
  }

  if (self_tested == 0) {
    /*
     * Compare test case results with correct answers and abort if bad
     */
    compare = TRUE;
    c_self_test(compare,prntu0,ds,out);

    self_tested = 1;
  }

  callnum++;

  /*
   * Free allocated memory
   */
  free(ab),free(array);
  free(b),free(bdr),free(bem);
  free(cband),free(cc),free(ch),free(chtau),free(cmu),free(cwt);
  free(dtaucpr);
  free(emu),free(eval),free(evecc),free(expbea);
  free(flyr),free(fl);
  free(gc),free(gl),free(gu);
  free(kk);
  free(ll);
  free(oprim);
  free(phasa),free(phast),free(phasm),free(phirad),free(pkag),free(plk),free(psi);
  free(rmu);
  free(tauc),free(taucpr);
  free(u0c),free(utaupr),free(uum);
  free(wk);
  free(xb),free(xba),free(xr);
  free(ylm0),free(ylmc),free(ylmu);
  free(z),free(zbu),free(zbeam),free(zbeamsp),
  free(zbeama),free(zbs),free(zj),free(zjg),free(zju),
  free(zgu),free(zz),free(zzg),free(zee),free(zu);

  return 0;
}

/*============================= end of c_disort() =======================*/
