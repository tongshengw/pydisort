#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
#undef  KK
#define KK(lyu) kk[lyu-1]
/*============================= c_gaussian_quadrature_test() ============*/

int c_gaussian_quadrature_test(int nstr, float *sza, double umu0)
{

  /* Test if the solar zenith angle coincides with one of
     the computational angles */

  int nn=0, iq=0, result=0;
  double umu0s=0.0, *cmu=NULL, *cwt=NULL;

  cmu = c_dbl_vector(0,nstr,"cmu");
  if (cmu==NULL) {
    printf("Error allocating cmu!\n");
    return -1;
  }

  cwt = c_dbl_vector(0,nstr,"cwt");
  if (cwt==NULL) {
    printf("Error allocating cwt!\n");
    return -1;
  }

  nn = nstr / 2.0;

  c_gaussian_quadrature ( nn, cmu, cwt );

  for (iq=1; iq<=nn; iq++) {
    if( fabs( (umu0 - CMU (iq)) / umu0 ) < 1.0e-4 ) {
      umu0s = umu0;
      if ( umu0 < CMU (iq) )
	umu0  = CMU (iq) * (1. - 1.1e-4);
      else
	umu0  = CMU (iq) * (1. + 1.1e-4);

      *sza   = acos (umu0)/DEG;
      printf("%s %s %s %f %s %f\n",	      "******* WARNING >>>>>> \n",
	      "SETDIS--beam angle=computational angle;\n",
	      "******* changing cosine of solar zenith angle, umu0, from ",
	      umu0s, "to", umu0 );
      result=-1;
    }
  }

  free(cwt);
  free(cmu);
  return result;
}

/*============================= end of c_gaussian_quadrature_test() =====*/
