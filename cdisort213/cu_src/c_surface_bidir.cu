#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_surface_bidir() =======================*/

/*
       Computes user's' surface bidirectional properties, STWL(41)

   I N P U T     V A R I A B L E S:

       ds     :  Disort input variables
       cmu    :  Computational polar angle cosines (Gaussian)
       delm0  :  Kronecker delta, delta-sub-m0
       mazim  :  Order of azimuthal component
       nn     :  Order of Double-Gauss quadrature (ds->nstr/2)
       callnum:  number of surface calls

    O U T P U T     V A R I A B L E S:

       bdr :  Fourier expansion coefficient of surface bidirectional
                 reflectivity (computational angles)
       rmu :  Surface bidirectional reflectivity (user angles)
       bem :  Surface directional emissivity (computational angles)
       emu :  Surface directional emissivity (user angles)

    I N T E R N A L     V A R I A B L E S:

       dref   :  Directional reflectivity
       gmu    :  The NMUG angle cosine quadrature points on (0,1)
                 NMUG is set in cdisort.h
       gwt    :  The NMUG angle cosine quadrature weights on (0,1)

   Called by- c_disort
   Calls- c_gaussian_quadrature, c_bidir_reflectivity
+---------------------------------------------------------------------*/

void c_surface_bidir(disort_state *ds,
                     double        delm0,
                     double       *cmu,
                     int           mazim,
                     int           nn,
                     double       *bdr,
                     double       *emu,
                     double       *bem,
                     double       *rmu,
		     int           callnum)
{
  static int
    pass1 = TRUE;
  register int
    iq,iu,jg,jq,k;
  double
    dref,sum;
  static double
    gmu[NMUG],gwt[NMUG];

  if (pass1) {
    pass1 = FALSE;
    c_gaussian_quadrature(NMUG/2,gmu,gwt);
    for (k = 1; k <= NMUG/2; k++) {
      GMU(k+NMUG/2) = -GMU(k);
      GWT(k+NMUG/2) =  GWT(k);
    }
  }

  memset(bdr,0,(ds->nstr/2)*((ds->nstr/2)+1)*sizeof(double));
  memset(bem,0,(ds->nstr/2)*sizeof(double));

  /*
   * Compute Fourier expansion coefficient of surface bidirectional reflectance
   * at computational angles eq. STWL (41)
   */
  if (ds->flag.lamber && mazim == 0) {
    for (iq = 1; iq <= nn; iq++) {
      BEM(iq) = 1.-ds->bc.albedo;
      for (jq = 0; jq <= nn; jq++) {
        BDR(iq,jq) = ds->bc.albedo;
      }
    }
  }
  else if (!ds->flag.lamber) {
    for (iq = 1; iq <= nn; iq++) {
      for (jq = 1; jq <= nn; jq++) {
        sum = 0.;
        for (k = 1; k <= NMUG; k++) {
          sum += GWT(k) *
	    c_bidir_reflectivity ( ds->wvnmlo, ds->wvnmhi, CMU(iq), CMU(jq),
				   M_PI * GMU(k), ds->flag.brdf_type, &ds->brdf, callnum)
	    * cos((double)mazim * M_PI * GMU(k) );
        }
        BDR(iq,jq) = .5*(2.-delm0)*sum;
      }
      if (ds->bc.fbeam > 0.) {
        sum = 0.;
        for(k = 1; k <= NMUG; k++) {
          sum += GWT (k) *
	    c_bidir_reflectivity ( ds->wvnmlo, ds->wvnmhi, CMU(iq), ds->bc.umu0,
				   M_PI * GMU(k), ds->flag.brdf_type, &ds->brdf, callnum )
	    * cos((double)mazim * M_PI * GMU(k) );
        }
        BDR(iq,0) = .5*(2.-delm0)*sum;
      }
    }
    if (mazim == 0) {
      /*
       * Integrate bidirectional reflectivity at reflection polar angle cosines -CMU- and incident angle
       * cosines -GMU- to get directional emissivity at computational angle cosines -CMU-.
       */
      for (iq = 1; iq <= nn; iq++) {
        dref = 0.;
        for (jg = 1; jg <= NMUG; jg++) {
          sum = 0.;
          for (k = 1; k <= NMUG/2; k++) {
            sum += GWT(k) * GMU(k) *
	      c_bidir_reflectivity ( ds->wvnmlo, ds->wvnmhi, CMU(iq), GMU(k),
				     M_PI * GMU(jg), ds->flag.brdf_type, &ds->brdf, callnum );
          }
          dref += GWT(jg)*sum;
        }
        BEM(iq) = 1.-dref;
      }
    }
  }
  /*
   * Compute Fourier expansion coefficient of surface bidirectional reflectance at user angles eq. STWL (41)
   */
  if(!ds->flag.onlyfl && ds->flag.usrang) {
    memset(emu,0,ds->numu*sizeof(double));
    memset(rmu,0,ds->numu*((ds->nstr/2)+1)*sizeof(double));
    for (iu = 1; iu <= ds->numu; iu++) {
      if (UMU(iu) > 0.) {
        if(ds->flag.lamber && mazim == 0) {
          for (iq = 0; iq <= nn; iq++) {
            RMU(iu,iq) = ds->bc.albedo;
          }
          EMU(iu) = 1.-ds->bc.albedo;
        }
        else if (!ds->flag.lamber) {
          for (iq = 1; iq <= nn; iq++) {
            sum = 0.;
            for (k = 1; k <= NMUG; k++) {
              sum += GWT(k) *
		c_bidir_reflectivity ( ds->wvnmlo, ds->wvnmhi, UMU(iu), CMU(iq),
				       M_PI * GMU(k), ds->flag.brdf_type, &ds->brdf, callnum )
		* cos( (double)mazim * M_PI * GMU(k) );
            }
            RMU(iu,iq) = .5*(2.-delm0)*sum;
          }
          if (ds->bc.fbeam > 0.) {
            sum = 0.;
            for (k = 1; k <= NMUG; k++) {
              sum += GWT(k) *
		c_bidir_reflectivity ( ds->wvnmlo, ds->wvnmhi, UMU(iu),
				       ds->bc.umu0, M_PI * GMU(k),
				       ds->flag.brdf_type, &ds->brdf, callnum )
		* cos( (double)mazim * M_PI * GMU(k) );
            }
            RMU(iu,0) = .5*(2.-delm0)*sum;
          }
          if (mazim == 0) {
            /*
             * Integrate bidirectional reflectivity at reflection angle cosines -UMU- and
             * incident angle cosines -GMU- to get directional emissivity at user angle cosines -UMU-.
             */
            dref = 0.;
            for (jg = 1; jg <= NMUG; jg++) {
              sum = 0.;
              for (k = 1; k <= NMUG/2; k++) {
                sum += GWT(k) * GMU(k) *
		  c_bidir_reflectivity ( ds->wvnmlo, ds->wvnmhi, UMU(iu), GMU(k),
					 M_PI*GMU(jg), ds->flag.brdf_type, &ds->brdf, callnum );
              }
              dref += GWT(jg)*sum;
            }
            EMU(iu) = 1.-dref;
          }
        }
      }
    }
  }

  return;
}

/*============================= end of c_surface_bidir() ================*/
