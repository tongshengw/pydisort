#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
#undef  KK
#define KK(lyu) kk[lyu-1]
/*============================= c_planck_func2() =========================*/

/*
  Computes Planck function integrated between two wavenumbers,
  except if wnmulo = wnmuhi, then the Planck function at wnumlo is returned

  I N P U T :  wnumlo : Lower wavenumber [inv cm] of spectral interval
               wnumhi : Upper wavenumber
               t      : Temperature [K]

  O U T P U T :  ans  : Integrated Planck function [Watts/sq m]
                         = integral (wnumlo to wnumhi) of 2h c*c nu*nu*nu/(exp(hc nu/(kT))-1),
                         where h = Plancks constant, c = speed of light, nu = wavenumber,
                         T=temperature,and k = Boltzmann constant

  REFERENCE : Specifications of the physical world: New value of the fundamental constants,
                Dimensions/N.B.S., Jan. 1974

  METHOD :  For  -wnumlo-  close to  -wnumhi-, a Simpson-rule quadrature is done
            to avoid ill-conditioning; otherwise

            (1)  For wavenumber (wnumlo or wnumhi) small, integral(0 to wnum) is calculated by expanding
                 the integrand in a power series and integrating term by term;

            (2)  Otherwise, integral(wnumlo/hi to infinity) is calculated by expanding the denominator of the
                 integrand in powers of the exponential and integrating term by term.

  ACCURACY :  At least 6 significant digits, assuming the physical constants are infinitely accurate

  ERRORS that are not trapped:

      * Power or exponential series may underflow, giving no significant digits.
        This may or may not be of concern, depending on the application.

      * Simpson-rule special case is skipped when denominator of integrand will cause overflow.
        In that case the normal procedure is used, which may be inaccurate if the wavenumber limits
        (wnumlo, wnumhi) are close together.
 ----------------------------------------------------------------------

        LOCAL VARIABLES

        a1,2,... :  Power series coefficients
        c2       :  h*c/k, in units cm*k (h = Planck's constant, c = speed of light, k = Boltzmann constant)
        D(I)     :  Exponential series expansion of integral of Planck function from wnumlo (i=1)
                    or wnumhi (i=2) to infinity
        ex       :  exp(-V(I))
        exm      :  pow(ex,m)
        mmax     :  No. of terms to take in exponential series
        mv       :  multiples of 'V(i)'
        P(I)     :  Power series expansion of integral of Planck function from zero to wnumlo (i=1) or wnumhi (i=2)
        sigma    :  Stefan-Boltzmann constant (W m-2 K-4)
        sigdpi   :  sigma/pi
        smallv   :  Number of times the power series is used (0,1,2)
        V(I)     :  c2*(wnumlo(i=1) or wnumhi(i=2))/temperature
        vcut     :  Power-series cutoff point
        vcp      :  Exponential series cutoff points
        vmax     :  Largest allowable argument of 'exp' function
  ----------------------------------------------------------------------*/

#define A1    (1./3.)
#define A2    (-1./8.)
#define A3    (1./60.)
#define A4    (-1./5040.)
#define A5    (1./272160.)
#define A6    (-1./13305600.)
#define C2    (1.438786)
#define SIGMA (5.67032e-8)
#define VCUT  (1.5)
#define PLKF(x) ({const double _x = (x); _x*_x*_x/(exp(_x)-1.);})

double __attribute__((weak)) c_planck_func2(double wnumlo,
                      double wnumhi,
                      double t)
{
  register int
    m,n,smallv,k,i,mmax;
  static int
    initialized = FALSE;
  double
    ans,del,val,val0,oldval,exm,
    ex,mv,vsq,wvn,arg,hh,
    d[2],p[2],v[2];
  const double
    vcp[7] = {10.25,5.7,3.9,2.9,2.3,1.9,0.0};
  static double
    sigdpi,vmax,conc,c1;

  if (!initialized) {
    sigdpi = SIGMA/M_PI;
    vmax   = log(DBL_MAX);
    conc   = 15./pow(M_PI,4.);
    c1     = 1.1911e-8;

    initialized = TRUE;
  }
  if (t < 0. || wnumhi < wnumlo || wnumlo < 0.) {
    c_errmsg("planck_func2--temperature or wavenumbers wrong",DS_ERROR);
  }
  if (t < 1.e-4) {
    return 0.;
  }
  if (wnumhi == wnumlo) {
    wvn    = wnumhi;
    arg    = exp(-C2*wvn/t);
    return c1*wvn*wvn*wvn*arg/(1.-arg);
  }

  v[0] = C2*wnumlo/t;
  v[1] = C2*wnumhi/t;

  if (v[0] > DBL_EPSILON && v[1] < vmax && (wnumhi-wnumlo)/wnumhi < 1.e-2) {
    /*
     * Wavenumbers are very close. Get integral by iterating Simpson rule to convergence.
     */
    hh     = v[1]-v[0];
    oldval = 0.;
    val0   = PLKF(v[0])+PLKF(v[1]);
    for (n = 1; n <= 10; n++) {
      del = hh/(2*n);
      val = val0;
      for (k = 1; k <=2*n-1; k++) {
        val += (double)(2*(1+k%2))*PLKF(v[0]+(double)k*del);
      }
      val *= del*A1;
      if (fabs((val-oldval)/val) <= 1.e-6) {
        return sigdpi*SQR(t*t)*conc*val;
      }
      oldval = val;
    }
    c_errmsg("planck_func2--Simpson rule did not converge",DS_WARNING);
    return sigdpi*SQR(t*t)*conc*val;
  }

  smallv = 0;
  for (i = 0; i <= 1; i++) {
    if(v[i] < VCUT) {
      /*
       * Use power series
       */
      smallv++;
      vsq  = v[i]*v[i];
      p[i] = conc*vsq*v[i]*(A1+v[i]*(A2+v[i]*(A3+vsq*(A4+vsq*(A5+vsq*A6)))));
    }
    else {
      /*
       * Use exponential series
       *
       * Find upper limit of series
       */
      mmax = 1;
      while (v[i] < vcp[mmax-1]) {
        mmax++;
      }

      ex   = exp(-v[i]);
      exm  = 1.;
      d[i] = 0.;

      for (m = 1; m <= mmax; m++) {
        mv    = (double)m*v[i];
        exm  *= ex;
        d[i] += exm*(6.+mv*(6.+mv*(3.+mv)))/SQR(m*m);
      }
      d[i] *= conc;
    }
  }

  if (smallv == 2) {
    /*
     * wnumlo and wnumhi both small
     */
    ans = p[1]-p[0];
  }
  else if (smallv == 1) {
    /*
     * wnumlo small, wnumhi large
     */
    ans = 1.-p[0]-d[1];
  }
  else {
    /*
     * wnumlo and wnumhi both large
     */
    ans = d[0]-d[1];
  }
  ans *= sigdpi*SQR(t*t);
  if (ans == 0.) {
    c_errmsg("planck_func2--returns zero; possible underflow",DS_WARNING);
  }

  return ans;
}

#undef A1
#undef A2
#undef A3
#undef A4
#undef A5
#undef A6
#undef C2
#undef SIGMA
#undef VCUT
#undef PLKF

/*============================= end of c_planck_func2() =================*/
