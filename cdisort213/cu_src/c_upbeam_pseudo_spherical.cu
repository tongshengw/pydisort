#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_upbeam_pseudo_spherical() =============*/

/*

       Finds the particular solution of beam source KS(10-11)

     Routines called:  sgeco, sgesl

   I N P U T     V A R I A B L E S:

       cc     :  capital-c-sub-ij in Eq. SS(5)
       cmu    :  abscissae for gauss quadrature over angle cosine
       xb0    :  EXPansion of beam source function Eq. KS(7)
       xb1    :  EXPansion of beam source function Eq. KS(7)
       xba    :  EXPansion of beam source function Eq. KS(7)
       (remainder are 'disort' input variables)

    O U T P U T    V A R I A B L E S:

       zbs0     :  solution vectors z-sub-zero of Eq. KS(10-11)
       zbs1     :  solution vectors z-sub-one  of Eq. KS(10-11)
       zbsa     :  alfa coefficient in Eq. KS(7)
       zbeam0,  :  permanent storage for -zbs0,zbs1,zbsa-, but rD-ordered
        zbeam1,
        zbeama

   I N T E R N A L    V A R I A B L E S:

       array  :  coefficient matrix in left-hand side of Eq. KS(10)
       ipvt   :  integer vector of pivot indices required by *linpack*
       wk     :  scratch array required by *linpack*

   Called by- c_disort
   Calls- c_sgeco, c_errmsg, c_sgesl
 -------------------------------------------------------------------*/

#undef  ARRAY
#define ARRAY(iq,jq) array[iq-1+(jq-1)*ds->nstr]

void c_upbeam_pseudo_spherical(disort_state *ds,
			       int           lc,
			       double       *array,
			       double       *cc,
			       double       *cmu,
			       int          *ipvt,
			       int           nn,
			       double       *wk,
			       disort_pair  *xb,
			       double       *xba,
			       disort_pair  *zbs,
			       double       *zbsa,
			       disort_pair  *zbeamsp,
			       double       *zbeama)
{

  register int
    iq,jq;
  double
    rcond,rmin;


  for (iq = 1; iq <= ds->nstr; iq++) {
    for (jq = 1; jq <= ds->nstr; jq++) {
      ARRAY(iq,jq) = -CC(iq,jq);
    }
    ARRAY(iq,iq) += 1.+XBA(lc)*CMU(iq);
    *zbsa     = XBA(lc);
    ZBS1(iq) = XB1(iq,lc);
  }

  /*
   * Find L-U (lower/upper triangular) decomposition of ARRAY and see
   * if it is nearly singular
   * (NOTE: ARRAY is altered)
   */

  rcond = 0.;
  c_sgeco(array,ds->nstr,ds->nstr,ipvt,&rcond,wk);

  if (1.+rcond == 1.) {
    c_errmsg("upbeam_pseudo_spherical--sgeco says matrix near singular",
	     DS_WARNING);
  }

  rmin = 1.0e-4;
  if ( rcond < rmin ) {
    /*     Dither alpha if rcond to small   */
    if(XBA(lc) ==0.0)       XBA(lc)=0.000000005;

    XBA(lc) = XBA(lc) * 1.00000005;

    for (iq = 1; iq <= ds->nstr; iq++) {
      for (jq = 1; jq <= ds->nstr; jq++) {
	ARRAY(iq,jq) = -CC(iq,jq);
      }
      ARRAY(iq,iq) += 1.0+XBA(lc)*CMU(iq);
      *zbsa     = XBA(lc);
      ZBS1(iq) = XB1(iq,lc);
    }
    /*     Solve linear equations KS(10-11) with dithered alpha */
    rcond = 0.;
    c_sgeco(array,ds->nstr,ds->nstr,ipvt,&rcond,wk);
    if (1.+rcond == 1.) {
      c_errmsg("upbeam_pseudo_spherical--sgeco says matrix near singular",
	       DS_WARNING);
    }
  }

  for (iq = 1; iq <= ds->nstr; iq++)  WK(iq) = ZBS1(iq);
  c_sgesl( array, ds->nstr, ds->nstr, ipvt, wk, 0 );

  for (iq = 1; iq <= ds->nstr; iq++) {
    ZBS1(iq) = WK(iq);
    ZBS0(iq) = XB0(iq,lc) + CMU(iq) * ZBS1(iq);
  }

  for (iq = 1; iq <= ds->nstr; iq++)  WK(iq) = ZBS0(iq);
  c_sgesl( array, ds->nstr, ds->nstr, ipvt, wk, 0 );
  for (iq = 1; iq <= ds->nstr; iq++)  ZBS0(iq) = WK(iq);

  /*   ... and now some index gymnastic for the inventive ones...  */

  ZBEAMA(lc)            = *zbsa;
  for (iq = 1; iq <= nn; iq++) {
    ZBEAM0( iq+nn, lc )   = ZBS0( iq );
    ZBEAM1( iq+nn, lc )   = ZBS1( iq );
    ZBEAM0( nn+1-iq, lc ) = ZBS0( iq+nn );
    ZBEAM1( nn+1-iq,lc )  = ZBS1( iq+nn );
  }

 return;

}


/*============================= end of c_upbeam_pseudo_spherical() ======*/
