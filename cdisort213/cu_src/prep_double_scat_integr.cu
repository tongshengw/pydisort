#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= prep_double_scat_integr () ==============*/

/*
       Prepares double scattering integration according to alternative
       Buras-Emde algorithm(201X).

                I N P U T   V A R I A B L E S

       nphase    number of angles for which original phase function
                     (ds->phase) is defined
       ntau
       nf        number of angular phase integration grid point
                     (zenith angle, theta)
       mu_phase  cos(theta) grid of phase function
       phas2     residual phase function

                O U T P U T   V A R I A B L E S

       mu_eq     cos(theta) phase integration grid points,
                     equidistant in abs(f_phas2)
       neg_phas  index whether phas2 is negative
       norm_phas normalization factor for phase integration

                I N T E R N A L   V A R I A B L E S

       f_phas2_abs absolute value of integrated phase function
                      phas2
       f_phas2     cumulative integrated phase function phas2
       df          step length for calculating mu_eq

   Called by- c_new_intensity_correction
   Calls- c_dbl_vector, locate
 -------------------------------------------------------------------*/

void prep_double_scat_integr (int nphase, int ntau,
			      int           nf,
			      double       *mu_phase,
			      double       *phas2,
			      double       *mu_eq,
			      int          *neg_phas,
			      double       *norm_phas)
{
  int it=0, i=0, lu=0;
  double *f_phas2_abs=NULL;
  double f_phas2=0.0, df=0.0;

  f_phas2_abs = c_dbl_vector(0,nphase,"f_phas2_abs");

  for (lu=1; lu<=ntau; lu++) {

    /* calculate integral of |phas2| (f_phas2_abs) */

    F_PHAS2_ABS(1) = 0.0;
    for (it=2; it<=nphase; it++)
      F_PHAS2_ABS(it) = F_PHAS2_ABS(it-1) +
	( MUP(it) - MUP(it-1) ) * 0.5 *
	( fabs( PHAS2(it,lu) ) + fabs ( PHAS2(it-1,lu) ) );

    /* define mu grid which is equidistant in f_phas2_abs (mu_eq);
       find areas of negative phas2 (neg_phas);
       define normalization (norm_phas) */

    f_phas2 = 0.0;
    df = F_PHAS2_ABS(nphase) / (nf-1);
    MU_EQ(1,lu) = -1.0;

    if ( PHAS2(1,lu) > 0.0 )
      NEG_PHAS(1,lu) = FALSE;
    else
      NEG_PHAS(1,lu) = TRUE;

    it = 1;
    for (i=2; i<=nf-1; i++) {
      f_phas2 += df;

      while ( F_PHAS2_ABS(it+1) < f_phas2 )
	it++;

      MU_EQ(i,lu) = MUP(it)
	+ ( f_phas2 - F_PHAS2_ABS(it) ) /
	( F_PHAS2_ABS(it+1) - F_PHAS2_ABS(it) ) *
	( MUP(it+1) - MUP(it) );

      if ( PHAS2(it,lu) > 0.0 && PHAS2(it+1,lu) > 0.0 )
	NEG_PHAS(i,lu) = FALSE;
      else {
	if ( PHAS2(it,lu) < 0.0 && PHAS2(it+1,lu) < 0.0 )
	  NEG_PHAS(i,lu) = TRUE;
	else {
	  if ( PHAS2(it,lu) + ( f_phas2 - F_PHAS2_ABS(it) ) /
	       ( F_PHAS2_ABS(it+1) - F_PHAS2_ABS(it) ) *
	       ( PHAS2(it+1,lu) - PHAS2(it,lu) ) > 0.0 )
	    NEG_PHAS(i,lu) = FALSE;
	  else
	    NEG_PHAS(i,lu) = TRUE;
	}
      }

    } /* end for i<nf */

    MU_EQ(nf,lu) = 1.0;
    if ( PHAS2(nphase,lu) > 0.0 )
      NEG_PHAS(nf,lu) = FALSE;
    else
      NEG_PHAS(nf,lu) = TRUE;

    NORM_PHAS(lu) = F_PHAS2_ABS(nphase) / ( (nf-1) * M_PI );

  } /* end for lu<ntau */

  free(f_phas2_abs);
}

/*============================= end of prep_double_scat_integr() ========*/
