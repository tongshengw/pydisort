#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_ratio() ===============================*/

/*
 * Calculate ratio a/b with overflow and underflow protection
 * (thanks to Prof. Jeff Dozier for some suggestions here).
 *
 * Modification in this C version: in the case b == 0., returns 1.+a.
 *
 * Called by: c_disort
 */

double c_ratio(double a,
             double b)
{
  static int
    initialized = FALSE;
  static double
    tiny,huge,powmax,powmin;
  double
    ans,absa,absb,powa,powb;

  if(!initialized) {
    tiny   = DBL_MIN;
    huge   = DBL_MAX;
    powmax = log10(huge);
    powmin = log10(tiny);

    initialized = TRUE;
  }

  if (c_fcmp(b,0.) == 0) {
    ans = 1.+a;
  }
  else if (c_fcmp(a,0.) == 0) {
    ans = 0.;
  }
  else {
    absa = fabs(a);
    absb = fabs(b);
    powa = log10(absa);
    powb = log10(absb);
    if (c_fcmp(absa,tiny) < 0 && c_fcmp(absb,tiny) < 0) {
      ans = 1.;
    }
    else if (c_fcmp(powa-powb,powmax) >= 0) {
      ans = huge;
    }
    else if(c_fcmp(powa-powb,powmin) <= 0) {
      ans = tiny;
    }
    else {
      ans = absa/absb;
    }

   /*
    * NOTE: Don't use old trick of determining sign from a*b because a*b
    *       may overflow or underflow.
    */
    if ( (a > 0. && b < 0.) || (a < 0. && b > 0.) ) {
      ans *= -1;
    }
  }

  return ans;
}

/*============================= end of c_ratio() ========================*/
