#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_sasum() ================================*/

/*
  Input--   n     Number of elements in vector to be summed
            sx    array, length n, containing vector

  OUTPUT--  ans   Sum from i = 1 to n of fabs(SX(i))

  NOTE: Fortran input incx removed because it is not used by
        disort or twostr
 ----------------------------------------------------------*/

double c_sasum(int     n,
             double *sx)
{
  register int
    i,m;
  double
    ans;

  ans = 0.;
  if (n <= 0) {
    return ans;
  }

  m = n%4;
  if (m != 0) {
    /*
     * clean-up loop so remaining vector length is a multiple of 4.
     */
    for (i = 1; i <= m; i++) {
      ans += fabs(SX(i));
    }
  }
  /*
   * unroll loop for speed
   */
  for (i = m+1; i <= n; i+=4) {
    ans += fabs(SX(i  ))
          +fabs(SX(i+1))
          +fabs(SX(i+2))
          +fabs(SX(i+3));
  }

  return ans;
}

/*============================= end of c_sasum() =========================*/
