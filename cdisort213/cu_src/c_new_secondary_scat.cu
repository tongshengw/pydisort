#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_new_secondary_scat() ==================*/

/*
   Calculates secondary scattered intensity, new method (see BDE)

                I N P U T   V A R I A B L E S

        ds        Disort state variables
        iu        index of user polar angle
        lu        index of user level
	it	  index where ctheta contained in mu grid of exact
	             phase function
        ctheta    cosine of scattering angle
        flyr      separated fraction f in Delta-M method
        layru     index of utau in multi-layered system
        tauc      cumulative optical depth at computational layers
        nf        number of angular phase integration grid point
                     (zenith angle, theta)
        phas2     residual phase function
        mu_eq     cos(theta) phase integration grid points,
                     equidistant in abs(f_phas2)
        neg_phas  index whether phas2 is negative
        norm_phas normalization factor for phase integration

                I N T E R N A L   V A R I A B L E S

        pspike  2*p"-p"*p", where p" is the residual phase function
        pspike1 2*p", where p" is the residual phase function
        pspike2 p"*p", where p" is the residual phase function
        wbar    mean value of single scattering albedo
        fbar    mean value of separated fraction f
        dtau    layer optical depth
        stau    sum of layer optical depths between top of atmopshere and layer layru
	umu0p
        nphase  number of angles for which original phase function
                   (ds->phase) is defined

   Called by- c_new_intensity_correction
   Calls- calc_phase_squared, c_xi_func
 -------------------------------------------------------------------*/

double c_new_secondary_scat(disort_state *ds,
			    int           iu,
			    int           lu,
			    int           it,
			    double        ctheta,
			    double       *flyr,
			    int           layru,
			    double       *tauc,
			    int           nf,
			    double       *phas2,
			    double       *mu_eq,
			    int          *neg_phas,
			    double        norm_phas)
{
  register int
    lyr;
  const double
    tiny = 1.e-4;
  double
    dtau,fbar,pspike,
    stau,umu0p,wbar;
  int nphase=ds->nphase;

  double pspike1=0.0, pspike2=0.0;

  /*
   * Calculate vertically averaged value of single scattering albedo and separated
   * fraction f, eq. STWL (A.15)
   */
  dtau = UTAU(lu)-TAUC(layru-1);
  wbar = SSALB(layru)*dtau;
  fbar = FLYR(layru)*wbar;
  stau = dtau;
  for (lyr = 1; lyr <= layru-1; lyr++) {
    wbar += DTAUC(lyr)*SSALB(lyr);
    fbar += DTAUC(lyr)*SSALB(lyr)*FLYR(lyr);
    stau += DTAUC(lyr);
  }

  if (wbar <= tiny || fbar <= tiny || stau <= tiny || ds->bc.fbeam <= tiny) {
    return 0.;
  }

  fbar /= wbar;
  wbar /= stau;

  /* Calculate pspike1=P" */

  pspike1 = PHAS2(it,lu) + ( ctheta - ds->MUP(it) ) /
    ( ds->MUP(it+1) - ds->MUP(it) ) * ( PHAS2(it+1,lu) - PHAS2(it,lu) );

  pspike2 = calc_phase_squared (ds->nphase, lu, ctheta, nf,
				ds->mu_phase, phas2, mu_eq, neg_phas,
				norm_phas);

  pspike = 2.*pspike1 - pspike2;

  umu0p = ds->bc.umu0/(1.-fbar*wbar);

  /*
   * Calculate IMS correction term, eq. STWL (A.13)
   */
  return ds->bc.fbeam/(4.*M_PI)*SQR(fbar*wbar)/(1.-fbar*wbar)*pspike*c_xi_func(-UMU(iu),umu0p,UTAU(lu));
}

/*============================= end of c_new_secondary_scat() ===========*/
