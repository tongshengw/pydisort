#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_albtrans_spherical() ===================*/

/*
    Calculates spherical albedo and transmissivity for the entire medium
    from the m=0 intensity components (this is a specialized version of fluxes)

    I N P U T    V A R I A B L E S:

       ds      :  Disort state variables
       cmu,cwt :  Abscissae, weights for Gaussian quadrature over angle cosine
       kk      :  Eigenvalues of coeff. matrix in eq. SS(7)
       gc      :  Eigenvectors at polar quadrature angles, SC(1)
       ll      :  Constants of integration in eq. SC(1), obtained by solving
                  scaled version of eq. SC(5); exponential term of eq. SC(12) not incl.
       nn      :  Order of double-Gauss quadrature (NSTR/2)

    O U T P U T   V A R I A B L E S:

       sflup   :  Up-flux at top (equivalent to spherical albedo due to
                  reciprocity).  For illumination from below it gives
                  spherical transmissivity

       sfldn   :  Down-flux at bottom (for single layer, equivalent to
                  spherical transmissivity due to reciprocity)

    I N T E R N A L   V A R I A B L E S:

       zint    :  Intensity of m=0 case, in eq. SC(1)

   Called by- c_albtrans
 --------------------------------------------------------------------*/

void c_albtrans_spherical(disort_state *ds,
                          double       *cmu,
                          double       *cwt,
                          double       *gc,
                          double       *kk,
                          double       *ll,
                          int           nn,
                          double       *taucpr,
                          double       *sflup,
                          double       *sfldn)
{
  register int
    iq,jq;
  double
    zint;

  *sflup = 0.;
  for (iq = nn+1; iq <= ds->nstr; iq++) {
    zint = 0.;
    for (jq = 1; jq <= nn; jq++) {
      zint += GC(iq,jq,1)*LL(jq,1)*exp(KK(jq,1)*TAUCPR(1));
    }
    for (jq = nn+1; jq <= ds->nstr; jq++) {
      zint += GC(iq,jq,1)*LL(jq,1);
    }
    *sflup += CWT(iq-nn)*CMU(iq-nn)*zint;
  }

  *sfldn = 0.;
  for (iq = 1; iq <= nn; iq++) {
    zint = 0.;
    for (jq = 1; jq <= nn; jq++) {
      zint += GC(iq,jq,ds->nlyr)*LL(jq,ds->nlyr);
    }
    for (jq = nn+1; jq <=ds->nstr; jq++) {
      zint += GC(iq,jq,ds->nlyr)*LL(jq,ds->nlyr)*exp(-KK(jq,ds->nlyr)*(TAUCPR(ds->nlyr)-TAUCPR(ds->nlyr-1)));
    }
    *sfldn += CWT(nn+1-iq)*CMU(nn+1-iq)*zint;
  }

  *sflup *= 2.;
  *sfldn *= 2.;

  return;
}

/*============================= end of c_albtrans_spherical() ============*/
