#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_albtrans_intensity() ===================*/

/*
   Computes azimuthally-averaged intensity at top and bottom of medium
   (related to albedo and transmission of medium by reciprocity principles;
   see Ref S2).  User polar angles are used as incident beam angles.
   (This is a very specializedversion of user_intensities)

   ** NOTE **  User input values of UMU (assumed positive) are temporarily in
               upper locations of  UMU  and corresponding negatives are in
               lower locations (this makes GU come out right); the contents
               of the temporary UMU array are:
                   -UMU(ds->numu),..., -UMU(1), UMU(1),..., UMU(ds->numu)

   I N P U T    V A R I A B L E S:

       ds     :  Disort state variables
       gu     :  Eigenvectors interpolated to user polar angles (i.e., g in eq. SC(1), STWL(31ab))
       kk     :  Eigenvalues of coeff. matrix in eq. SS(7), STWL(23b)
       ll     :  Constants of integration in eq. SC(1), obtained by solving scaled version of eq. SC(5);
                 exponential term of eq. SC(12) not included
       nn     :  Order of double-Gauss quadrature (NSTR/2)
       taucpr :  Cumulative optical depth (delta-M-scaled)

   O U T P U T    V A R I A B L E:

       out->u0u : Diffuse azimuthally-averaged intensity at top and bottom of medium (directly transmitted component,
                  corresponding to bndint in user_intensities, is omitted).

   I N T E R N A L    V A R I A B L E S:

       dtau   :  Optical depth of a computational layer
       palint :  Non-boundary-forced intensity component
       utaupr :  Optical depths of user output levels (delta-M scaled)
       wk     :  Scratch vector for saving 'EXP' evaluations
       All the exponential factors (i.e., exp1, expn,... etc.)
       come from the substitution of constants of integration in
       eq. SC(12) into eqs. S1(8-9).  All have negative arguments.

   Called by- c_albtrans
 -------------------------------------------------------------------*/

void c_albtrans_intensity(disort_state *ds,
			  disort_output *out,
                          double       *gu,
                          double       *kk,
                          double       *ll,
                          int           nn,
                          double       *taucpr,
                          double       *wk)
{
  register int
    iq,iu,iumax,iumin,lc,lu;
  double
    denom,dtau,exp1,exp2,expn,mu,palint,sgn,utaupr[2];

  UTAUPR(1) = 0.;
  UTAUPR(2) = TAUCPR(ds->nlyr);

  for (lu = 1; lu <= 2; lu++) {
    if (lu == 1) {
      iumin = ds->numu/2+1;
      iumax = ds->numu;
      sgn   = 1.;
    }
    else {
      iumin = 1;
      iumax = ds->numu/2;
      sgn   = -1.;
    }

    /*
     * Loop over polar angles at which albedos/transmissivities desired
     * ( upward angles at top boundary, downward angles at bottom )
     */
    for (iu = iumin; iu <= iumax; iu++) {
      mu = UMU(iu);
      /*
       * Integrate from top to bottom computational layer
       */
      palint = 0.;
      for (lc = 1; lc <= ds->nlyr; lc++) {
        dtau = TAUCPR(lc)-TAUCPR(lc-1);
        exp1 = exp((UTAUPR(lu)-TAUCPR(lc-1))/mu);
        exp2 = exp((UTAUPR(lu)-TAUCPR(lc  ))/mu);
        /*
         * KK is negative
         */
        for (iq = 1; iq <= nn; iq++) {
          WK(iq) = exp(KK(iq,lc)*dtau);
          denom  = 1.+mu*KK(iq,lc);
          if (fabs(denom) < 0.0001) {
            /*
             * L'Hospital limit
             */
            expn = dtau/mu*exp2;
          }
          else {
            expn = (exp1*WK(iq)-exp2)*sgn/denom;
          }
          palint += GU(iu,iq,lc)*LL(iq,lc)*expn;
        }
        /*
         * KK is positive
         */
        for (iq = nn+1; iq <= ds->nstr; iq++) {
          denom = 1.+mu*KK(iq,lc);
          if (fabs(denom) < 0.0001) {
            expn = -dtau/mu*exp1;
          }
          else {
            expn = (exp1-exp2*WK(ds->nstr+1-iq))*sgn/denom;
          }
          palint += GU(iu,iq,lc)*LL(iq,lc)*expn;
        }
      }
      U0U(iu,lu) = palint;
    }
  }

  return;
}

/*============================= end of c_albtrans_intensity() ============*/
