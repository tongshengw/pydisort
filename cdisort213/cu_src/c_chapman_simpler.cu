#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
#undef  KK
#define KK(lyu) kk[lyu-1]
/*============================= c_chapman_simpler() ======================*/

double c_chapman_simpler(int     lc,
                 double  taup,
                 int     nlyr,
                 double *zd,
                 double *dtau_c,
                 double  zenang,
                 double  r)
{
  register int
    id,j;
  double
    zenrad,xp,xpsinz,
    sum,fact,fact2,rj,rjp1,dhj,dsj;

  zenrad = zenang*DEG;
  xp     = r+ZD(lc)+(ZD(lc-1)-ZD(lc))*taup;
  xpsinz = xp*sin(zenrad);

  if (zenang > 90. && xpsinz < r) {
    return 1.e+20;
  }

  /*
   * Find index of layer in which the screening height lies
   */
  id = lc;
  if (zenang > 90.) {
    for (j= lc; j <= nlyr; j++) {
      if (xpsinz < (ZD(j-1)+r) && (xpsinz >= ZD(j)+r)) {
        id = j;
      }
    }
  }

  sum = 0.;
  for (j = 1; j <= id; j++) {
    fact  = 1.;
    fact2 = 1.;
    /*
     * Include factor of 2 for zenang > 90., second sum in eq. B2 (DS)
     */
    if (j > lc) {
      fact = 2.;
    }
    else if (j == lc && lc == id && zenang > 90.) {
      fact2 = -1.;
    }

    rj   = r+ZD(j-1);
    rjp1 = r+ZD(j  );
    if (j == lc && id == lc) {
      rjp1 = xp;
    }

    dhj = ZD(j-1)-ZD(j);
    if (id > lc && j == id) {
      dsj = sqrt(rj*rj-xpsinz*xpsinz);
    }
    else {
      dsj = sqrt(rj*rj-xpsinz*xpsinz)-fact2*sqrt(rjp1*rjp1-xpsinz*xpsinz);
    }
    sum += DTAU_C(j)*fact*dsj/dhj;
  }
  /*
   * Add third term in eq. B2 (DS)
   */
  if (id > lc) {
    dhj  = ZD(lc-1)-ZD(lc);
    dsj  = sqrt(xp*xp-xpsinz*xpsinz)-sqrt(SQR(ZD(lc)+r)-xpsinz*xpsinz);
    sum += DTAU_C(lc)*dsj/dhj;
  }

  return sum;
}

/*============================= end of c_chapman_simpler() ===============*/
