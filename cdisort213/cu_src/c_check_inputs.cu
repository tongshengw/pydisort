#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_check_inputs() ========================*/

/*
 * Checks the input dimensions and variables
 *
 * Calls- c_write_bad_var, c_dref, c_errmsg
 * Called by- c_disort
 */

int c_check_inputs(disort_state *ds,
		    int           scat_yes,
		    int           deltam,
		    int           corint,
		    double       *tauc,
		    int           callnum)
{
  int
    inperr = FALSE;
  register int
    irmu,iu,j,k,lc,lu, nu;
  double
    flxalb,rmu,umumin;

  if (ds->nstr < 2 || ds->nstr%2 != 0) {
    inperr = c_write_bad_var(VERBOSE,"ds.nstr");
  }
  if (ds->nstr == 2) {
    c_errmsg("check_inputs()--2 streams not recommended;\n\nUse specialized 2-stream code c_twostr() instead",DS_WARNING);
  }
  if (ds->nlyr < 1) {
    inperr = c_write_bad_var(VERBOSE,"ds.nlyr");
  }

  for (lc = 1; lc <= ds->nlyr; lc++) {
    if (DTAUC(lc) < 0.) {
      inperr = c_write_bad_var(VERBOSE,"ds.dtauc");
    }
    if (SSALB(lc) < 0.0 || SSALB(lc) > 1.0) {
      inperr = c_write_bad_var(VERBOSE,"ds.ssalb");
    }
    if (ds->flag.ibcnd == GENERAL_BC) {
      if (ds->flag.planck) {
        if (lc == 1 && TEMPER(0) < 0.) {
          inperr = c_write_bad_var(VERBOSE,"ds.temper");
        }
        if (TEMPER(lc) < 0.) {
          inperr = c_write_bad_var(VERBOSE,"ds.temper");
        }
      }
    }
    else if (ds->flag.ibcnd == SPECIAL_BC) {
      ds->flag.planck = FALSE;
    }
    else {
      c_errmsg("check_inputs---unrecognized ds->flag.ibcnd",DS_ERROR);
    }
  }

  if (ds->nmom < 0 || (scat_yes  && ds->nmom < ds->nstr)) {
    inperr = c_write_bad_var(VERBOSE,"ds.nmom");
  }

  for (lc = 1; lc <= ds->nlyr; lc++) {
    for (k = 0; k <= ds->nmom; k++) {
      if (PMOM(k,lc) < -1. || PMOM(k,lc) > 1.) {
        inperr = c_write_bad_var(VERBOSE,"PMOM(k,lc)");
      }
    }
  }

  if( ds->flag.spher == TRUE ) {
    for (lc = 1; lc <= ds->nlyr; lc++) {
      if (ds->ZD(lc) > ds->ZD(lc-1)) {
        inperr     = c_write_bad_var(ds->flag.quiet,"zd");
      }
    }
  }

  if (ds->flag.ibcnd == GENERAL_BC) {
    if (ds->flag.usrtau) {
      if (ds->ntau < 1) {
        inperr = c_write_bad_var(VERBOSE,"ds.ntau");
      }
      for (lu = 1; lu <= ds->ntau; lu++) {
	/* Do a relative check to see if we are just beyond the bottom boundary */
	/* This might happen due to numerical rounding off problems.  ak20110224*/
        if (fabs(UTAU(lu)-TAUC(ds->nlyr)) <= 1.e-6*TAUC(ds->nlyr)) {
          UTAU(lu) = TAUC(ds->nlyr);
        }
        if(UTAU(lu) < 0. || UTAU(lu) > TAUC(ds->nlyr)) {
          inperr = c_write_bad_var(VERBOSE,"ds.utau");
        }
      }
    }
  }

  if (ds->flag.usrang) {
    if (ds->numu < 0) {
      inperr = c_write_bad_var(VERBOSE,"ds.numu");
    }
    if (!ds->flag.onlyfl && ds->numu == 0) {
      inperr = c_write_bad_var(VERBOSE,"ds.numu");
    }
    nu = ds->numu;
    if (ds->flag.ibcnd == SPECIAL_BC ) nu = ds->numu/2;
    for (iu = 1; iu <= nu; iu++) {
      if (UMU(iu) < -1. || UMU(iu) > 1. || UMU(iu) == 0.) {
        inperr = c_write_bad_var(VERBOSE,"ds.umu");
      }
      if (ds->flag.ibcnd == SPECIAL_BC && UMU(iu) < 0.) {
        inperr = c_write_bad_var(VERBOSE,"ds.umu");
      }
      if (iu > 1) {
        if (UMU(iu) < UMU(iu-1)) {
          inperr = c_write_bad_var(VERBOSE,"ds.umu");
        }
      }
    }
  }

  if (!ds->flag.onlyfl && ds->flag.ibcnd != SPECIAL_BC) {
    if (ds->nphi <= 0) {
      inperr = c_write_bad_var(VERBOSE,"ds.nphi");
    }
    for (j=1; j <=ds->nphi; j++) {
      if (PHI(j) < 0. || PHI(j) > 360.) {
        inperr = c_write_bad_var(VERBOSE,"ds.phi");
      }
    }
  }

  if (ds->flag.ibcnd != GENERAL_BC && ds->flag.ibcnd != SPECIAL_BC) {
    inperr = c_write_bad_var(VERBOSE,"ds.flag.ibcnd");
  }

  if (ds->flag.ibcnd == GENERAL_BC) {
    if (ds->bc.fbeam < 0.) {
      inperr = c_write_bad_var(VERBOSE,"ds.bc.fbeam");
    }
    else if (ds->bc.fbeam > 0.) {
      umumin = 0.;
      if( ds->flag.spher == TRUE ) {
	umumin = -1.;
      }
      if (ds->bc.umu0 <= umumin || ds->bc.umu0 > 1.) {
        inperr = c_write_bad_var(VERBOSE,"ds.bc.umu0");
      }
      if (ds->bc.phi0 < 0. || ds->bc.phi0 > 360.) {
        inperr = c_write_bad_var(VERBOSE,"ds.bc.phi0");
      }
    }

    if (ds->bc.fisot < 0.) {
      inperr = c_write_bad_var(VERBOSE,"ds.bc.fisot");
    }

    if (ds->flag.lamber) {
      if (ds->bc.albedo < 0. || ds->bc.albedo > 1.) {
        inperr = c_write_bad_var(VERBOSE,"ds.bc.albedo");
      }
    }
    else {
      /*
       * Make sure flux albedo at dense mesh of incident angles does not assume unphysical values
       */
      for (irmu = 0; irmu <= 100; irmu++) {
        rmu    = (double)irmu*0.01;
        flxalb = c_dref(ds->wvnmlo, ds->wvnmhi, rmu, ds->flag.brdf_type, &ds->brdf, callnum);
        if (flxalb < 0. || flxalb > 1.) {
          inperr = c_write_bad_var(VERBOSE,"bidir_reflectivity()");
        }
      }
    }
  }
  else {
    if (ds->bc.albedo < 0. || ds->bc.albedo > 1.) {
      inperr = c_write_bad_var(VERBOSE,"ds.bc.albedo");
    }
  }

  if (ds->flag.planck && ds->flag.ibcnd != SPECIAL_BC) {
    if (ds->wvnmlo < 0. || ds->wvnmhi < ds->wvnmlo) {
      inperr = c_write_bad_var(VERBOSE,"ds.wvnmlo,hi");
    }
    if (ds->bc.temis < 0. || ds->bc.temis > 1.) {
      inperr = c_write_bad_var(VERBOSE,"ds.bc.temis");
    }
    if (ds->bc.btemp < 0.) {
      inperr = c_write_bad_var(VERBOSE,"ds.bc.btemp");
    }
    if (ds->bc.ttemp < 0.) {
      inperr = c_write_bad_var(VERBOSE,"ds.bc.ttemp");
    }
  }

  if (ds->accur < 0. || ds->accur > 1.e-2) {
    inperr = c_write_bad_var(VERBOSE,"ds.accur");
  }

  if (inperr) {
    c_errmsg("DISORT--input and/or dimension errors",DS_WARNING);
    return 1;
  }

  if (ds->flag.planck && ds->flag.quiet == VERBOSE) {
    for (lc = 1; lc <= ds->nlyr; lc++) {
      if (fabs(TEMPER(lc)-TEMPER(lc-1)) > 10.) {
        c_errmsg("check_inputs--vertical temperature step may be too large for good accuracy",DS_WARNING);
      }
    }
  }
  if(!corint && (!ds->flag.onlyfl && ds->bc.fbeam > 0. && scat_yes && deltam)) {
    c_errmsg("check_inputs--intensity correction is off;\nintensities may be less accurate",DS_WARNING);
  }

  return 0;
}

/*============================= end of c_check_inputs() =================*/
