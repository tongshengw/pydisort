#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
#undef  KK
#define KK(lyu) kk[lyu-1]
/*============================= c_twostr_solns() =========================*/

/*
    Calculates the homogenous and particular solutions to the
    radiative transfer equation in the two-stream approximation,
    for each layer in the medium.

    I n p u t     v a r i a b l e s:

      ds         : 'Disort' state variables
      ch         : Chapman correction factor
      chtau      :
      cmu        : Abscissa for gauss quadrature over angle cosine
      ncut       : Number of computational layer where absorption optical depth exceeds -abscut-
      oprim      : Delta-m scaled single scattering albedo
      pkag,c     : Planck function in each layer
      flag.spher : spher = true => spherical geometry invoked
      taucpr     : Cumulative optical depth (delta-m-scaled)
      ggprim     :

   O u t p u t     v a r i a b l e s:

      kk         :  Eigenvalues
      rr         :  Eigenvectors at polar quadrature angles
      ts         :  twostr_xyz structure variables (see cdisort.h)
  ----------------------------------------------------------------------*/

void c_twostr_solns(disort_state *ds,
                    double       *ch,
                    double       *chtau,
                    double        cmu,
                    int           ncut,
                    double       *oprim,
                    double       *pkag,
                    double       *pkagc,
                    double       *taucpr,
                    double       *ggprim,
                    double       *kk,
                    double       *rr,
                    twostr_xyz   *ts)
{
  register int
    lc;
  static int
    initialized = FALSE;
  static double
    big,large,small,little;
  double
    q_1,q_2,qq,q0a,q0,q1a,q2a,q1,q2,
    deltat,denomb,z0p,z0m,arg,sgn,fact3,denomp,
    beta,fact1,fact2;

  if (!initialized) {
    /*
     * The calculation of the particular solutions require some care; small,little,
       big, and large have been set so that no problems should occur in double precision.
     */
    small  = 1.e+30*DBL_MIN;
    little = 1.e+20*DBL_MIN;
    big    = sqrt(DBL_MAX)/1.e+10;
    large  = log(DBL_MAX)-20.;

    initialized = TRUE;
  }

  /*----------------  Begin loop on computational layers  ---------------------*/

  for (lc = 1; lc <= ncut; lc++) {
    /*
     * Calculate eigenvalues -kk- and eigenvector -rr-, eqs. KST(20-21)
     */
    beta   = 0.5*(1.-3.*GGPRIM(lc)*cmu*cmu);
    fact1  = 1.-OPRIM(lc);
    fact2  = 1.-OPRIM(lc)+2.*OPRIM(lc)*beta;
    KK(lc) = (1./cmu)*sqrt(fact1*fact2);
    RR(lc) = (sqrt(fact2)-sqrt(fact1))/(sqrt(fact2)+sqrt(fact1));

    if (ds->bc.fbeam > 0.) {
      /*
       * Set coefficients in KST(22) for beam source
       */
      q_1 = ds->bc.fbeam/(4.*M_PI)*OPRIM(lc)*(1.-3.*GGPRIM(lc)*cmu*ds->bc.umu0);
      q_2 = ds->bc.fbeam/(4.*M_PI)*OPRIM(lc)*(1.+3.*GGPRIM(lc)*cmu*ds->bc.umu0);

      if (ds->bc.umu0 >= 0.) {
        qq = q_2;
      }
      else {
        qq = q_1;
      }

      if (ds->flag.spher) {
        q0a = exp(-CHTAU(lc-1));
        q0  = q0a*qq;
        if (q0 <= small) {
          q1a = 0.;
          q2a = 0.;
        }
        else {
          q1a = exp(-CHTAU(lc-1  ));
          q2a = exp(-CHTAU(lc));
        }
      }
      else {
        q0a = exp(-TAUCPR(lc-1)/ds->bc.umu0);
        q0  = q0a*qq;
        if (q0 <= small) {
          q1a = 0.;
          q2a = 0.;
        }
        else {
          q1a = exp(-(TAUCPR(lc-1)+TAUCPR(lc))/(2.*ds->bc.umu0));
          q2a = exp(-TAUCPR(lc)/ds->bc.umu0);
        }
      }
      q1 = q1a*qq;
      q2 = q2a*qq;

      /*
       * Calculate alpha coefficient
       */
      deltat     = TAUCPR(lc)-TAUCPR(lc-1);
      ZB_A(lc)   = 1./CH(lc);
      if (fabs(ZB_A(lc)*TAUCPR(lc-1)) > large || fabs(ZB_A(lc)*TAUCPR(lc)) > large) {
        ZB_A(lc) = 0.;
      }

      /*
       * Dither alpha if it is close to an eigenvalue
       */
      denomb = fact1*fact2-SQR(ZB_A(lc)*cmu);
      if (denomb < 1.e-03) {
        ZB_A(lc) = 1.02*ZB_A(lc);
      }
      q0 = q0a*q_1;
      q2 = q2a*q_1;

      /*
       * Set constants in eq. KST(22)
       */
      if (deltat < 1.e-07) {
        XB_1D(lc) = 0.;
      }
      else {
        XB_1D(lc) = 1./deltat*(q2*exp(ZB_A(lc)*TAUCPR(lc))-q0*exp(ZB_A(lc)*TAUCPR(lc-1)));
      }
      XB_0D(lc) = q0*exp(ZB_A(lc)*TAUCPR(lc-1))-XB_1D(lc)*TAUCPR(lc-1);
      q0        = q0a*q_2;
      q2        = q2a*q_2;

      if (deltat < 1.e-07) {
        XB_1U(lc) = 0.;
      }
      else {
        XB_1U(lc) = 1./deltat*(q2*exp(ZB_A(lc)*TAUCPR(lc))-q0*exp(ZB_A(lc)*TAUCPR(lc-1)));
      }
      XB_0U(lc) = q0*exp(ZB_A(lc)*TAUCPR(lc-1))-XB_1U(lc)*TAUCPR(lc-1);

      /*
       * Calculate particular solutions for incident beam source in pseudo-spherical geometry, eqs. KST(24-25)
       */
      denomb    = fact1*fact2-SQR(ZB_A(lc)*cmu);
      YB_1D(lc) = (OPRIM(lc)*beta*XB_1D(lc)+(1.-OPRIM(lc)*(1.-beta)+ZB_A(lc)*cmu)*XB_1U(lc))/denomb;
      YB_1U(lc) = (OPRIM(lc)*beta*XB_1U(lc)+(1.-OPRIM(lc)*(1.-beta)-ZB_A(lc)*cmu)*XB_1D(lc))/denomb;
      z0p       = XB_0U(lc)-cmu*YB_1D(lc);
      z0m       = XB_0D(lc)+cmu*YB_1U(lc);
      YB_0D(lc) = (OPRIM(lc)*beta*z0m+(1.-OPRIM(lc)*(1.-beta)+ZB_A(lc)*cmu)*z0p)/denomb;
      YB_0U(lc) = (OPRIM(lc)*beta*z0p+(1.-OPRIM(lc)*(1.-beta)-ZB_A(lc)*cmu)*z0m)/denomb;
    }

    if(ds->flag.planck) {
      /*
       * Set coefficients in KST(22) for thermal source
       * Calculate alpha coefficient
       */
      q0     = (1.-OPRIM(lc))*PKAG(lc-1);
      q1     = (1.-OPRIM(lc))*PKAGC(lc);
      q2     = (1.-OPRIM(lc))*PKAG(lc);
      deltat = TAUCPR(lc)-TAUCPR(lc-1);

      if ((q2 < q0*1.e-02 || q2 <= little) && q1 > little && q0 > little) {
        /*
         * Case 1: source small at bottom layer; alpha eq. KS(50)
         */
        ZP_A(lc) = MIN(2./deltat*log(q0/q1),big);
        if (ZP_A(lc)*TAUCPR(lc-1) >= log(big)) {
          XP_0(lc) = big;
        }
        else {
          XP_0(lc) = q0;
        }
        XP_1(lc) = 0.;
      }
      else if ((q2 <= q1*1.e-02 || q2 <= little) && (q1 <= q0*1.e-02 || q1 <= little) && q0 > little) {
        /*
         * Case 2: Source small at center and bottom of layer
         */
        ZP_A(lc) = big/TAUCPR(ncut);
        XP_0(lc) = q0;
        XP_1(lc) = 0.;
      }
      else if (q2 <= little && q1 <= little && q0 <= little) {
        /*
         * Case 3: All sources zero
         */
        ZP_A(lc) = 0.;
        XP_0(lc) = 0.;
        XP_1(lc) = 0.;
      }
      else if ( ( fabs((q2-q0)/q2) < 1.e-04 && fabs((q2-q1)/q2) < 1.e-04 ) || deltat < 1.e-04) {
        /*
         * Case 4: Sources same at center, bottom and top of layer or layer optically very thin
         */
        ZP_A(lc) = 0.;
        XP_0(lc) = q0;
        XP_1(lc) = 0.;
      }
      else {
        /*
         *  Case 5: Normal case
         */
        arg = MAX(SQR(q1/q2)-q0/q2,0.);
        /*
         * alpha eq. (44). For source that has its maximum at the top of the layer, use negative solution
         */
        sgn = 1.;
        if (PKAG(lc-1) > PKAG(lc)) {
         sgn = -1.;
        }
        fact3 = log(q1/q2+sgn*sqrt(arg));

        /* Be careful with log of numbers close to one */
        if (fabs(fact3) <= 0.005) {
          /* numbers close to one */
          q1    = 0.99*q1;
          fact3 = log(q1/q2+sgn*sqrt(arg));
        }

        ZP_A(lc) = 2./deltat*fact3;
        if (fabs(ZP_A(lc)*TAUCPR(lc)) > log(DBL_MAX)-log(q0*100.)) {
          ZP_A(lc) = 0.;
        }

        /*
         * Dither alpha if it is close to an eigenvalue
         */
        denomp = fact1*fact2-SQR(ZP_A(lc)*cmu);
        if (denomp < 1.e-03) {
          ZP_A(lc) *= 1.01;
        }

        /*
         * Set constants in eqs. KST(22)
         */
        if(deltat < 1.e-07) {
          XP_1(lc) = 0.;
        }
        else {
          XP_1(lc) = 1./deltat*(q2*exp(ZP_A(lc)*TAUCPR(lc))-q0*exp(ZP_A(lc)*TAUCPR(lc-1)));
        }
        XP_0(lc) = q0*exp(ZP_A(lc)*TAUCPR(lc-1))-XP_1(lc)*TAUCPR(lc-1);
      }

      /*
       * Calculate particular solutions eqs. KST(24-25) for internal thermal so
       */
      denomp    = fact1*fact2-SQR(ZP_A(lc)*cmu);
      YP_1D(lc) = (OPRIM(lc)*beta*XP_1(lc)+(1.-OPRIM(lc)*(1.-beta)+ZP_A(lc)*cmu)*XP_1(lc))/denomp;
      YP_1U(lc) = (OPRIM(lc)*beta*XP_1(lc)+(1.-OPRIM(lc)*(1.-beta)-ZP_A(lc)*cmu)*XP_1(lc))/denomp;
      z0p       = XP_0(lc)-cmu*YP_1D(lc);
      z0m       = XP_0(lc)+cmu*YP_1U(lc);
      YP_0D(lc) = (OPRIM(lc)*beta*z0m+(1.-OPRIM(lc)*(1.-beta)+ZP_A(lc)*cmu)*z0p)/denomp;
      YP_0U(lc) = (OPRIM(lc)*beta*z0p+(1.-OPRIM(lc)*(1.-beta)-ZP_A(lc)*cmu)*z0m)/denomp;
    }
  }

  return;
}

/*============================= end of c_twostr_solns() ==================*/
