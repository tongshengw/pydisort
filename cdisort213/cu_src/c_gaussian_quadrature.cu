#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_gaussian_quadrature() =================*/

/*
   Compute weights and abscissae for ordinary Gaussian quadrature
   on the interval (0,1);  that is, such that
       sum(i=1 to M) ( GWT(i) f(GMU(i)) )
   is a good approximation to integral(0 to 1) ( f(x) dx )

   INPUT :     m        order of quadrature rule

   OUTPUT :    GMU(I)   array of abscissae (I = 1 TO M)
               GWT(I)   array of weights (I = 1 TO M)

   REFERENCE:  Davis, P.J. and P. Rabinowitz, Methods of Numerical
                 Integration, Academic Press, New York, pp. 87, 1975

   METHOD:     Compute the abscissae as roots of the Legendre polynomial P-sub-M using a cubically convergent
               refinement of Newton's method.  Compute the weights from eq. 2.7.3.8 of Davis/Rabinowitz.  Note
               that Newton's method can very easily diverge; only a very good initial guess can guarantee convergence.
               The initial guess used here has never led to divergence even for M up to 1000.

   ACCURACY:   Relative error no better than TOL or computer precision (DBL_EPSILON), whichever is larger

   INTERNAL VARIABLES:
    iter      : Number of Newton Method iterations
    pm2,pm1,p : 3 successive Legendre polynomials
    ppr       : Derivative of Legendre polynomial
    p2pri     : 2nd derivative of Legendre polynomial
    tol       : Convergence criterion for Legendre poly root iteration
    x,xi      : Successive iterates in cubically-convergent version of Newtons Method (seeking roots of Legendre poly.)

   Called by- c_dref, c_disort_set, c_surface_bidir
   Calls- c_errmsg
 -------------------------------------------------------------------*/

/* Maximum allowed iterations of Newton Method */
#define MAXIT 1000

void c_gaussian_quadrature(int    m,
                           double *gmu,
                           double *gwt)
{
  static int
    initialized = FALSE;
  register int
    iter,k,lim,nn,np1;
  double
    cona,t,en,nnp1,p=0,p2pri,pm1,pm2,ppr,
    prod,tmp,x,xi;
  static double
    tol;

  if (!initialized) {
    tol         = 10.*DBL_EPSILON;
    initialized = TRUE;
  }

  if (m < 1) {
    c_errmsg("gaussian_quadrature--Bad value of m",DS_ERROR);
  }

  if (m == 1) {
    GMU(1) = 0.5;
    GWT(1) = 1.0;
    return;
  }

  en   = (double)m;
  np1  = m+1;
  nnp1 = m*np1;
  cona = (double)(m-1)/(8*m*m*m);
  lim  = m/2;
  for (k = 1; k <= lim; k++) {
    /*
     * Initial guess for k-th root of Legendre polynomial, from Davis/Rabinowitz (2.7.3.3a)
     */
    t = (double)(4*k-1)*M_PI/(4*m+2);
    x = cos(t+cona/tan(t));

    /*
     * Upward recurrence for Legendre polynomials
     */
    for (iter = 1; iter <= MAXIT+1; iter++) {
      if (iter > MAXIT) {
        c_errmsg("gaussian_quadrature--max iteration count",DS_ERROR);
      }
      pm2 = 1.;
      pm1 = x;
      for (nn = 2; nn <= m; nn++) {
        p   = ((double)(2*nn-1)*x*pm1-(double)(nn-1)*pm2)/nn;
        pm2 = pm1;
        pm1 = p;
      }
      /*
       * Newton Method
       */
      tmp   = 1./(1.-x*x);
      ppr   = en*(pm2-x*p)*tmp;
      p2pri = (2.*x*ppr-nnp1*p)*tmp;
      xi    = x-p/ppr*(1.+p/ppr*p2pri/(2.*ppr));
      /*
       * Check for convergence
       */
      if (fabs(xi-x) <= tol) {
        break;
      }
      else {
        x = xi;
      }
    }

    /*
     * Iteration finished--calculate weights, abscissae for (-1,1)
     */
    GMU(k)     = -x;
    GWT(k)     = 2./(tmp*SQR(en*pm2));
    GMU(np1-k) = -GMU(k);
    GWT(np1-k) =  GWT(k);
  }

  /*
   * Set middle abscissa and weight for rules of odd order
   */
  if (m%2 != 0) {
    GMU(lim+1) = 0.;
    prod       = 1.;
    for (k = 3; k <= m; k+=2) {
      prod *= (double)k/(k-1);
    }
    GWT(lim+1) = 2./SQR(prod);
  }
  /*
   * Convert from (-1,1) to (0,1)
   */
  for (k = 1; k <= m; k++) {
    GMU(k) = 0.5*GMU(k)+0.5;
    GWT(k) = 0.5*GWT(k);
  }

  return;
}

#undef MAXIT

/*============================= end of c_gaussian_quadrature() ==========*/
