#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
#undef  KK
#define KK(lyu) kk[lyu-1]
/*============================= c_twostr_solve_bc() ======================*/

/*
 Construct right-hand side vector -b- for general boundary conditions
 and solve system of equations obtained from the boundary conditions
 and the continuity-of-intensity-at-layer-interface equations.

 Routines called: c_sgbfa, c_sgbsl

 I n p u t      v a r i a b l e s:

       ds       : 'Disort' state variables
       ts       :  twostr_xyz structure variables (see cdisort.h)
       bplanck  :  Bottom boundary thermal emission
       cband    :  Left-hand side matrix of linear system eqs. KST(38-41)
                   in banded form required by linpack solution routines
       cmu      :  Abscissa for gauss quadrature over angle cosine
       expbea   :  Transmission of incident beam, EXP(-taucpr/ch)
       lyrcut   :  Logical flag for truncation of comput. layer
       ncol     :  Counts of columns in -cband-
       nn       :  Order of double-gauss quadrature (nstr/2)
       ncut     :  Total number of computational layers considered
       tplanck  :  Top boundary thermal emission
       taucpr   :  Cumulative optical depth (delta-m-scaled)
       kk       :
       rr       :
       ipvt     :

 O u t p u t     v a r i a b l e s:

       b        :  Right-hand side vector of eqs. KST(38-41) going into
                   sgbsl; returns as solution vector of eqs. KST(38-41)
                   constants of integration
       ll       :  Permanent storage for -b-, but re-ordered

 I n t e r n a l    v a r i a b l e s:

       diag     : diag[].super, diag[].on, diag[].sub

 ---------------------------------------------------------------------*/

void c_twostr_solve_bc(disort_state *ds,
                       twostr_xyz   *ts,
                       double        bplanck,
                       double       *cband,
                       double        cmu,
                       double       *expbea,
                       int           lyrcut,
                       int           nn,
                       int           ncut,
                       double        tplanck,
                       double       *taucpr,
                       double       *kk,
                       double       *rr,
                       int          *ipvt,
                       double       *b,
                       double       *ll,
                       twostr_diag  *diag)
{
  int
    info;
  register int
    irow,lc,nloop,nrow,job;
  double
    wk0,wk1,wk,rpp1_m,rp_m,rpp1_p,rp_p,sum,refflx;
  register double
    fact1,fact2,fact3,fact4;

  /*
   * First top row, top boundary condition
   */
  irow = 1;
  lc   = 1;
  /*
   * SUBD(irow) is undefined
   */
  DIAG(irow)   = RR(lc)*exp(-KK(lc)*TAUCPR(lc));
  SUPERD(irow) = 1.;
  /*
   * next from layer no. 2 to nlyr-1
   */
  nloop = ncut-1;
  for (lc = 1; lc <= nloop; lc++) {
    irow++;
    wk0          = exp(-KK(lc  )*(TAUCPR(lc  )-TAUCPR(lc-1)));
    wk1          = exp(-KK(lc+1)*(TAUCPR(lc+1)-TAUCPR(lc  )));
    SUBD(irow)   = 1.-RR(lc)*RR(lc+1);
    DIAG(irow)   = (RR(lc)-RR(lc+1))*wk0;
    SUPERD(irow) = -(1.-SQR(RR(lc+1)))*wk1;
    irow++;
    SUBD(irow)   = (1.-SQR(RR(lc)))*wk0;
    DIAG(irow)   = (RR(lc)-RR(lc+1))*wk1;
    SUPERD(irow) = -(1.-RR(lc+1)*RR(lc));
  }
  /*
   * bottom layer
   */
  irow++;
  lc = ncut;
  /*
   * SUPERD(irow) = undefined
   */
  wk = exp(-KK(lc)*(TAUCPR(lc)-TAUCPR(lc-1)));
  if (lyrcut) {
    SUBD(irow) = 1.;
    DIAG(irow) = RR(lc)*wk;
  }
  else {
    SUBD(irow) = 1.-2.*ds->bc.albedo*cmu*RR(lc);
    DIAG(irow) = (RR(lc)-2.*ds->bc.albedo*cmu)*wk;
  }

  /*
   * NOTE: If not allocating memory with swappablecalloc(), need to zero out b here.
   */

  /*
   * Construct -b-, for parallel beam + bottom reflection + thermal emission at top and/or bottom
   *
   * Top boundary, right-hand-side of eq. KST(28)
   */
  lc   = 1;
  irow = 1;
  B(irow) = -YB_0D(lc)-YP_0D(lc)+ds->bc.fisot+tplanck;
  /*
   * Continuity condition for layer interfaces, right-hand-side of eq. KST(29)
   */
  for (lc = 1; lc <= nloop; lc++) {
    fact1     = exp(-ZB_A(lc+1)*TAUCPR(lc));
    fact2     = exp(-ZP_A(lc+1)*TAUCPR(lc));
    fact3     = exp(-ZB_A(lc  )*TAUCPR(lc));
    fact4     = exp(-ZP_A(lc  )*TAUCPR(lc));
    rpp1_m    = fact1*(YB_0D(lc+1)+YB_1D(lc+1)*TAUCPR(lc))+fact2*(YP_0D(lc+1)+YP_1D(lc+1)*TAUCPR(lc));
    rp_m      = fact3*(YB_0D(lc  )+YB_1D(lc  )*TAUCPR(lc))+fact4*(YP_0D(lc  )+YP_1D(lc  )*TAUCPR(lc));
    rpp1_p    = fact1*(YB_0U(lc+1)+YB_1U(lc+1)*TAUCPR(lc))+fact2*(YP_0U(lc+1)+YP_1U(lc+1)*TAUCPR(lc));
    rp_p      = fact3*(YB_0U(lc  )+YB_1U(lc  )*TAUCPR(lc))+fact4*(YP_0U(lc  )+YP_1U(lc  )*TAUCPR(lc));
    B(++irow) = rpp1_p-rp_p-RR(lc+1)*(rpp1_m-rp_m);
    B(++irow) = rpp1_m-rp_m-RR(lc  )*(rpp1_p-rp_p);
  }
  /*
   * Bottom boundary
   */
  lc = ncut;
  if (lyrcut) {
    /*
     * Right-hand-side of eq. KST(30)
     */
    B(++irow) = -exp(-ZB_A(ncut)*TAUCPR(ncut))*(YB_0U(ncut)+YB_1U(ncut)*TAUCPR(ncut))
                -exp(-ZP_A(ncut)*TAUCPR(ncut))*(YP_0U(ncut)+YP_1U(ncut)*TAUCPR(ncut));
  }
  else {
    sum = cmu*ds->bc.albedo*(exp(-ZB_A(ncut)*TAUCPR(ncut))*(YB_0D(ncut)+YB_1D(ncut)*TAUCPR(ncut))
                            +exp(-ZP_A(ncut)*TAUCPR(ncut))*(YP_0D(ncut)+YP_1D(ncut)*TAUCPR(ncut)));
   if (ds->bc.umu0 <= 0.) {
     refflx = 0.;
   }
   else {
     refflx = 1.;
   }
   B(++irow) = 2.*sum+ds->bc.albedo*ds->bc.umu0*ds->bc.fbeam/M_PI*refflx*EXPBEA(ncut)+(1.-ds->bc.albedo)*bplanck
               -exp(-ZB_A(ncut)*TAUCPR(ncut))*(YB_0U(ncut)+YB_1U(ncut)*TAUCPR(ncut))
               -exp(-ZP_A(ncut)*TAUCPR(ncut))*(YP_0U(ncut)+YP_1U(ncut)*TAUCPR(ncut));

 }
 /*
  * solve for constants of integration by inverting matrix KST(38-41)
  */
  nrow = irow;

  /*
   * NOTE: If not allocating memory with swappablecalloc(), need to zero out cband here.
   */

  for (irow = 1; irow <= nrow; irow++) {
    CBAND(1,irow) = 0.;
    CBAND(3,irow) = DIAG(irow);
  }
  for (irow = 1; irow <= nrow-1; irow++) {
    CBAND(2,irow+1) = SUPERD(irow);
  }
  for (irow = 2; irow <= nrow; irow++) {
    CBAND(4,irow-1) = SUBD(irow);
  }

  c_sgbfa(cband,(9*(ds->nstr/2)-2),nrow,1,1,ipvt,&info);
  job = 0;
  c_sgbsl(cband,(9*(ds->nstr/2)-2),nrow,1,1,ipvt,b,job);

  /*
   * unpack
   */
  irow = 0;
  for (lc = 1; lc <= ncut; lc++) {
    /* downward direction */
    LL(1,lc) = B(++irow);

    /* upward direction */
    LL(2,lc) = B(++irow);
  }

  return;
}

/*============================= end of c_twostr_solve_bc() ===============*/
