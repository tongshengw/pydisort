#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
#undef  KK
#define KK(lyu) kk[lyu-1]
/*============================= c_inter() ===============================*/

/*-------------------------------------------------------------------
 * Copyright (C) 1994 Arve Kylling
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 1, or (at your option)
 * any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY of FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * To obtain a copy of the GNU General Public License write to the
 * Free Software Foundation, Inc., 675 Mass Ave, Cambridge, MA 02139,
 * USA.
 *-------------------------------------------------------------------
 *
 *     Interpolates at the x-point arg from x-value array xarr and
 *     y-value array yarr. xarr and yarr are expected to have
 *     descending arguments, i.e. for atmospheric applications
 *     xarr typically holds the altitude and inter expects
 *     xarr(1) = altitude at top of atmosphere.
 *
 *     Input variables:
 *     dim       Array dimension of xarr and yarr
 *     npoints   No. points in arrays xarr and yarr
 *     itype     Interpolation type
 *     arg       Interpolation argument
 *     xarr      array of x values
 *     yarr      array of y values
 *
 *     Output variables:
 *     ynew      Interpolated function value at arg
 *     hh        gradient or scale height value
 *
 * This code was translated to c from fortran by Robert Buras
 *
 */

double c_inter( int     npoints,
		int     itype,
		double  arg,
		float  *xarr,
		double *yarr,
		double *hh )
{
  int iq=0, ip=0;
  double ynew=0.0;

  if ( arg <= XARR (1) && arg >= XARR (npoints) ) {
    for (iq=1;iq<=npoints-1;iq++)
      if ( arg <= XARR (iq) && arg >= XARR (iq+1) )
	ip=iq;
    if ( arg == XARR (npoints) )
      ip = npoints - 1;
  }
  else {
    if ( arg > XARR (1) )
      ip = 1;
    else {
      if ( arg < XARR (npoints) )
	ip = npoints - 1;
    }
  }

  /* Interpolate function value at arg from data points ip to ip+1 */

  switch(itype) {
  case 1:
    /*     exponential interpolation */
    if ( YARR (ip+1) == YARR (ip) ) {
      *hh = 0.0;
      ynew = YARR (ip);
    }
    else {
      *hh = -( XARR (ip+1) - XARR (ip) ) /
	log( YARR (ip+1) / YARR (ip));
      ynew = YARR (ip) * exp(- ( arg - XARR (ip) ) / *hh );
    }
    break;
  case 2:
    /*     linear interpolation */
    *hh = ( YARR (ip+1) - YARR (ip) ) / ( XARR (ip+1) - XARR (ip) );
    ynew = YARR (ip) + *hh * ( arg - XARR (ip) );
    break;
  default:
    printf("Error, unknown itype %d (line %d, function '%s' in '%s')\n",
	     itype, __LINE__, __func__, __FILE__);
    return -999.0;
  }

  return ynew;
}

/*============================= end of c_inter() ========================*/
