#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_print_albtrans() =======================*/

/*
   Print planar albedo and transmissivity of medium as a function of
   incident beam angle

   Called by- c_albtrans
 --------------------------------------------------------------------*/

void c_print_albtrans(disort_state  *ds,
                      disort_output *out)
{
  register int
    iu;

  printf("\n\n\n *******  Flux Albedo and/or Transmissivity of entire medium  ********\n");
  printf(" Beam Zen Ang   cos(Beam Zen Ang)      Albedo   Transmissivity\n");
  for (iu = 1; iu <= ds->numu; iu++) {
    printf("%13.4f%20.6f%12.5f%17.4e\n",acos(UMU(iu))/DEG,UMU(iu),ALBMED(iu),TRNMED(iu));
  }

  return;
}

/*============================= end of c_print_albtrans() ================*/
