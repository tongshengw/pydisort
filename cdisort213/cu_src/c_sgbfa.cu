#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_sgbfa() ================================*/

/*
    Factors a real band matrix by elimination.
    Revision date:  8/1/82
    Author:  Moler, C. B. (U. of New Mexico)
    c_sgbfa is usually called by c_sgbco, but it can be called
    directly with a saving in time if rcond is not needed.

    Inputs:  same as c_sgbco
    Outputs:
        abd,ipvt    same as c_sgbco
        info    int,
                = 0  normal value.
                = k  if  u(k,k) == 0.  This is not an error
                     condition for this subroutine, but it does
                     indicate that sgbsl will divide by zero if
                     called.  Use  rcond  in c_sgbco for a reliable
                     indication of singularity.
    (see c_sgbco for description of band storage mode)

    NOTE: using memset() to zero columns in abd
 ----------------------------------------------------------------*/

void c_sgbfa(double *abd,
             int     lda,
             int     n,
             int     ml,
             int     mu,
             int    *ipvt,
             int    *info)
{
  register int
    i0,j,j0,j1,ju,jz,k,kp1,l,lm,m,mm,nm1;
  double
    t;

  m     = ml+mu+1;
  *info = 0;
  /*
   * zero initial fill-in columns
   */
  j0 = mu+2;
  j1 = IMIN(n,m)-1;
  for (jz = j0; jz <= j1; jz++) {
    i0 = m+1-jz;
    memset(&ABD(i0,jz),0,(ml-i0+1)*sizeof(double));
  }
  jz = j1;
  ju = 0;

  /*
   * Gaussian elimination with partial pivoting
   */
  nm1 = n-1;
  for (k = 1; k <= nm1; k++) {
    kp1 = k+1;
   /*
    * zero next fill-in column
    */
    jz++;
    if (jz <= n) {
      memset(&ABD(1,jz),0,ml*sizeof(double));
    }
    /*
     * find L = pivot index
     */
    lm      = IMIN(ml,n-k);
    l       = c_isamax(lm+1,&ABD(m,k))+m-1;
    IPVT(k) = l+k-m;
    if (ABD(l,k) == 0.) {
     /*
      * zero pivot implies this column already triangularized
      */
      *info = k;
    }
    else {
      /*
       * interchange if necessary
       */
      if (l != m) {
        t        = ABD(l,k);
        ABD(l,k) = ABD(m,k);
        ABD(m,k) = t;
      }
      /*
       * compute multipliers
       */
      t = -1./ABD(m,k);
      c_sscal(lm,t,&ABD(m+1,k));
      /*
       * row elimination with column indexing
       */
      ju = IMIN(IMAX(ju,mu+IPVT(k)),n);
      mm = m;
      for (j = kp1; j <= ju; j++) {
        l--;
        mm--;
        t = ABD(l,j);
        if (l != mm) {
          ABD(l,j)  = ABD(mm,j);
          ABD(mm,j) = t;
        }
        c_saxpy(lm,t,&ABD(m+1,k),&ABD(mm+1,j));
      }
    }
  }
  IPVT(n) = n;
  if (ABD(m,n) == 0.) {
    *info = n;
  }

  return;
}

/*============================= end of c_sgbfa() =========================*/
