#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_intensity_components() ================*/

/*
    Calculates the Fourier intensity components at the quadrature
    angles for azimuthal expansion terms (mazim) in eq. SD(2),STWL(6)

    I N P U T    V A R I A B L E S:

       ds      :  Disort state variables
       kk      :  Eigenvalues of coeff. matrix in eq. SS(7), STWL(23b)
       gc      :  Eigenvectors at polar quadrature angles in eq. SC(1)
       ll      :  Constants of integration in eq. SC(1), obtained by solving scaled version of eq. SC(5);
                  exponential term of eq. SC(12) not included
       lyrcut  :  Logical flag for truncation of computational layer
       mazim   :  Order of azimuthal component
       ncut    :  Number of computational layer where absorption optical depth exceeds ABSCUT
       nn      :  Order of double-Gauss quadrature (NSTR/2)
       taucpr  :  Cumulative optical depth (delta-M-scaled)
       utaupr  :  Optical depths of user output levels in delta-M coordinates;  equal to UTAU if no delta-M
       zz      :  Beam source vectors in eq. SS(19), STWL(24b)
       plk     :  Thermal source vectors z0,z1 by solving eq. SS(16), Y-sub-zero, Y-sub-one in STWL(26ab);
                  plk[].zero, plk[].one (see cdisort.h)

    O U T P U T   V A R I A B L E S:

       uum     :  Fourier components of the intensity in eq. SD(12) (at polar quadrature angles)

    I N T E R N A L   V A R I A B L E S:

       fact    :  exp(-utaupr/umu0)
       zint    :  intensity of m=0 case, in eq. SC(1)

   Called by- c_disort
 -------------------------------------------------------------------*/

void c_intensity_components(disort_state *ds,
                            double       *gc,
                            double       *kk,
                            int          *layru,
                            double       *ll,
                            int           lyrcut,
                            int           mazim,
                            int           ncut,
                            int           nn,
                            double       *taucpr,
                            double       *utaupr,
                            double       *zz,
                            disort_pair  *plk,
                            double       *uum)
{
  register int
    iq,jq,lu,lyu;
  register double
    zint;

  /*
   * Loop over user levels
   */
  for (lu = 1; lu <= ds->ntau; lu++) {
    lyu = LAYRU(lu);
    if (lyrcut && lyu > ncut) {
      continue;
    }
    for (iq = 1; iq <= ds->nstr; iq++) {
      zint = 0.;
      for (jq = 1; jq <= nn; jq++) {
        zint += GC(iq,jq,lyu)*LL(jq,lyu)*exp(-KK(jq,lyu)*(UTAUPR(lu)-TAUCPR(lyu  )));
      }
      for (jq = nn+1; jq <=ds->nstr; jq++) {
        zint += GC(iq,jq,lyu)*LL(jq,lyu)*exp(-KK(jq,lyu)*(UTAUPR(lu)-TAUCPR(lyu-1)));
      }
      UUM(iq,lu) = zint;
      if (ds->bc.fbeam > 0.) {
        UUM(iq,lu) = zint+ZZ(iq,lyu)*exp(-UTAUPR(lu)/ds->bc.umu0);
      }
      if (ds->flag.planck && mazim == 0) {
        UUM(iq,lu) += ZPLK0(iq,lyu)+ZPLK1(iq,lyu)*UTAUPR(lu);
      }
    }
  }

  return;
}

/*============================= end of c_intensity_components() =========*/
