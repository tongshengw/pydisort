#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_sgbco() ================================*/

/*
     Factors a real band matrix by Gaussian elimination and estimates the
     condition of the matrix.
     Revision date:  8/1/82
     Author:  Moler, C.B. (Univ. of New Mexico)

     If  RCOND  is not needed, sgbfa is slightly faster.
     To solve  A*X = B , follow sgbco by sgbsl.

     Inputs:
        abd     double(LDA,N), contains the matrix in band storage.
                The columns of the matrix are stored in the columns of abd
                and the diagonals of the matrix are stored in rows
                ml+1 through 2*ml+mu+1 of  abd.
                See the comments below for details.
        lda     int, the leading dimension of the array abd.
                lda must be >= 2*ml+mu+1.
        n       int,the order of the original matrix.
        ml      int, number of diagonals below the main diagonal.
                0 <= ml < n.
        mu      int, number of diagonals above the main diagonal.
                0 <= mu < n.
                more efficient if  ml <= mu.

     Outputs:
        abd     an upper triangular matrix in band storage and
                the multipliers which were used to obtain it.
                The factorization can be written  A = L*U  where
                L  is a product of permutation and unit lower
                triangular matrices and  U  is upper triangular.
        ipvt    int[n], an integer vector of pivot indices.
        rcond   double, an estimate of the reciprocal condition of A.
                For the system  A*X = B, relative perturbations
                in A and B of size epsilon may cause relative
                perturbations in  X  of size  epsilon/rcond.
                If rcond  is so small that the logical expression
                   1.+RCOND == 1.
                is true, then  A  may be singular to working
                precision.  In particular, rcond is zero if exact
                singularity is detected or the estimate underflows.
        z       double[n], a work vector whose contents are usually
                unimportant. If A is close to a singular matrix, then
                z is an approximate null vector in the sense that
                norm(a*z) = rcond*norm(a)*norm(z).

     Band storage:
           If A is a band matrix, the following program segment
           will set up the input (with unit-offset arrays):
                   ml = (band width below the diagonal)
                   mu = (band width above the diagonal)
                   m = ml+mu+1
                   for (j = 1; j <= n; j++) {
                     i1 = IMAX(1,j-mu);
                     i2 = IMIN(n,j+ml);
                     for (i = i1; i <= i2; i++) {
                       k = i-j+m;
                       ABD(K,J) = A(I,J);
                     }
                   }
           This uses rows ml+1 through 2*ml+mu+1 of abd.
           In addition, the first ml rows in abd are used for
           elements generated during the triangularization.
           The total number of rows needed in abd is 2*ml+mu+1.
           The ml+mu by ml+mu upper left triangle and the
           ml by ml lower right triangle are not referenced.

     Example:  if the original matrix is

           11 12 13  0  0  0
           21 22 23 24  0  0
            0 32 33 34 35  0
            0  0 43 44 45 46
            0  0  0 54 55 56
            0  0  0  0 65 66

      then  n = 6, ml = 1, mu = 2, lda >= 5  and abd should contain
            *  *  *  +  +  +  , * = not used
            *  * 13 24 35 46  , + = used for pivoting
            * 12 23 34 45 56
           11 22 33 44 55 66
           21 32 43 54 65  *

 --------------------------------------------------------------------*/

void c_sgbco(double *abd,
             int     lda,
             int     n,
             int     ml,
             int     mu,
             int    *ipvt,
             double *rcond,
             double *z)
{
  int
    info;
  register int
    is,j,ju,k,kb,kp1,l,la,lm,lz,m,mm;
  double
    anorm,ek,s,sm,t,wk,wkm,ynorm;

  /*
   * compute 1-norm of A
   */
  anorm = 0.;
  l  = ml+1;
  is = l+mu;
  for (j = 1; j <= n; j++) {
    anorm = MAX(anorm,c_sasum(l,&ABD(is,j)));
    if (is > ml+1) {
      is--;
    }
    if (j <= mu) {
      l++;
    }
    if (j >= n-ml) {
      l--;
    }
  }
  /*
   * factor
   */
  c_sgbfa(abd,lda,n,ml,mu,ipvt,&info);
  /*
   * rcond = 1/(norm(A)*(estimate of norm(inverse(A)))) .
   * estimate = norm(Z)/norm(Y) where  A*Z = Y  and  trans(A)*Y = E.
   * trans(A) is the transpose of A.  The components of E are
   * chosen to cause maximum local growth in the elements of W where
   * trans(U)*W = E. The vectors are frequently rescaled to avoid overflow.
   * solve trans(U)*W = E
   */
  ek = 1.;

  memset(z,0,n*sizeof(double));

  m  = ml+mu+1;
  ju = 0;
  for (k = 1; k <= n; k++) {
    if (Z(k) != 0.) {
      ek = F77_SIGN(ek,-Z(k));
    }
    if (fabs(ek-Z(k)) > fabs(ABD(m,k))) {
      s = fabs(ABD(m,k))/fabs(ek-Z(k));
      c_sscal(n,s,z);
      ek *= s;
    }
    wk  =  ek-Z(k);
    wkm = -ek-Z(k);
    s   = fabs(wk);
    sm  = fabs(wkm);
    if (ABD(m,k) != 0.) {
      wk  /= ABD(m,k);
      wkm /= ABD(m,k);
    }
    else {
      wk  = 1.;
      wkm = 1.;
    }
    kp1 = k+1;
    ju  = IMIN(IMAX(ju,mu+IPVT(k)),n);
    mm  = m;
    if (kp1 <= ju) {
      for (j = kp1; j <= ju; j++) {
        mm--;
        sm   += fabs(Z(j)+wkm*ABD(mm,j));
        Z(j) += wk*ABD(mm,j);
        s    += fabs(Z(j));
      }
      if (s < sm) {
        t  = wkm-wk;
        wk = wkm;
        mm = m;
        for (j = kp1; j <= ju; j++) {
          mm--;
          Z(j) += t*ABD(mm,j);
        }
      }
    }
    Z(k) = wk;
  }

  s = 1./c_sasum(n,z);
  c_sscal(n,s,z);

  /*
   * solve trans(L)*Y = W
   */
  for (kb = 1; kb <= n; kb++) {
    k  = n+1-kb;
    lm = IMIN(ml,n-k);
    if (k < n) {
      Z(k) += c_sdot(lm,&ABD(m+1,k),&Z(k+1));
    }
    if (fabs(Z(k)) > 1.) {
      s = 1./fabs(Z(k));
      c_sscal(n,s,z);
    }

    l    = IPVT(k);
    t    = Z(l);
    Z(l) = Z(k);
    Z(k) = t;
  }

  s = 1./c_sasum(n,z);
  c_sscal(n,s,z);

  ynorm = 1.;
  /*
   * solve L*V = Y
   */
  for (k = 1; k <= n; k++) {
    l    = IPVT(k);
    t    = Z(l);
    Z(l) = Z(k);
    Z(k) = t;
    lm   = IMIN(ml,n-k);
    if (k < n) {
      c_saxpy(lm,t,&ABD(m+1,k),&Z(k+1));
    }
    if (fabs(Z(k)) > 1.) {
      s = 1./fabs(Z(k));
      c_sscal(n,s,z);
      ynorm *= s;
    }
  }

  s = 1./c_sasum(n,z);
  c_sscal(n,s,z);

  ynorm *= s;
  /*
   * solve  U*Z = W
   */
  for (kb = 1; kb <= n; kb++) {
    k = n+1-kb;
    if (fabs(Z(k)) > fabs(ABD(m,k))) {
      s = fabs(ABD(m,k))/fabs(Z(k));
      c_sscal(n,s,z);
      ynorm *= s;
    }
    if (ABD(m,k) != 0.) {
      Z(k) /= ABD(m,k);
    }
    else {
      Z(k) = 1.;
    }
    lm = IMIN(k,m)-1;
    la = m-lm;
    lz = k-lm;
    t  = -z[k-1];
    c_saxpy(lm,t,&ABD(la,k),&Z(lz));
  }

  /*
   * make znorm = 1.
   */
  s = 1./c_sasum(n,z);
  c_sscal(n,s,z);

  ynorm *= s;
  if(anorm != 0.) {
    *rcond = ynorm/anorm;
  }
  else {
    *rcond = 0.;
  }

  return;
}

/*============================= end of c_sgbco() =========================*/
