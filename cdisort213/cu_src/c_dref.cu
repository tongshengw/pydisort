#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_dref() ================================*/

/*
  Flux albedo for given angle of incidence, given a bidirectional reflectivity.

  INPUTS
    wvnmlo    :  Lower wavenumber (inv-cm) of spectral interval
    wvnmhi    :  Upper wavenumber (inv-cm) of spectral interval
    mu        :  Cosine of incidence angle
    brdf_type :  BRDF type
    brdf      :  pointer to disort_brdf structure
    callnum   :  number of surface calls

  INTERNAL VARIABLES

    gmu    : The NMUG angle cosine quadrature points on (0,1)
             NMUG is set in cdisort.h
    gwt    : The NMUG angle cosine quadrature weights on (0,1)

   Called by- c_check_inputs
   Calls- c_gaussian_quadrature, c_errmsg, c_bidir_reflectivity
 --------------------------------------------------------------------*/

double c_dref(double       wvnmlo,
              double       wvnmhi,
              double       mu,
	      int          brdf_type,
	      disort_brdf *brdf,
	      int          callnum )
{
  static int
    pass1 = TRUE;
  register int
    jg,k;
  double
    ans,sum;
  static double
    gmu[NMUG],gwt[NMUG];

  if (pass1) {
    pass1 = FALSE;
    c_gaussian_quadrature(NMUG/2,gmu,gwt);
    for (k = 1; k <= NMUG/2; k++) {
      GMU(k+NMUG/2) = -GMU(k);
      GWT(k+NMUG/2) =  GWT(k);
    }
  }

  if (fabs(mu) > 1.) {
    c_errmsg("dref--input argument error(s)",DS_ERROR);
  }

  ans = 0.;
  /*
   * Loop over azimuth angle difference
   */
  for (jg = 1; jg <= NMUG; jg++) {
    /*
     * Loop over angle of reflection
     */
    sum = 0.;
    for (k = 1; k <= NMUG/2; k++) {
      sum += GWT(k) * GMU(k) *
	c_bidir_reflectivity ( wvnmlo, wvnmhi, GMU(k), mu, M_PI*GMU(jg), brdf_type, brdf, callnum );
    }
    ans += GWT(jg)*sum;
  }
  if (ans < 0. || ans > 1.) {
    c_errmsg("DREF--albedo value not in [0,1]",DS_WARNING);
  }

  return ans;
}

/*============================= end of c_dref() =========================*/
