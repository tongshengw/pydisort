#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
#undef  KK
#define KK(lyu) kk[lyu-1]
/*============================= c_twostr_set() ===========================*/

/*
 Perform miscellaneous setting-up operations

 Routines called: c_errmsg

 Input :  ds         'Disort' input variables

 Output:  ntau,utau  If ds->flag.usrtau = FALSE
          bplanck    Intensity emitted from bottom boundary
          ch         The Chapman factor
          cmu        Computational polar angle
          expbea     Transmission of direct beam
          flyr       Truncated fraction in delta-m method
          layru      Computational layer in which utau falls
          lyrcut     Flag as to whether radiation will be zeroed below layer ncut
          ncut       Computational layer where absorption optical depth first exceeds abscut
          nn         nstr/2 = 1
          nstr       No.of streams (=2)
          oprim      Delta-m-scaled single-scatter albedo
          pkag,c     Planck function in each layer
          taucpr     Delta-m-scaled optical depth
          tplanck    Intensity emitted from top boundary
          utaupr     Delta-m-scaled version of utau

 Internal Variables
          abscut     Absorption optical depth, medium is cut off below this depth
          tempc      Temperature at center of layer, assumed to be average of
                     layer boundary temperatures
  ---------------------------------------------------------------------*/

void c_twostr_set(disort_state *ds,
                  double       *bplanck,
                  double       *ch,
                  double       *chtau,
                  double       *cmu,
                  int           deltam,
                  double       *dtaucpr,
                  double       *expbea,
                  double       *flyr,
                  double       *gg,
                  double       *ggprim,
                  int          *layru,
                  int          *lyrcut,
                  int          *ncut,
                  int          *nn,
                  double       *oprim,
                  double       *pkag,
                  double       *pkagc,
                  double        radius,
                  double       *tauc,
                  double       *taucpr,
                  double       *tplanck,
                  double       *utaupr,
                  emission_func_t emi_func)
{
  static int
    firstpass = TRUE;
  register int
    lc,lu,lev;
  double
    zenang,abstau,chtau_tmp,f,tempc,taup,
    abscut = 10.;

  if (firstpass) {
    firstpass = FALSE;
    ds->nstr  = 2;
    *nn       = ds->nstr/2;
  }

  if (!ds->flag.usrtau) {
    /*
     * Set output levels at computational layer boundaries
     */
    ds->ntau = ds->nlyr+1;
    for (lc = 0; lc <= ds->ntau-1; lc++) {
      UTAU(lc+1) = TAUC(lc);
    }
  }
  /*
   * Apply delta-m scaling and move description of computational layers to local variables
   */

  /*
   * NOTE: If not using swappablecalloc() to dynamically allocate memory, then need to zero-out
   *       taucpr, expbea, flyr, oprim here.
   */

  abstau = 0.;
  for (lc = 1; lc <= ds->nlyr; lc++) {
    if (abstau < abscut) {
      *ncut = lc;
    }
    abstau += (1.-SSALB(lc))*DTAUC(lc);
    if (!deltam) {
      OPRIM(lc)   = SSALB(lc);
      TAUCPR(lc)  = TAUC(lc);
      f           = 0.;
      GGPRIM(lc)  = GG(lc);
      DTAUCPR(lc) = DTAUC(lc);
    }
    else {
     /*
      * Do delta-m transformation eqs. WW(20a,20b,14)
      */
      f           = SQR(GG(lc));
      TAUCPR(lc)  = TAUCPR(lc-1)+(1.-f*SSALB(lc))*DTAUC(lc);
      OPRIM(lc)   = SSALB(lc)*(1.-f)/(1.-f*SSALB(lc));
      GGPRIM(lc)  = (GG(lc)-f)/(1.-f);
      DTAUCPR(lc) = TAUCPR(lc)-TAUCPR(lc-1);
    }
    FLYR(lc) = f;
  }
  /*
   * If no thermal emission, cut off medium below absorption optical
   * depth = abscut (note that delta-m transformation leaves absorption
   * optical depth invariant). Not worth the trouble for one-layer problems, though.
   */
  *lyrcut = FALSE;
  if (abstau >= abscut && !ds->flag.planck && ds->nlyr > 1) {
    *lyrcut = TRUE;
  }
  if (!*lyrcut) {
    *ncut = ds->nlyr;
  }
  /*
   * Calculate Chapman function if spherical geometry, set expbea and ch for beam source.
   */
  if (ds->bc.fbeam > 0.) {
    CHTAU(0) = 0.;
    EXPBEA(0) = 1.;
    zenang    = acos(ds->bc.umu0)/DEG;

    if(ds->flag.spher == TRUE && ds->bc.umu0 < 0.) {
      EXPBEA(0) = exp(-c_chapman(1,0.,tauc,ds->nlyr,ds->zd,ds->dtauc,zenang,radius));
    }
    if (ds->flag.spher == TRUE) {
      for (lc = 1; lc <= *ncut; lc++) {
        taup        = TAUCPR(lc-1)+DTAUCPR(lc)/2.;
        CHTAU(lc  ) = c_chapman(lc, 0.0,      taucpr,ds->nlyr,ds->zd,dtaucpr,zenang,radius);
        chtau_tmp   = c_chapman(lc, 0.5,taucpr,ds->nlyr,ds->zd,dtaucpr,zenang,radius);
        CH(lc)      = taup/chtau_tmp;
        EXPBEA(lc)  = exp(-CHTAU(lc));
      }
    }
    else {
      for (lc = 1; lc <= *ncut; lc++) {
        CH(lc)     = ds->bc.umu0;
        EXPBEA(lc) = exp(-TAUCPR(lc)/ds->bc.umu0);
      }
    }
  }
  /*
   * Set arrays defining location of user output levels within delta-m-scaled computational mesh
   */
  for (lu = 1; lu <= ds->ntau; lu++) {
    for (lc = 1; lc <= ds->nlyr-1; lc++) {
      if (UTAU(lu) >= TAUC(lc-1) && UTAU(lu) <= TAUC(lc)) {
        break;
      }
    }
    UTAUPR(lu) = UTAU(lu);
    if (deltam) {
      UTAUPR(lu) = TAUCPR(lc-1)+(1.-SSALB(lc)*FLYR(lc))*(UTAU(lu)-TAUC(lc-1));
    }
    LAYRU(lu) = lc;
  }

  /*
   * Set computational polar angle cosine for double gaussian
   * quadrature; cmu = 0.5, or  single gaussian quadrature; cmu = 1./sqrt(3
   * See KST for discussion of which is better for your specific applicatio
   */
  if(ds->flag.planck && ds->bc.fbeam == 0.) {
    *cmu = 0.5;
  }
  else {
    *cmu = sqrt(1./3.);
  }
  /*
   * Calculate planck functions
   */
  if (!ds->flag.planck) {
    *bplanck = 0.;
    *tplanck = 0.;
    /*
     * NOTE: If not using swappablecalloc() for dynamic memory allocation, need to zero-out
     *       pkag and pkagc here.
     */
  }
  else {
    *tplanck = emi_func(ds->wvnmlo,ds->wvnmhi,ds->bc.ttemp)*ds->bc.temis;
    *bplanck = emi_func(ds->wvnmlo,ds->wvnmhi,ds->bc.btemp);
    for (lev = 0; lev <= ds->nlyr; lev++) {
      PKAG(lev) = emi_func(ds->wvnmlo,ds->wvnmhi,TEMPER(lev));
    }
    for (lc = 1; lc <=ds->nlyr; lc++) {
      tempc     = .5*(TEMPER(lc-1)+TEMPER(lc));
      PKAGC(lc) = emi_func(ds->wvnmlo,ds->wvnmhi,tempc);
    }
  }

  return;
}

/*============================= end of c_twostr_set() ====================*/
