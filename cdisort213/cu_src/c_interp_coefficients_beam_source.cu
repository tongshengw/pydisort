#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_interp_coefficients_beam_source() =======*/

/*
     Find coefficients at user angle, necessary for later use in
     c_interp_source()
*/

/*

    I N P U T      V A R I A B L E S:

       cmu    :   Computational polar angles
       chtau  :   The optical depth in spherical geometry.
       delmo  :   Kronecker delta, delta-sub-m0
       fbeam  :   incident beam radiation at top
       gl     :   Phase function Legendre coefficients multiplied by (2l+1) and single-scatter albedo
       lc:    :   layer index
       mazim  :   order of azimuthal component
       nstr   :   number of streams
       numu   :   number of user angles
       taucpr :   delta-m-scaled optical depth
       xba    :   alfa in eq. KS(7)
       ylmu   :   Normalized associated Legendre polynomial at the user angles -umu-
       ylm0   :   Normalized associated Legendre polynomial at the beam angle

    O U T P U T     V A R I A B L E S:

       zb0u   :   x-sub-zero in KS(7) at user angles -umu-
       zb1u   :   x-sub-one in KS(7) at user angles -umu-
       zju    :  Solution vector Z-sub-zero after solving eq. SS(19), STWL(24b), at user angles -umu-

   Called by- c_disort

*/

void c_interp_coefficients_beam_source(disort_state   *ds,
				       double         *chtau,
				       double          delm0,
				       double          fbeam,
				       double         *gl,
				       int             lc,
				       int             mazim,
				       int             nstr,
				       int             numu,
				       double         *taucpr,
				       disort_triplet *zbu,
				       double         *xba,
				       double         *zju,
				       double         *ylm0,
				       double         *ylmu)
{
  register int
    iu,k;
  double
    deltat,sum,q0a,q2a,q0,q2;

  /*     Calculate x-sub-zero in STWJ(6d) */
  deltat = TAUCPR(lc) - TAUCPR(lc-1);

  q0a = exp(-CHTAU(lc-1));
  q2a = exp(-CHTAU(lc));

  for (iu = 1; iu <= numu; iu++) {
    sum = 0.0;
    for (k = mazim; k <= nstr-1; k++) {
      sum = sum + GL(k,lc)*YLMU(k,iu)*YLM0(k);
    }
    ZJU(iu) = (2.0-delm0)*fbeam*sum/(4.0*M_PI);
  }

  for (iu = 1; iu <= numu; iu++) {

    q0 = q0a*ZJU(iu);
    q2 = q2a*ZJU(iu);

    /*     x-sub-zero and x-sub-one in Eqs. KS(48-49)   */

    ZB1U(iu,lc)=(1./deltat)*(q2*exp(XBA(lc)*TAUCPR(lc))
			     -q0*exp(XBA(lc)*TAUCPR(lc-1)));
    ZB0U(iu,lc) = q0*exp(XBA(lc)*TAUCPR(lc-1))-ZB1U(iu,lc)*TAUCPR(lc-1);
  }

  return;

}
/*============================= end of c_interp_coefficients_beam_source() =*/
