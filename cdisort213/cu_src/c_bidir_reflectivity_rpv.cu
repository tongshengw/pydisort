#include "hip/hip_runtime.h"
// includes
#include<alloc.h>
#include<cdisort.h>
#include<locate.h>

DISPATCH_MACRO
/*============================= c_bidir_reflectivity_rpv() ==============*/

/*
  Computes the Rahman, Pinty, Verstraete BRDF.  The incident
  and outgoing cosine zenith angles are MU1 and MU2, respectively,
  and the relative azimuthal angle is PHI.  In this case the incident
  direction is where the radiation is coming from, so MU1>0 and
  the hot spot is MU2=MU1 and PHI=180 (the azimuth convention is
  different from the original Frank Evans code).
  The reference is:
  Rahman, Pinty, Verstraete, 1993: Coupled Surface-Atmosphere
  Reflectance (CSAR) Model. 2. Semiempirical Surface Model Usable
  With NOAA Advanced Very High Resolution Radiometer Data,
  J. Geophys. Res., 98, 20791-20801.

  Translated from fortran to C by Robert Buras; original name RPV_REFLECTION

  INPUT:

    rho0   :  BRDF rpv: rho0
    k      :  BRDF rpv: k
    theta  :  BRDF rpv: theta
    sigma  :  BRDF rpv snow: sigma
    t1     :  BRDF rpv snow: t1
    t2     :  BRDF rpv snow: t2
    scale  :  BRDF rpv: scale
    mu1    :  Cosine of angle of reflection (positive)
    mu2    :  Cosine of angle of incidence (positive)
    phi    :  Difference of azimuth angles of incidence and reflection
                 (radians)
    badmu  :  minimally allowed value for mu1 and mu2

  LOCAL VARIABLES:

    ans    :  Return value

   Called by- c_bidir_reflectivity
-------------------------------------------------------------------------*/

double c_bidir_reflectivity_rpv ( rpv_brdf_spec *brdf,
                                  double         mu1,
				  double         mu2,
				  double         phi,
				  double         badmu )
{
  double
    m, f, h, cosphi, sin1, sin2, cosg, tan1, tan2, capg,
    hspot, t, g;
  double ans;

  /* This function needs more checking; some constraints are
     required to avoid albedos larger than 1; in particular,
     the BDREF is limited to 5 times the hotspot value to
     avoid extremely large values at low polar angles */


  /* Azimuth convention different from Frank Evans:
     Here PHI=0 means the backward direction while
     while in DISORT PHI=0 means forward. */
  phi = M_PI - phi;

  /* Don't allow mu's smaller than BADMU because
     the albedo is larger than 1 for those */
  if ( badmu > 0.0 ) {
    if ( mu1 < badmu )
      mu1 = badmu;
    if ( mu2 < badmu )
      mu2 = badmu;
  }

  /* Hot spot */
  hspot = brdf->rho0 * ( pow ( 2.0 * mu1 * mu1 * mu1 , brdf->k - 1.0 ) *
		   ( 1.0 - brdf->theta ) / ( 1.0 + brdf->theta ) / ( 1.0 + brdf->theta )
		   *  ( 2.0 - brdf->rho0 )
		   + brdf->sigma / mu1 ) * ( brdf->t1 * exp ( M_PI * brdf->t2 ) + 1.0 );

  /* Hot spot region */
  /* is this bug??? phi <= 1e-4 would be more sensible ... RPB */
  if (phi == 1e-4 && mu1 == mu2)
    return hspot * brdf->scale;

  m = pow ( mu1 * mu2 * ( mu1 + mu2 ) , brdf->k - 1.0 );
  cosphi = cos(phi);
  sin1 = sqrt ( 1.0 - mu1 * mu1 );
  sin2 = sqrt ( 1.0 - mu2 * mu2 );
  cosg = mu1 * mu2 + sin1 * sin2 * cosphi;
  g = acos ( cosg );
  f = ( 1.0 - brdf->theta * brdf->theta ) /
    pow ( 1.0 + 2.0 * brdf->theta * cosg + brdf->theta * brdf->theta , 1.5);

  tan1 = sin1 / mu1;
  tan2 = sin2 / mu2;
  capg = sqrt( tan1 * tan1 + tan2 * tan2 - 2.0 * tan1 * tan2 * cosphi );
  h = 1.0 + ( 1.0 - brdf->rho0 ) / ( 1.0 + capg );
  t = 1.0 + brdf->t1 * exp ( brdf->t2 * ( M_PI - g ) );

  ans = brdf->rho0 * ( m * f * h + brdf->sigma / mu1 ) * t * brdf->scale;

 if (ans < 0.0)
   ans = 0.0;

 return ans;
}

/*============================= end of c_bidir_reflectivity_rpv() =======*/
